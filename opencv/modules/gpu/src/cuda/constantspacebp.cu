#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/devmem2d.hpp"
#include "saturate_cast.hpp"
#include "safe_call.hpp"

using namespace cv::gpu;
using namespace cv::gpu::impl;

#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38F
#endif

#ifndef SHRT_MAX
#define SHRT_MAX 32767
#endif

template <typename T> 
struct TypeLimits {};

template <> 
struct TypeLimits<short>
{
    static __device__ short max() {return SHRT_MAX;}
};

template <> 
struct TypeLimits<float>
{
    static __device__ float max() {return FLT_MAX;}
};

///////////////////////////////////////////////////////////////
/////////////////////// load constants ////////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{
    __constant__ int cndisp;

    __constant__ float cmax_data_term;
    __constant__ float cdata_weight;
    __constant__ float cmax_disc_term;
    __constant__ float cdisc_single_jump;

    __constant__ size_t cimg_step;
    __constant__ size_t cmsg_step1;
    __constant__ size_t cmsg_step2;
    __constant__ size_t cdisp_step1;
    __constant__ size_t cdisp_step2;

    __constant__ uchar* cleft;
    __constant__ uchar* cright;
    __constant__ uchar* ctemp1;
    __constant__ uchar* ctemp2;
}

namespace cv { namespace gpu { namespace csbp 
{
    void load_constants(int ndisp, float max_data_term, float data_weight, float max_disc_term, float disc_single_jump, 
                        const DevMem2D& left, const DevMem2D& right, const DevMem2D& temp1, const DevMem2D& temp2)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cndisp), &ndisp, sizeof(int)) );

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmax_data_term),    &max_data_term,    sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdata_weight),      &data_weight,      sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmax_disc_term),    &max_disc_term,    sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisc_single_jump), &disc_single_jump, sizeof(float)) );
        
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cimg_step), &left.step, sizeof(size_t)) );

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cleft),  &left.ptr,  sizeof(left.ptr)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cright), &right.ptr, sizeof(right.ptr)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::ctemp1), &temp1.ptr, sizeof(temp1.ptr)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::ctemp2), &temp2.ptr, sizeof(temp2.ptr)) );
    }
}}}

///////////////////////////////////////////////////////////////
/////////////////////// init data cost ////////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{       
    template <int channels> 
    struct DataCostPerPixel
    {
        static __device__ float compute(const uchar* left, const uchar* right)
        {
            float tb = 0.114f * abs((int)left[0] - right[0]);
            float tg = 0.587f * abs((int)left[1] - right[1]);
            float tr = 0.299f * abs((int)left[2] - right[2]);

            return fmin(cdata_weight * (tr + tg + tb), cdata_weight * cmax_data_term);
        }
    };

    template <> 
    struct DataCostPerPixel<1>
    {
        static __device__ float compute(const uchar* left, const uchar* right)
        {
            return fmin(cdata_weight * abs((int)*left - *right), cdata_weight * cmax_data_term);
        }
    };

    template <typename T>
    __global__ void get_first_k_initial_local(T* data_cost_selected_, T* selected_disp_pyr, int h, int w, int nr_plane)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (y < h && x < w)
        {
            T* selected_disparity = selected_disp_pyr + y * cmsg_step1 + x;
            T* data_cost_selected = data_cost_selected_ + y * cmsg_step1 + x;
            T* data_cost = (T*)ctemp1 + y * cmsg_step1 + x;

            int nr_local_minimum = 0;

            T prev = data_cost[0 * cdisp_step1];
            T cur  = data_cost[1 * cdisp_step1];
            T next = data_cost[2 * cdisp_step1];

            for (int d = 1; d < cndisp - 1 && nr_local_minimum < nr_plane; d++)
            {
                if (cur < prev && cur < next)
                {
                    data_cost_selected[nr_local_minimum * cdisp_step1] = cur;
                    selected_disparity[nr_local_minimum * cdisp_step1] = d;
                    
                    data_cost[d * cdisp_step1] = TypeLimits<T>::max();

                    nr_local_minimum++;
                }
                prev = cur;
                cur = next;
                next = data_cost[(d + 1) * cdisp_step1];
            }

            for (int i = nr_local_minimum; i < nr_plane; i++)
            {
                T minimum = TypeLimits<T>::max();
                int id = 0;

                for (int d = 0; d < cndisp; d++)
                {
                    cur = data_cost[d * cdisp_step1];
                    if (cur < minimum)
                    {
                        minimum = cur;
                        id = d;
                    }
                }
                data_cost_selected[i * cdisp_step1] = minimum;
                selected_disparity[i * cdisp_step1] = id;

                data_cost[id * cdisp_step1] = TypeLimits<T>::max();
            }
        }
    }

    template <typename T, int winsz, int channels> 
    __global__ void data_init(int level, int rows, int cols, int h)
    {
        int x_out = blockIdx.x;        
        int y_out = blockIdx.y % h;
        int d = (blockIdx.y / h) * blockDim.z + threadIdx.z;

        int tid = threadIdx.x;

        if (d < cndisp)
        {
            int x0 = x_out << level;
            int y0 = y_out << level;

            int len = min(y0 + winsz, rows) - y0;

            float val = 0.0f;
            if (x0 + tid < cols)
            {
                if (x0 + tid - d < 0)
                    val = cdata_weight * cmax_data_term * len;
                else
                {
                    const uchar* lle =  cleft + y0 * cimg_step + channels * (x0 + tid    );
                    const uchar* lri = cright + y0 * cimg_step + channels * (x0 + tid - d);

                    for(int y = 0; y < len; ++y)
                    {                                                
                        val += DataCostPerPixel<channels>::compute(lle, lri);

                        lle += cimg_step;
                        lri += cimg_step;
                    }
                }
            }

            extern __shared__ float smem[];
            float* dline = smem + winsz * threadIdx.z;

            dline[tid] = val;

            __syncthreads();

            if (winsz >= 256) { if (tid < 128) { dline[tid] += dline[tid + 128]; } __syncthreads(); }
            if (winsz >= 128) { if (tid <  64) { dline[tid] += dline[tid + 64]; } __syncthreads(); }

            if (winsz >= 64) if (tid < 32) dline[tid] += dline[tid + 32];
            if (winsz >= 32) if (tid < 16) dline[tid] += dline[tid + 16];
            if (winsz >= 16) if (tid <  8) dline[tid] += dline[tid + 8];
            if (winsz >=  8) if (tid <  4) dline[tid] += dline[tid + 4];
            if (winsz >=  4) if (tid <  2) dline[tid] += dline[tid + 2];            
            if (winsz >=  2) if (tid <  1) dline[tid] += dline[tid + 1];

            T* data_cost = (T*)ctemp1 + y_out * cmsg_step1 + x_out;

            if (tid == 0)                 
                data_cost[cdisp_step1 * d] = saturate_cast<T>(dline[0]);
        }
    }
}

namespace cv { namespace gpu { namespace csbp 
{
    template <typename T, int winsz> 
    void data_init_caller(int rows, int cols, int h, int w, int level, int ndisp, int channels, const hipStream_t& stream)
    {
        const int threadsNum = 256;
        const size_t smem_size = threadsNum * sizeof(float);
        
        dim3 threads(winsz, 1, threadsNum/winsz);
        dim3 grid(w, h, 1);        
        grid.y *= divUp(ndisp, threads.z);
        
        switch (channels)
        {
        case 1: csbp_kernels::data_init<T, winsz, 1><<<grid, threads, smem_size, stream>>>(level, rows, cols, h); break;
        case 3: csbp_kernels::data_init<T, winsz, 3><<<grid, threads, smem_size, stream>>>(level, rows, cols, h); break;
        default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__);
        }            
    }
      
    typedef void (*DataInitCaller)(int cols, int rows, int w, int h, int level, int ndisp, int channels, const hipStream_t& stream);

    template <typename T>
    void get_first_k_initial_local_caller(const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost_selected, int h, int w, int nr_plane, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(w, threads.x);
        grid.y = divUp(h, threads.y);

        csbp_kernels::get_first_k_initial_local<T><<<grid, threads, 0, stream>>>((T*)data_cost_selected.ptr, (T*)disp_selected_pyr.ptr, h, w, nr_plane);
    }

    typedef void (*GetFirstKInitialLocalCaller)(const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost_selected, int h, int w, int nr_plane, const hipStream_t& stream);

    void init_data_cost(int rows, int cols, const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost_selected,
                        size_t msg_step, int msg_type, int h, int w, int level, int nr_plane, int ndisp, int channels, const hipStream_t& stream)
    {

        static const DataInitCaller data_init_callers[8][9] = 
            {
                {0, 0, 0, 0, 0, 0, 0, 0, 0}, 
                {0, 0, 0, 0, 0, 0, 0, 0, 0}, 
                {0, 0, 0, 0, 0, 0, 0, 0, 0},
                {data_init_caller<short, 1>, data_init_caller<short, 2>, data_init_caller<short, 4>, data_init_caller<short, 8>, 
                 data_init_caller<short, 16>, data_init_caller<short, 32>, data_init_caller<short, 64>, data_init_caller<short, 128>, 
                 data_init_caller<short, 256>},
                {0, 0, 0, 0, 0, 0, 0, 0, 0},
                {data_init_caller<float, 1>, data_init_caller<float, 2>, data_init_caller<float, 4>, data_init_caller<float, 8>, 
                 data_init_caller<float, 16>, data_init_caller<float, 32>, data_init_caller<float, 64>, data_init_caller<float, 128>, 
                 data_init_caller<float, 256>},
                {0, 0, 0, 0, 0, 0, 0, 0, 0}, 
                {0, 0, 0, 0, 0, 0, 0, 0, 0}
            };

        static const GetFirstKInitialLocalCaller get_first_k_initial_local_callers[8] = 
            {
                0, 0, 0,
                get_first_k_initial_local_caller<short>,
                0,
                get_first_k_initial_local_caller<float>,
                0, 0
            };
                
        DataInitCaller data_init_caller = data_init_callers[msg_type][level];
        GetFirstKInitialLocalCaller get_first_k_initial_local_caller = get_first_k_initial_local_callers[msg_type];
        if (!data_init_caller || !get_first_k_initial_local_caller)
            cv::gpu::error("Unsupported message type or levels count", __FILE__, __LINE__);
        
        size_t disp_step = msg_step * h;
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step1), &disp_step, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step1),  &msg_step,  sizeof(size_t)) );

        data_init_caller(rows, cols, h, w, level, ndisp, channels, stream);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );

        get_first_k_initial_local_caller(disp_selected_pyr, data_cost_selected, h, w, nr_plane, stream);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}}}

///////////////////////////////////////////////////////////////
////////////////////// compute data cost //////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{
    template <typename T, int channels>
    __global__ void compute_data_cost(T* selected_disp_pyr, T* data_cost_, int h, int w, int level, int nr_plane)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;        

        if (y < h && x < w)
        {
            int y0 = y << level;
            int yt = (y + 1) << level;
            
            int x0 = x << level;
            int xt = (x + 1) << level;

            T* selected_disparity = selected_disp_pyr + y/2 * cmsg_step2 + x/2;
            T* data_cost = data_cost_ + y * cmsg_step1 + x;

            for(int d = 0; d < nr_plane; d++)
            {
                float val = 0.0f;
                for(int yi = y0; yi < yt; yi++)
                {
                    for(int xi = x0; xi < xt; xi++)
                    {                    
                        int sel_disp = selected_disparity[d * cdisp_step2];
                        int xr = xi - sel_disp;

                        if (xr < 0) 
                            val += cdata_weight * cmax_data_term;
                        else 
                        {
                            const uchar* left_x = cleft + yi * cimg_step + xi * channels;
                            const uchar* right_x = cright + yi * cimg_step + xr * channels;

                            val += DataCostPerPixel<channels>::compute(left_x, right_x);
                        }
                    }
                }
                data_cost[cdisp_step1 * d] = saturate_cast<T>(val);
            }
        }
    }
}

namespace cv { namespace gpu { namespace csbp 
{
    template <typename T> 
    void compute_data_cost_caller(const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost, 
                                  int h, int w, int level, int nr_plane, int channels, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(w, threads.x);
        grid.y = divUp(h, threads.y);

        switch(channels)
        {
        case 1: csbp_kernels::compute_data_cost<T, 1><<<grid, threads, 0, stream>>>((T*)disp_selected_pyr.ptr, (T*)data_cost.ptr, h, w, level, nr_plane); break;
        case 3: csbp_kernels::compute_data_cost<T, 3><<<grid, threads, 0, stream>>>((T*)disp_selected_pyr.ptr, (T*)data_cost.ptr, h, w, level, nr_plane); break;
        default: cv::gpu::error("Unsupported channels count", __FILE__, __LINE__);
        }           
    }
      
    typedef void (*ComputeDataCostCaller)(const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost, 
                           int h, int w, int level, int nr_plane, int channels, const hipStream_t& stream);

    void compute_data_cost(const DevMem2D& disp_selected_pyr, const DevMem2D& data_cost, size_t msg_step1, size_t msg_step2, int msg_type,
                           int h, int w, int h2, int level, int nr_plane, int channels, const hipStream_t& stream)
    {
        static const ComputeDataCostCaller callers[8] = 
            {
                0, 0, 0,
                compute_data_cost_caller<short>,
                0,
                compute_data_cost_caller<float>,
                0, 0
            };

        size_t disp_step1 = msg_step1 * h;
        size_t disp_step2 = msg_step2 * h2;
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step1), &disp_step1, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step2), &disp_step2, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step1),  &msg_step1,  sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step2),  &msg_step2,  sizeof(size_t)) );

        ComputeDataCostCaller caller = callers[msg_type];
        if (!caller)
            cv::gpu::error("Unsopported message type", __FILE__, __LINE__);
        
        caller(disp_selected_pyr, data_cost, h, w, level, nr_plane, channels, stream);
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() ); 
    }
}}}

///////////////////////////////////////////////////////////////
//////////////////////// init message /////////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{
    template <typename T>
    __device__ void get_first_k_element_increase(T* u_new, T* d_new, T* l_new, T* r_new, 
                                                 const T* u_cur, const T* d_cur, const T* l_cur, const T* r_cur,
                                                 T* data_cost_selected, T* disparity_selected_new, T* data_cost_new, 
                                                 const T* data_cost_cur, const T* disparity_selected_cur, 
                                                 int nr_plane, int nr_plane2)
    {
        for(int i = 0; i < nr_plane; i++)
        {
            T minimum = TypeLimits<T>::max();
            int id = 0;
            for(int j = 0; j < nr_plane2; j++)
            {
                T cur = data_cost_new[j * cdisp_step1];
                if(cur < minimum)
                {
                    minimum = cur;
                    id = j;
                }
            }

            data_cost_selected[i * cdisp_step1] = data_cost_cur[id * cdisp_step1];
            disparity_selected_new[i * cdisp_step1] = disparity_selected_cur[id * cdisp_step1];

            u_new[i * cdisp_step1] = u_cur[id * cdisp_step2];
            d_new[i * cdisp_step1] = d_cur[id * cdisp_step2];
            l_new[i * cdisp_step1] = l_cur[id * cdisp_step2];
            r_new[i * cdisp_step1] = r_cur[id * cdisp_step2];

            data_cost_new[id * cdisp_step1] = TypeLimits<T>::max();
        }
    }

    template <typename T>
    __global__ void init_message(T* u_new_, T* d_new_, T* l_new_, T* r_new_, 
                                 const T* u_cur_, const T* d_cur_, const T* l_cur_, const T* r_cur_, 
                                 T* selected_disp_pyr_new, const T* selected_disp_pyr_cur, 
                                 T* data_cost_selected_, T* data_cost_, 
                                 int h, int w, int nr_plane, int h2, int w2, int nr_plane2)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y < h && x < w)
        {            
            const T* u_cur = u_cur_ + min(h2-1, y/2 + 1) * cmsg_step2 + x/2;
            const T* d_cur = d_cur_ + max(0, y/2 - 1)    * cmsg_step2 + x/2;
            const T* l_cur = l_cur_ + y/2                * cmsg_step2 + min(w2-1, x/2 + 1);
            const T* r_cur = r_cur_ + y/2                * cmsg_step2 + max(0, x/2 - 1);

            T* disparity_selected_cur_backup = (T*)ctemp2 + y * cmsg_step1 + x;
            T* data_cost_new = (T*)ctemp1 + y * cmsg_step1 + x;

            const T* disparity_selected_cur = selected_disp_pyr_cur + y/2 * cmsg_step2 + x/2;
            T* data_cost = data_cost_ + y * cmsg_step1 + x;

            for(int d = 0; d < nr_plane2; d++)
            {
                int idx2 = d * cdisp_step2;

                disparity_selected_cur_backup[d * cdisp_step1] = disparity_selected_cur[idx2];                
                T val  = data_cost[d * cdisp_step1] + u_cur[idx2] + d_cur[idx2] + l_cur[idx2] + r_cur[idx2];
                data_cost_new[d * cdisp_step1] = val;
            }

            T* data_cost_selected = data_cost_selected_ + y * cmsg_step1 + x;
            T* disparity_selected_new = selected_disp_pyr_new + y * cmsg_step1 + x;

            T* u_new = u_new_ + y * cmsg_step1 + x;
            T* d_new = d_new_ + y * cmsg_step1 + x;
            T* l_new = l_new_ + y * cmsg_step1 + x;
            T* r_new = r_new_ + y * cmsg_step1 + x;

            u_cur = u_cur_ + y/2 * cmsg_step2 + x/2;
            d_cur = d_cur_ + y/2 * cmsg_step2 + x/2;
            l_cur = l_cur_ + y/2 * cmsg_step2 + x/2;
            r_cur = r_cur_ + y/2 * cmsg_step2 + x/2;

            get_first_k_element_increase(u_new, d_new, l_new, r_new, u_cur, d_cur, l_cur, r_cur,
                                         data_cost_selected, disparity_selected_new, data_cost_new,
                                         data_cost, disparity_selected_cur_backup, nr_plane, nr_plane2);
        }
    }
}

namespace cv { namespace gpu { namespace csbp 
{
    template <typename T>
    void init_message_caller(const DevMem2D& u_new, const DevMem2D& d_new, const DevMem2D& l_new, const DevMem2D& r_new, 
                      const DevMem2D& u_cur, const DevMem2D& d_cur, const DevMem2D& l_cur, const DevMem2D& r_cur, 
                      const DevMem2D& selected_disp_pyr_new, const DevMem2D& selected_disp_pyr_cur, 
                      const DevMem2D& data_cost_selected, const DevMem2D& data_cost, 
                      int h, int w, int nr_plane, int h2, int w2, int nr_plane2, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(w, threads.x);
        grid.y = divUp(h, threads.y);
        
        csbp_kernels::init_message<T><<<grid, threads, 0, stream>>>((T*)u_new.ptr, (T*)d_new.ptr, (T*)l_new.ptr, (T*)r_new.ptr, 
                                                         (const T*)u_cur.ptr, (const T*)d_cur.ptr, (const T*)l_cur.ptr, (const T*)r_cur.ptr,
                                                         (T*)selected_disp_pyr_new.ptr, (const T*)selected_disp_pyr_cur.ptr, 
                                                         (T*)data_cost_selected.ptr, (T*)data_cost.ptr, 
                                                         h, w, nr_plane, h2, w2, nr_plane2);
    }

    typedef void (*InitMessageCaller)(const DevMem2D& u_new, const DevMem2D& d_new, const DevMem2D& l_new, const DevMem2D& r_new, 
                      const DevMem2D& u_cur, const DevMem2D& d_cur, const DevMem2D& l_cur, const DevMem2D& r_cur, 
                      const DevMem2D& selected_disp_pyr_new, const DevMem2D& selected_disp_pyr_cur, 
                      const DevMem2D& data_cost_selected, const DevMem2D& data_cost, 
                      int h, int w, int nr_plane, int h2, int w2, int nr_plane2, const hipStream_t& stream);

    void init_message(const DevMem2D& u_new, const DevMem2D& d_new, const DevMem2D& l_new, const DevMem2D& r_new, 
                      const DevMem2D& u_cur, const DevMem2D& d_cur, const DevMem2D& l_cur, const DevMem2D& r_cur, 
                      const DevMem2D& selected_disp_pyr_new, const DevMem2D& selected_disp_pyr_cur, 
                      const DevMem2D& data_cost_selected, const DevMem2D& data_cost, size_t msg_step1, size_t msg_step2, int msg_type, 
                      int h, int w, int nr_plane, int h2, int w2, int nr_plane2, const hipStream_t& stream)
    {
        static const InitMessageCaller callers[8] = 
            {
                0, 0, 0,
                init_message_caller<short>,
                0,
                init_message_caller<float>,
                0, 0
            };
        
        size_t disp_step1 = msg_step1 * h;
        size_t disp_step2 = msg_step2 * h2;
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step1), &disp_step1, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step2), &disp_step2, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step1),   &msg_step1, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step2),   &msg_step2, sizeof(size_t)) );

        InitMessageCaller caller = callers[msg_type];
        if (!caller)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);

        caller(u_new, d_new, l_new, r_new, u_cur, d_cur, l_cur, r_cur, 
               selected_disp_pyr_new, selected_disp_pyr_cur, data_cost_selected, data_cost, 
               h, w, nr_plane, h2, w2, nr_plane2, stream);
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}}}

///////////////////////////////////////////////////////////////
////////////////////  calc all iterations /////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{
    template <typename T>
    __device__ void message_per_pixel(const T* data, T* msg_dst, const T* msg1, const T* msg2, const T* msg3, 
                                      const T* dst_disp, const T* src_disp, int nr_plane, T* temp)
    {
        T minimum = TypeLimits<T>::max();

        for(int d = 0; d < nr_plane; d++)
        {
            int idx = d * cdisp_step1;
            T val  = data[idx] + msg1[idx] + msg2[idx] + msg3[idx];

            if(val < minimum) 
                minimum = val;

            msg_dst[idx] = val;
        }

        float sum = 0;
        for(int d = 0; d < nr_plane; d++)
        {
            float cost_min = minimum + cmax_disc_term;
            T src_disp_reg = src_disp[d * cdisp_step1];

            for(int d2 = 0; d2 < nr_plane; d2++)
                cost_min = fmin(cost_min, msg_dst[d2 * cdisp_step1] + cdisc_single_jump * abs(dst_disp[d2 * cdisp_step1] - src_disp_reg));
            
            temp[d * cdisp_step1] = saturate_cast<T>(cost_min);
            sum += cost_min;
        }
        sum /= nr_plane;

        for(int d = 0; d < nr_plane; d++)
            msg_dst[d * cdisp_step1] = saturate_cast<T>(temp[d * cdisp_step1] - sum);
    }

    template <typename T>
    __global__ void compute_message(T* u_, T* d_, T* l_, T* r_, const T* data_cost_selected, const T* selected_disp_pyr_cur, 
                                    int h, int w, int nr_plane, int i)
    {
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + i) & 1);

        if (y > 0 && y < h - 1 && x > 0 && x < w - 1)
        {
            const T* data = data_cost_selected + y * cmsg_step1 + x;

            T* u = u_ + y * cmsg_step1 + x;
            T* d = d_ + y * cmsg_step1 + x;
            T* l = l_ + y * cmsg_step1 + x;
            T* r = r_ + y * cmsg_step1 + x;
            
            const T* disp = selected_disp_pyr_cur + y * cmsg_step1 + x;
            
            T* temp = (T*)ctemp1 + y * cmsg_step1 + x;

            message_per_pixel(data, u, r - 1, u + cmsg_step1, l + 1, disp, disp - cmsg_step1, nr_plane, temp);
            message_per_pixel(data, d, d - cmsg_step1, r - 1, l + 1, disp, disp + cmsg_step1, nr_plane, temp);
            message_per_pixel(data, l, u + cmsg_step1, d - cmsg_step1, l + 1, disp, disp - 1, nr_plane, temp);
            message_per_pixel(data, r, u + cmsg_step1, d - cmsg_step1, r - 1, disp, disp + 1, nr_plane, temp);
        }
    }
}

namespace cv { namespace gpu { namespace csbp 
{
    template <typename T>
    void compute_message_caller(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                                const DevMem2D& selected_disp_pyr_cur, int h, int w, int nr_plane, int t, const hipStream_t& stream)
    {            
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(w, threads.x << 1);
        grid.y = divUp(h, threads.y);

        csbp_kernels::compute_message<T><<<grid, threads, 0, stream>>>((T*)u.ptr, (T*)d.ptr, (T*)l.ptr, (T*)r.ptr, 
                                                            (const T*)data_cost_selected.ptr, (const T*)selected_disp_pyr_cur.ptr, 
                                                            h, w, nr_plane, t & 1);
    }

    typedef void (*ComputeMessageCaller)(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                                         const DevMem2D& selected_disp_pyr_cur, int h, int w, int nr_plane, int t, const hipStream_t& stream);

    void calc_all_iterations(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                             const DevMem2D& selected_disp_pyr_cur, size_t msg_step, int msg_type, int h, int w, int nr_plane, int iters, const hipStream_t& stream)
    {
        static const ComputeMessageCaller callers[8] = 
            {
                0, 0, 0,
                compute_message_caller<short>,
                0,
                compute_message_caller<float>,
                0, 0
            };
        
        size_t disp_step = msg_step * h;
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step1), &disp_step, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step1),  &msg_step,  sizeof(size_t)) );

        ComputeMessageCaller caller = callers[msg_type];
        if (!caller)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);

        for(int t = 0; t < iters; ++t)
        {
            caller(u, d, l, r, data_cost_selected, selected_disp_pyr_cur, h, w, nr_plane, t, stream);
            
            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}

///////////////////////////////////////////////////////////////
/////////////////////////// output ////////////////////////////
///////////////////////////////////////////////////////////////

namespace csbp_kernels
{
    template <typename T>
    __global__ void compute_disp(const T* u_, const T* d_, const T* l_, const T* r_, 
                                 const T* data_cost_selected, const T* disp_selected_pyr, 
                                 short* disp, size_t res_step, int cols, int rows, int nr_plane) 
    {   
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y > 0 && y < rows - 1 && x > 0 && x < cols - 1)
        {
            const T* data = data_cost_selected + y * cmsg_step1 + x;
            const T* disp_selected = disp_selected_pyr + y * cmsg_step1 + x;

            const T* u = u_ + (y+1) * cmsg_step1 + (x+0);
            const T* d = d_ + (y-1) * cmsg_step1 + (x+0);
            const T* l = l_ + (y+0) * cmsg_step1 + (x+1);
            const T* r = r_ + (y+0) * cmsg_step1 + (x-1);
            
            int best = 0;
            T best_val = TypeLimits<T>::max();
            for (int i = 0; i < nr_plane; ++i) 
            {
                int idx = i * cdisp_step1;
                T val = data[idx]+ u[idx] + d[idx] + l[idx] + r[idx];

                if (val < best_val) 
                {
                    best_val = val;
                    best = saturate_cast<short>(disp_selected[idx]);
                }
            }

            disp[res_step * y + x] = best;
        }
    }
}

namespace cv { namespace gpu { namespace csbp 
{
    template <typename T>
    void compute_disp_caller(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                      const DevMem2D& disp_selected, const DevMem2D& disp, int nr_plane, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);
        
        csbp_kernels::compute_disp<T><<<grid, threads, 0, stream>>>((const T*)u.ptr, (const T*)d.ptr, (const T*)l.ptr, (const T*)r.ptr, 
                                                         (const T*)data_cost_selected.ptr, (const T*)disp_selected.ptr, 
                                                         (short*)disp.ptr, disp.step / sizeof(short), disp.cols, disp.rows, nr_plane);
    }

    typedef void (*ComputeDispCaller)(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                      const DevMem2D& disp_selected, const DevMem2D& disp, int nr_plane, const hipStream_t& stream);

    void compute_disp(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data_cost_selected, 
                      const DevMem2D& disp_selected, size_t msg_step, int msg_type, const DevMem2D& disp, int nr_plane, const hipStream_t& stream)
    {
        static const ComputeDispCaller callers[8] = 
            {
                0, 0, 0,
                compute_disp_caller<short>,
                0,
                compute_disp_caller<float>,
                0, 0
            };
        
        size_t disp_step = disp.rows * msg_step;
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cdisp_step1), &disp_step, sizeof(size_t)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(csbp_kernels::cmsg_step1),  &msg_step,  sizeof(size_t)) );

        ComputeDispCaller caller = callers[msg_type];
        if (!caller)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);

        caller(u, d, l, r, data_cost_selected, disp_selected, disp, nr_plane, stream);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}}}