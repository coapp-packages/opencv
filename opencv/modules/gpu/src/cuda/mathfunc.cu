#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"
#include "transform.hpp"

using namespace cv::gpu;

#ifndef CV_PI
#define CV_PI   3.1415926535897932384626433832795f
#endif

//////////////////////////////////////////////////////////////////////////////////////
// Cart <-> Polar

namespace cv { namespace gpu { namespace mathfunc
{
    struct Nothing
    {
        static __device__ void calc(int, int, float, float, float*, size_t, float)
        {
        }
    };
    struct Magnitude
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = sqrtf(x_data * x_data + y_data * y_data);
        }
    };
    struct MagnitudeSqr
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
        }
    };
    struct Atan2
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
        {
            dst[y * dst_step + x] = scale * atan2f(y_data, x_data);
        }
    };
    template <typename Mag, typename Angle>
    __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step, 
                                float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float x_data = xptr[y * x_step + x];
            float y_data = yptr[y * y_step + x];

            Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
            Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
        }
    }

    struct NonEmptyMag
    {
        static __device__ float get(const float* mag, size_t mag_step, int x, int y)
        {
            return mag[y * mag_step + x];
        }
    };
    struct EmptyMag
    {
        static __device__ float get(const float*, size_t, int, int)
        {
            return 1.0f;
        }
    };
    template <typename Mag>
    __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
        float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float mag_data = Mag::get(mag, mag_step, x, y);
            float angle_data = angle[y * angle_step + x];
            float sin_a, cos_a;

            sincosf(scale * angle_data, &sin_a, &cos_a);

            xptr[y * x_step + x] = mag_data * cos_a;
            yptr[y * y_step + x] = mag_data * sin_a;
        }
    }

    template <typename Mag, typename Angle>
    void cartToPolar_caller(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(x.cols, threads.x);
        grid.y = divUp(x.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(180.0f / CV_PI) : 1.f;

        cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
            x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), 
            mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void cartToPolar_gpu(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, bool magSqr, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2][2][2] = 
        {
            {
                {
                    cartToPolar_caller<Magnitude, Atan2>,
                    cartToPolar_caller<Magnitude, Nothing>
                },
                {
                    cartToPolar_caller<MagnitudeSqr, Atan2>,
                    cartToPolar_caller<MagnitudeSqr, Nothing>,
                }
            },
            {
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>
                },
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>,
                }
            }
        };

        callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
    }

    template <typename Mag>
    void polarToCart_caller(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(mag.cols, threads.x);
        grid.y = divUp(mag.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(CV_PI / 180.0f) : 1.0f;

        polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(), 
            angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void polarToCart_gpu(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2] = 
        {
            polarToCart_caller<NonEmptyMag>,
            polarToCart_caller<EmptyMag>
        };

        callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
    }

//////////////////////////////////////////////////////////////////////////////////////
// Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ uchar operator()(const T1& src1, const T2& src2)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, 0);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<uint, uint>(src1, src2, dst);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<float, float>(src1, src2, dst);
    }


//////////////////////////////////////////////////////////////////////////////
// Per-element bit-wise logical matrix operations

    struct Mask8U
    {
        explicit Mask8U(PtrStep mask): mask(mask) {}
        __device__ bool operator()(int y, int x) { return mask.ptr(y)[x]; }
        PtrStep mask;
    };
    struct MaskTrue { __device__ bool operator()(int y, int x) { return true; } };

    // Unary operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct UnOp<T, UN_OP_NOT>{ __device__ T operator()(T x) { return ~x; } };

    template <typename T, int cn, typename UnOp, typename Mask>
    __global__ void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, UnOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* srcy = (const T*)src.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(srcy[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_un_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_un_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_un_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_un_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_un_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_un_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_un_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_un_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_un_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_un_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, Mask8U(mask), stream);
    }

    // Binary operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct BinOp<T, BIN_OP_OR>{ __device__ T operator()(T lhs, T rhs) { return lhs | rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_AND>{ __device__ T operator()(T lhs, T rhs) { return lhs & rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>{ __device__ T operator()(T lhs, T rhs) { return lhs ^ rhs; } };

    template <typename T, int cn, typename BinOp, typename Mask>
    __global__ void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, BinOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* src1y = (const T*)src1.ptr(y);
            const T* src2y = (const T*)src2.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(src1y[cn * x + i], src2y[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_bin_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_bin_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_bin_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_bin_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_bin_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_bin_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_bin_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_bin_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_bin_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_bin_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }  



//////////////////////////////////////////////////////////////////////////////
// Min max

    enum { MIN, MAX };  

    template <typename T> struct MinMaxTypeTraits {};
    template <> struct MinMaxTypeTraits<unsigned char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<signed char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<unsigned short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<signed short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<int> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<float> { typedef float best_type; };
    template <> struct MinMaxTypeTraits<double> { typedef double best_type; };

    template <typename T, int op> struct Cmp {};
    
    template <typename T>
    struct Cmp<T, MIN> 
    {
        static __device__ void call(unsigned int tid, unsigned int offset, volatile T* optval)
        {
            optval[tid] = min(optval[tid], optval[tid + offset]); 
        }
    };

    template <typename T>
    struct Cmp<T, MAX> 
    {
        static __device__ void call(unsigned int tid, unsigned int offset, volatile T* optval)
        {
            optval[tid] = max(optval[tid], optval[tid + offset]);
        }
    };


    template <int nthreads, int op, typename T>
    __global__ void opt_kernel(int cols, int rows, const PtrStep src, PtrStep optval)
    {
        typedef typename MinMaxTypeTraits<T>::best_type best_type;
        __shared__ best_type soptval[nthreads];

        unsigned int x0 = blockIdx.x * blockDim.x;
        unsigned int y0 = blockIdx.y * blockDim.y;
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        if (x0 + threadIdx.x < cols && y0 + threadIdx.y < rows)
            soptval[tid] = ((const T*)src.ptr(y0 + threadIdx.y))[x0 + threadIdx.x];
        else
            soptval[tid] = ((const T*)src.ptr(y0))[x0];

        __syncthreads();

        if (nthreads >= 512) if (tid < 256) { Cmp<best_type, op>::call(tid, 256, soptval); __syncthreads(); }
        if (nthreads >= 256) if (tid < 128) { Cmp<best_type, op>::call(tid, 128, soptval); __syncthreads(); }
        if (nthreads >= 128) if (tid < 64) { Cmp<best_type, op>::call(tid, 64, soptval); __syncthreads(); }

        if (tid < 32)
        {
            if (nthreads >= 64) Cmp<best_type, op>::call(tid, 32, soptval);
            if (nthreads >= 32) Cmp<best_type, op>::call(tid, 16, soptval);
            if (nthreads >= 16) Cmp<best_type, op>::call(tid, 8, soptval);
            if (nthreads >= 8) Cmp<best_type, op>::call(tid, 4, soptval);
            if (nthreads >= 4) Cmp<best_type, op>::call(tid, 2, soptval);
            if (nthreads >= 2) Cmp<best_type, op>::call(tid, 1, soptval);
        }

        if (tid == 0) ((T*)optval.ptr(blockIdx.y))[blockIdx.x] = (T)soptval[0];
    }

   
    template <typename T>
    void min_max_caller(const DevMem2D src, double* minval, double* maxval)
    {
        dim3 threads(32, 8);

        // Allocate memory for aux. buffers
        DevMem2D minval_buf[2]; DevMem2D maxval_buf[2];
        minval_buf[0].cols = divUp(src.cols, threads.x); 
        minval_buf[0].rows = divUp(src.rows, threads.y);
        minval_buf[1].cols = divUp(minval_buf[0].cols, threads.x); 
        minval_buf[1].rows = divUp(minval_buf[0].rows, threads.y);
        maxval_buf[0].cols = divUp(src.cols, threads.x); 
        maxval_buf[0].rows = divUp(src.rows, threads.y);
        maxval_buf[1].cols = divUp(maxval_buf[0].cols, threads.x); 
        maxval_buf[1].rows = divUp(maxval_buf[0].rows, threads.y);
        cudaSafeCall(hipMallocPitch(&minval_buf[0].data, &minval_buf[0].step, minval_buf[0].cols * sizeof(T), minval_buf[0].rows));
        cudaSafeCall(hipMallocPitch(&minval_buf[1].data, &minval_buf[1].step, minval_buf[1].cols * sizeof(T), minval_buf[1].rows));
        cudaSafeCall(hipMallocPitch(&maxval_buf[0].data, &maxval_buf[0].step, maxval_buf[0].cols * sizeof(T), maxval_buf[0].rows));
        cudaSafeCall(hipMallocPitch(&maxval_buf[1].data, &maxval_buf[1].step, maxval_buf[1].cols * sizeof(T), maxval_buf[1].rows));

        int curbuf = 0;
        dim3 cursize(src.cols, src.rows);
        dim3 grid(divUp(cursize.x, threads.x), divUp(cursize.y, threads.y));

        opt_kernel<256, MIN, T><<<grid, threads>>>(cursize.x, cursize.y, src, minval_buf[curbuf]);
        opt_kernel<256, MAX, T><<<grid, threads>>>(cursize.x, cursize.y, src, maxval_buf[curbuf]);
        cursize = grid;

        while (cursize.x > 1 || cursize.y > 1)
        {
            grid.x = divUp(cursize.x, threads.x); 
            grid.y = divUp(cursize.y, threads.y);  
            opt_kernel<256, MIN, T><<<grid, threads>>>(cursize.x, cursize.y, minval_buf[curbuf], minval_buf[1 - curbuf]);
            opt_kernel<256, MAX, T><<<grid, threads>>>(cursize.x, cursize.y, maxval_buf[curbuf], maxval_buf[1 - curbuf]);
            curbuf = 1 - curbuf;
            cursize = grid;
        }

        cudaSafeCall(hipDeviceSynchronize());

        // Copy results from device to host
        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf[curbuf].ptr(0), sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf[curbuf].ptr(0), sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;

        // Release aux. buffers
        cudaSafeCall(hipFree(minval_buf[0].data));
        cudaSafeCall(hipFree(minval_buf[1].data));
        cudaSafeCall(hipFree(maxval_buf[0].data));
        cudaSafeCall(hipFree(maxval_buf[1].data));
    }

    template void min_max_caller<unsigned char>(const DevMem2D, double*, double*);
    template void min_max_caller<signed char>(const DevMem2D, double*, double*);
    template void min_max_caller<unsigned short>(const DevMem2D, double*, double*);
    template void min_max_caller<signed short>(const DevMem2D, double*, double*);
    template void min_max_caller<int>(const DevMem2D, double*, double*);
    template void min_max_caller<float>(const DevMem2D, double*, double*);
    template void min_max_caller<double>(const DevMem2D, double*, double*);

}}}
