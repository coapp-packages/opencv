#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"
#include "transform.hpp"

using namespace cv::gpu;

#ifndef CV_PI
#define CV_PI   3.1415926535897932384626433832795f
#endif

//////////////////////////////////////////////////////////////////////////////////////
// Cart <-> Polar

namespace cv { namespace gpu { namespace mathfunc
{
    struct Nothing
    {
        static __device__ void calc(int, int, float, float, float*, size_t, float)
        {
        }
    };
    struct Magnitude
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = sqrtf(x_data * x_data + y_data * y_data);
        }
    };
    struct MagnitudeSqr
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
        }
    };
    struct Atan2
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
        {
            dst[y * dst_step + x] = scale * atan2f(y_data, x_data);
        }
    };
    template <typename Mag, typename Angle>
    __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step, 
                                float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float x_data = xptr[y * x_step + x];
            float y_data = yptr[y * y_step + x];

            Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
            Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
        }
    }

    struct NonEmptyMag
    {
        static __device__ float get(const float* mag, size_t mag_step, int x, int y)
        {
            return mag[y * mag_step + x];
        }
    };
    struct EmptyMag
    {
        static __device__ float get(const float*, size_t, int, int)
        {
            return 1.0f;
        }
    };
    template <typename Mag>
    __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
        float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float mag_data = Mag::get(mag, mag_step, x, y);
            float angle_data = angle[y * angle_step + x];
            float sin_a, cos_a;

            sincosf(scale * angle_data, &sin_a, &cos_a);

            xptr[y * x_step + x] = mag_data * cos_a;
            yptr[y * y_step + x] = mag_data * sin_a;
        }
    }

    template <typename Mag, typename Angle>
    void cartToPolar_caller(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(x.cols, threads.x);
        grid.y = divUp(x.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(180.0f / CV_PI) : 1.f;

        cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
            x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), 
            mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void cartToPolar_gpu(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, bool magSqr, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2][2][2] = 
        {
            {
                {
                    cartToPolar_caller<Magnitude, Atan2>,
                    cartToPolar_caller<Magnitude, Nothing>
                },
                {
                    cartToPolar_caller<MagnitudeSqr, Atan2>,
                    cartToPolar_caller<MagnitudeSqr, Nothing>,
                }
            },
            {
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>
                },
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>,
                }
            }
        };

        callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
    }

    template <typename Mag>
    void polarToCart_caller(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(mag.cols, threads.x);
        grid.y = divUp(mag.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(CV_PI / 180.0f) : 1.0f;

        polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(), 
            angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void polarToCart_gpu(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2] = 
        {
            polarToCart_caller<NonEmptyMag>,
            polarToCart_caller<EmptyMag>
        };

        callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
    }

//////////////////////////////////////////////////////////////////////////////////////
// Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ uchar operator()(const T1& src1, const T2& src2, int, int)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, 0);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<uint, uint>(src1, src2, dst);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<float, float>(src1, src2, dst);
    }


//////////////////////////////////////////////////////////////////////////////
// Per-element bit-wise logical matrix operations


    __global__ void bitwise_not_kernel(int cols, int rows, const PtrStep src, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            dst.ptr(y)[x] = ~src.ptr(y)[x];
        }
    }


    void bitwise_not_caller(const DevMem2D src, int elemSize, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(divUp(src.cols * elemSize, threads.x), divUp(src.rows, threads.y), 1);

        bitwise_not_kernel<<<grid, threads, 0, stream>>>(src.cols * elemSize, src.rows, src, dst);

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    __global__ void bitwise_or_kernel(int cols, int rows, const PtrStep src1, const PtrStep src2, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            dst.ptr(y)[x] = src1.ptr(y)[x] | src2.ptr(y)[x];
        }
    }


    void bitwise_or_caller(int cols, int rows, const PtrStep src1, const PtrStep src2, int elemSize, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(divUp(cols * elemSize, threads.x), divUp(rows, threads.y), 1);

        bitwise_or_kernel<<<grid, threads, 0, stream>>>(cols * elemSize, rows, src1, src2, dst);

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }


    __global__ void bitwise_and_kernel(int cols, int rows, const PtrStep src1, const PtrStep src2, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            dst.ptr(y)[x] = src1.ptr(y)[x] & src2.ptr(y)[x];
        }
    }


    void bitwise_and_caller(int cols, int rows, const PtrStep src1, const PtrStep src2, int elemSize, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(divUp(cols * elemSize, threads.x), divUp(rows, threads.y), 1);

        bitwise_and_kernel<<<grid, threads, 0, stream>>>(cols * elemSize, rows, src1, src2, dst);

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }



    __global__ void bitwise_xor_kernel(int cols, int rows, const PtrStep src1, const PtrStep src2, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            dst.ptr(y)[x] = src1.ptr(y)[x] ^ src2.ptr(y)[x];
        }
    }


    void bitwise_xor_caller(int cols, int rows, const PtrStep src1, const PtrStep src2, int elemSize, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(divUp(cols * elemSize, threads.x), divUp(rows, threads.y), 1);

        bitwise_xor_kernel<<<grid, threads, 0, stream>>>(cols * elemSize, rows, src1, src2, dst);

        if (stream == 0)
            cudaSafeCall(hipDeviceSynchronize());
    }
}}}
