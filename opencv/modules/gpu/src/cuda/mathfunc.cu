#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"
#include "transform.hpp"

using namespace cv::gpu;

#ifndef CV_PI
#define CV_PI   3.1415926535897932384626433832795f
#endif

//////////////////////////////////////////////////////////////////////////////////////
// Cart <-> Polar

namespace cv { namespace gpu { namespace mathfunc
{
    struct Nothing
    {
        static __device__ void calc(int, int, float, float, float*, size_t, float)
        {
        }
    };
    struct Magnitude
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = sqrtf(x_data * x_data + y_data * y_data);
        }
    };
    struct MagnitudeSqr
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
        }
    };
    struct Atan2
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
        {
            dst[y * dst_step + x] = scale * atan2f(y_data, x_data);
        }
    };
    template <typename Mag, typename Angle>
    __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step, 
                                float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float x_data = xptr[y * x_step + x];
            float y_data = yptr[y * y_step + x];

            Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
            Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
        }
    }

    struct NonEmptyMag
    {
        static __device__ float get(const float* mag, size_t mag_step, int x, int y)
        {
            return mag[y * mag_step + x];
        }
    };
    struct EmptyMag
    {
        static __device__ float get(const float*, size_t, int, int)
        {
            return 1.0f;
        }
    };
    template <typename Mag>
    __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
        float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float mag_data = Mag::get(mag, mag_step, x, y);
            float angle_data = angle[y * angle_step + x];
            float sin_a, cos_a;

            sincosf(scale * angle_data, &sin_a, &cos_a);

            xptr[y * x_step + x] = mag_data * cos_a;
            yptr[y * y_step + x] = mag_data * sin_a;
        }
    }

    template <typename Mag, typename Angle>
    void cartToPolar_caller(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(x.cols, threads.x);
        grid.y = divUp(x.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(180.0f / CV_PI) : 1.f;

        cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
            x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), 
            mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void cartToPolar_gpu(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, bool magSqr, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2][2][2] = 
        {
            {
                {
                    cartToPolar_caller<Magnitude, Atan2>,
                    cartToPolar_caller<Magnitude, Nothing>
                },
                {
                    cartToPolar_caller<MagnitudeSqr, Atan2>,
                    cartToPolar_caller<MagnitudeSqr, Nothing>,
                }
            },
            {
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>
                },
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>,
                }
            }
        };

        callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
    }

    template <typename Mag>
    void polarToCart_caller(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(mag.cols, threads.x);
        grid.y = divUp(mag.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(CV_PI / 180.0f) : 1.0f;

        polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(), 
            angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void polarToCart_gpu(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2] = 
        {
            polarToCart_caller<NonEmptyMag>,
            polarToCart_caller<EmptyMag>
        };

        callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
    }

//////////////////////////////////////////////////////////////////////////////////////
// Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ uchar operator()(const T1& src1, const T2& src2, int, int)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, 0);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<uint, uint>(src1, src2, dst);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<float, float>(src1, src2, dst);
    }


//////////////////////////////////////////////////////////////////////////////
// Per-element bit-wise logical matrix operations

    struct Mask8U
    {
        explicit Mask8U(PtrStep mask): mask(mask) {}
        __device__ bool operator()(int y, int x) { return mask.ptr(y)[x]; }
        PtrStep mask;
    };
    struct MaskTrue { __device__ bool operator()(int y, int x) { return true; } };

    // Unary operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct UnOp<T, UN_OP_NOT>{ __device__ T operator()(T x) { return ~x; } };

    template <typename T, int cn, typename UnOp, typename Mask>
    __global__ void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, UnOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* srcy = (const T*)src.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(srcy[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_un_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_un_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_un_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_un_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_un_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_un_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_un_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_un_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_un_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_un_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, Mask8U(mask), stream);
    }

    // Binary operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct BinOp<T, BIN_OP_OR>{ __device__ T operator()(T lhs, T rhs) { return lhs | rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_AND>{ __device__ T operator()(T lhs, T rhs) { return lhs & rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>{ __device__ T operator()(T lhs, T rhs) { return lhs ^ rhs; } };

    template <typename T, int cn, typename BinOp, typename Mask>
    __global__ void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, BinOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* src1y = (const T*)src1.ptr(y);
            const T* src2y = (const T*)src2.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(src1y[cn * x + i], src2y[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_bin_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_bin_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_bin_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_bin_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_bin_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_bin_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_bin_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_bin_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_bin_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_bin_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }  
}}}
