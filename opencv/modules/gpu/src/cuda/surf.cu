#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
// Copyright (c) 2010, Paul Furgale, Chi Hay Tong
//
// The original code was written by Paul Furgale and Chi Hay Tong 
// and later optimized and prepared for integration into OpenCV by Itseez.
//
//M*/

#include "internal_shared.hpp"
#include "surf_key_point.h"
#include "opencv2/gpu/device/limits_gpu.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

#define CV_PI 3.1415926535897932384626433832795f

namespace cv { namespace gpu { namespace surf
{
    ////////////////////////////////////////////////////////////////////////
    // Help funcs

    // Wrapper for host reference to pass into kernel
    template <typename T> 
    class DeviceReference
    {
    public:
        explicit DeviceReference(T& host_val) : d_ptr(0), h_ptr(&host_val)
        {
            cudaSafeCall( hipMalloc((void**)&d_ptr, sizeof(T)) );
            cudaSafeCall( hipMemcpy(d_ptr, h_ptr, sizeof(T), hipMemcpyHostToDevice) );
        }

        ~DeviceReference()
        {            
            cudaSafeCall( hipMemcpy(h_ptr, d_ptr, sizeof(T), hipMemcpyDeviceToHost) );
            cudaSafeCall( hipFree(d_ptr) );
        }

        // Casting to device pointer
        operator T*() {return d_ptr;}
        operator const T*() const {return d_ptr;}
    private:
        T* d_ptr;
        T* h_ptr;
    };

    __device__ void clearLastBit(int* f)
    {
        *f &= ~0x1;
    }
    __device__ void clearLastBit(float& f)
    {
        clearLastBit((int*)&f);
    }

    __device__ void setLastBit(int* f)
    {
        *f |= 0x1;
    }
    __device__ void setLastBit(float& f)
    {
        setLastBit((int*)&f);
    }

    ////////////////////////////////////////////////////////////////////////
    // Global parameters

    // The maximum number of features (before subpixel interpolation) that memory is reserved for.
    __constant__ int c_max_candidates;
    // The maximum number of features that memory is reserved for.
    __constant__ int c_max_features;
    // The number of intervals in the octave.
    __constant__ int c_nIntervals;
    // Mask sizes derived from the mask parameters
    __constant__ float c_mask_width;
    // Mask sizes derived from the mask parameters
    __constant__ float c_mask_height;
    // Mask sizes derived from the mask parameters
    __constant__ float c_dxy_center_offset;
    // Mask sizes derived from the mask parameters
    __constant__ float c_dxy_half_width;
    // Mask sizes derived from the mask parameters
    __constant__ float c_dxy_scale;
    // The scale associated with the first interval of the first octave
    __constant__ float c_initialScale;
    //! The interest operator threshold
    __constant__ float c_threshold;

    // Ther octave
    __constant__ int c_octave;
    // The width of the octave buffer.
    __constant__ int c_x_size;
    // The height of the octave buffer.
    __constant__ int c_y_size;
    // The size of the octave border in pixels.
    __constant__ int c_border;
    // The step size used in this octave in pixels.
    __constant__ int c_step;

    ////////////////////////////////////////////////////////////////////////
    // Integral image texture

    texture<float, 2, hipReadModeElementType> sumTex(0, hipFilterModeLinear, hipAddressModeClamp);

    __device__ float iiAreaLookupCDHalfWH(float cx, float cy, float halfWidth, float halfHeight)
    {
        float result = 0.f;

        result += tex2D(sumTex, cx - halfWidth, cy - halfHeight);
        result -= tex2D(sumTex, cx + halfWidth, cy - halfHeight);
        result -= tex2D(sumTex, cx - halfWidth, cy + halfHeight);
        result += tex2D(sumTex, cx + halfWidth, cy + halfHeight);

        return result;
    }

    ////////////////////////////////////////////////////////////////////////
    // Hessian

    __device__ float evalDyy(float x, float y, float t, float mask_width, float mask_height, float fscale)
    {
        float Dyy = 0.f;

        Dyy +=     iiAreaLookupCDHalfWH(x, y, mask_width, mask_height);
        Dyy -= t * iiAreaLookupCDHalfWH(x, y, mask_width, fscale);

        Dyy *=  1.0f / (fscale * fscale);

        return Dyy;
    }

    __device__ float evalDxx(float x, float y, float t, float mask_width, float mask_height, float fscale)
    {
    	float Dxx = 0.f;
	
	    Dxx +=     iiAreaLookupCDHalfWH(x, y, mask_height, mask_width);
	    Dxx -= t * iiAreaLookupCDHalfWH(x, y, fscale     , mask_width);

	    Dxx *=  1.0f / (fscale * fscale);

	    return Dxx;
    }
    
    __device__ float evalDxy(float x, float y, float fscale)
    {
    	float center_offset =  c_dxy_center_offset  * fscale;
	    float half_width    =  c_dxy_half_width     * fscale;

	    float Dxy = 0.f;

	    Dxy += iiAreaLookupCDHalfWH(x - center_offset, y - center_offset, half_width, half_width);
	    Dxy -= iiAreaLookupCDHalfWH(x - center_offset, y + center_offset, half_width, half_width);
	    Dxy += iiAreaLookupCDHalfWH(x + center_offset, y + center_offset, half_width, half_width);
	    Dxy -= iiAreaLookupCDHalfWH(x + center_offset, y - center_offset, half_width, half_width);
	
	    Dxy *= 1.0f / (fscale * fscale);

	    return Dxy;
    }

    __device__ float calcScale(int hidx_z)
    {
        float d = (c_initialScale * (1 << c_octave)) / (c_nIntervals - 2);
        return c_initialScale * (1 << c_octave) + d * (hidx_z - 1.0f) + 0.5f;
    }
    
    __global__ void fasthessian(PtrStepf hessianBuffer)
    {
  	    // Determine the indices in the Hessian buffer
        int hidx_x = threadIdx.x + blockIdx.x * blockDim.x;
        int hidx_y = threadIdx.y + blockIdx.y * blockDim.y;
        int hidx_z = threadIdx.z;

        float fscale = calcScale(hidx_z);

	    // Compute the lookup location of the mask center
        float x = hidx_x * c_step + c_border;
        float y = hidx_y * c_step + c_border;

	    // Scale the mask dimensions according to the scale
        if (hidx_x < c_x_size && hidx_y < c_y_size && hidx_z < c_nIntervals)
        {
	        float mask_width =  c_mask_width  * fscale;
	        float mask_height = c_mask_height * fscale;

	        // Compute the filter responses
	        float Dyy = evalDyy(x, y, c_mask_height, mask_width, mask_height, fscale);
	        float Dxx = evalDxx(x, y, c_mask_height, mask_width, mask_height, fscale);
	        float Dxy = evalDxy(x, y, fscale);
	
	        // Combine the responses and store the Laplacian sign
	        float result = (Dxx * Dyy) - c_dxy_scale * (Dxy * Dxy);

	        if (Dxx + Dyy > 0.f)
	            setLastBit(result);
	        else
	            clearLastBit(result);

	        hessianBuffer.ptr(c_y_size * hidx_z + hidx_y)[hidx_x] = result;
        }
    }   

    void fasthessian_gpu(PtrStepf hessianBuffer, int nIntervals, int x_size, int y_size)
    {
        dim3 threads;
        threads.x = 16;
        threads.y = 8;
        threads.z = nIntervals;

        dim3 grid;
        grid.x = divUp(x_size, threads.x);
        grid.y = divUp(y_size, threads.y);
        grid.z = 1;

  	    fasthessian<<<grid, threads>>>(hessianBuffer);

        cudaSafeCall( hipDeviceSynchronize() );
	}

    ////////////////////////////////////////////////////////////////////////
    // NONMAX

    __global__ void nonmaxonly(PtrStepf hessianBuffer, int4* maxPosBuffer, unsigned int* maxCounter)
    {        
        #if defined (__CUDA_ARCH__) && __CUDA_ARCH__ >= 110

        extern __shared__ float fh_vals[];

        // The hidx variables are the indices to the hessian buffer.
        int hidx_x = threadIdx.x + blockIdx.x * (blockDim.x - 2);
        int hidx_y = threadIdx.y + blockIdx.y * (blockDim.y - 2);
        int hidx_z = threadIdx.z;
        int localLin = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

        // Is this thread within the hessian buffer?
        if (hidx_x < c_x_size && hidx_y < c_y_size && hidx_z < c_nIntervals)
        {
            fh_vals[localLin] = hessianBuffer.ptr(c_y_size * hidx_z + hidx_y)[hidx_x];
        }
        __syncthreads();
    
        // Is this location one of the ones being processed for nonmax suppression.
        // Blocks overlap by one so we don't process the border threads.
        bool inBounds2 = threadIdx.x > 0 && threadIdx.x < blockDim.x-1 && hidx_x < c_x_size - 1 
            &&           threadIdx.y > 0 && threadIdx.y < blockDim.y-1 && hidx_y < c_y_size - 1
            &&           threadIdx.z > 0 && threadIdx.z < blockDim.z-1;

        float val = fh_vals[localLin];

        if (inBounds2 && val >= c_threshold)
        {
            // Check to see if we have a max (in its 26 neighbours)
            int zoff = blockDim.x * blockDim.y;
            bool condmax  =  val > fh_vals[localLin                     + 1]
            &&               val > fh_vals[localLin                     - 1]
            &&               val > fh_vals[localLin        - blockDim.x + 1]
            &&               val > fh_vals[localLin        - blockDim.x    ]
            &&               val > fh_vals[localLin        - blockDim.x - 1]
            &&               val > fh_vals[localLin        + blockDim.x + 1]
            &&               val > fh_vals[localLin        + blockDim.x    ]
            &&               val > fh_vals[localLin        + blockDim.x - 1]
      
            &&               val > fh_vals[localLin - zoff              + 1]
            &&               val > fh_vals[localLin - zoff                 ]
            &&               val > fh_vals[localLin - zoff              - 1]
            &&               val > fh_vals[localLin - zoff - blockDim.x + 1]
            &&               val > fh_vals[localLin - zoff - blockDim.x    ]
            &&               val > fh_vals[localLin - zoff - blockDim.x - 1]
            &&               val > fh_vals[localLin - zoff + blockDim.x + 1]
            &&               val > fh_vals[localLin - zoff + blockDim.x    ]
            &&               val > fh_vals[localLin - zoff + blockDim.x - 1]
      
            &&               val > fh_vals[localLin + zoff              + 1]
            &&               val > fh_vals[localLin + zoff                 ]
            &&               val > fh_vals[localLin + zoff              - 1]
            &&               val > fh_vals[localLin + zoff - blockDim.x + 1]
            &&               val > fh_vals[localLin + zoff - blockDim.x    ]
            &&               val > fh_vals[localLin + zoff - blockDim.x - 1]
            &&               val > fh_vals[localLin + zoff + blockDim.x + 1]
            &&               val > fh_vals[localLin + zoff + blockDim.x    ]
            &&               val > fh_vals[localLin + zoff + blockDim.x - 1]
            ;

            if(condmax) 
            {
                unsigned int i = atomicInc(maxCounter,(unsigned int) -1);
      
                if (i < c_max_candidates) 
                {
	                int4 f = {hidx_x, hidx_y, threadIdx.z, c_octave};
	                maxPosBuffer[i] = f;	
                }
            }
        }  

        #endif
    }

    void nonmaxonly_gpu(PtrStepf hessianBuffer, int4* maxPosBuffer, unsigned int& maxCounter, 
        int nIntervals, int x_size, int y_size)
    {
        dim3 threads;
        threads.x = 16;
        threads.y = 8;
        threads.z = nIntervals;

        dim3 grid;
        grid.x = divUp(x_size, threads.x - 2);
        grid.y = divUp(y_size, threads.y - 2);
        grid.z = 1;

        const size_t smem_size = threads.x * threads.y * threads.z * sizeof(float);

        DeviceReference<unsigned int> maxCounterWrapper(maxCounter);

        nonmaxonly<<<grid, threads, smem_size>>>(hessianBuffer, maxPosBuffer, maxCounterWrapper);

        cudaSafeCall( hipDeviceSynchronize() );
    }

    ////////////////////////////////////////////////////////////////////////
    // INTERPOLATION
    
    #define MID_IDX 1
    __global__ void fh_interp_extremum(PtrStepf hessianBuffer, const int4* maxPosBuffer, 
        KeyPoint_GPU* featuresBuffer, unsigned int* featureCounter)
    {        
        #if defined (__CUDA_ARCH__) && __CUDA_ARCH__ >= 110

        int hidx_x = maxPosBuffer[blockIdx.x].x - 1 + threadIdx.x;
        int hidx_y = maxPosBuffer[blockIdx.x].y - 1 + threadIdx.y;
        int hidx_z = maxPosBuffer[blockIdx.x].z - 1 + threadIdx.z;

        __shared__ float fh_vals[3][3][3];
        __shared__ KeyPoint_GPU p;

        fh_vals[threadIdx.z][threadIdx.y][threadIdx.x] = hessianBuffer.ptr(c_y_size * hidx_z + hidx_y)[hidx_x];
        __syncthreads();

        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
        {
            __shared__ float H[3][3];

            //dxx
            H[0][0] =    fh_vals[MID_IDX    ][MID_IDX + 1][MID_IDX    ] 
	        -       2.0f*fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX    ]
	        +            fh_vals[MID_IDX    ][MID_IDX - 1][MID_IDX    ];

            //dyy
            H[1][1] =    fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX + 1] 
	        -       2.0f*fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX    ]
	        +            fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX - 1];
      
            //dss
            H[2][2] =    fh_vals[MID_IDX + 1][MID_IDX    ][MID_IDX    ] 
	        -       2.0f*fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX    ]
	        +            fh_vals[MID_IDX - 1][MID_IDX    ][MID_IDX    ];

            //dxy
            H[0][1]= 0.25f*
                (fh_vals[MID_IDX    ][MID_IDX + 1][MID_IDX + 1] -
		         fh_vals[MID_IDX    ][MID_IDX - 1][MID_IDX + 1] -
		         fh_vals[MID_IDX    ][MID_IDX + 1][MID_IDX - 1] + 
		         fh_vals[MID_IDX    ][MID_IDX - 1][MID_IDX - 1]);
      
            //dxs
            H[0][2]= 0.25f*
                (fh_vals[MID_IDX + 1][MID_IDX + 1][MID_IDX    ] -
		         fh_vals[MID_IDX + 1][MID_IDX - 1][MID_IDX    ] -
		         fh_vals[MID_IDX - 1][MID_IDX + 1][MID_IDX    ] + 
		         fh_vals[MID_IDX - 1][MID_IDX - 1][MID_IDX    ]);

            //dys
            H[1][2]= 0.25f*
                (fh_vals[MID_IDX + 1][MID_IDX    ][MID_IDX + 1] -
		         fh_vals[MID_IDX + 1][MID_IDX    ][MID_IDX - 1] -
		         fh_vals[MID_IDX - 1][MID_IDX    ][MID_IDX + 1] + 
		         fh_vals[MID_IDX - 1][MID_IDX    ][MID_IDX - 1]);

            //dyx = dxy
            H[1][0] = H[0][1];

            //dsx = dxs
            H[2][0] = H[0][2];

            //dsy = dys
            H[2][1] = H[1][2];

            __shared__ float dD[3];

            //dx
            dD[0] = 0.5f*(fh_vals[MID_IDX    ][MID_IDX + 1][MID_IDX    ] -
	 	        fh_vals[MID_IDX    ][MID_IDX - 1][MID_IDX    ]);
            //dy
            dD[1] = 0.5f*(fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX + 1] -
		        fh_vals[MID_IDX    ][MID_IDX    ][MID_IDX - 1]);
            //ds
            dD[2] = 0.5f*(fh_vals[MID_IDX + 1][MID_IDX    ][MID_IDX    ] -
    		    fh_vals[MID_IDX - 1][MID_IDX    ][MID_IDX    ]);

            __shared__ float invdet;
            invdet = 1.f /
                (
                H[0][0]*H[1][1]*H[2][2] 
                +   H[0][1]*H[1][2]*H[2][0]
                +   H[0][2]*H[1][0]*H[2][1]
                -   H[0][0]*H[1][2]*H[2][1]
                -   H[0][1]*H[1][0]*H[2][2]
                -   H[0][2]*H[1][1]*H[2][0]
                );

            //   // 1-based entries of a 3x3 inverse
            //   /*             [ |a22 a23|   |a12 a13|  |a12 a13|]     */
            //   /*             [ |a32 a33|  -|a32 a33|  |a22 a23|]     */
            //   /*             [                                 ]     */
            //   /*             [ |a21 a23|   |a11 a13|  |a11 a13|]     */
            //   /*    A^(-1) = [-|a31 a33|   |a31 a33| -|a21 a23|] / d */
            //   /*             [                                 ]     */
            //   /*             [ |a21 a22|   |a11 a12|  |a11 a12|]     */
            //   /*             [ |a31 a32|  -|a31 a32|  |a21 a22|]     */

            __shared__ float Hinv[3][3];
            Hinv[0][0] =  invdet*(H[1][1]*H[2][2]-H[1][2]*H[2][1]);
            Hinv[0][1] = -invdet*(H[0][1]*H[2][2]-H[0][2]*H[2][1]);
            Hinv[0][2] =  invdet*(H[0][1]*H[1][2]-H[0][2]*H[1][1]);

            Hinv[1][0] = -invdet*(H[1][0]*H[2][2]-H[1][2]*H[2][0]);
            Hinv[1][1] =  invdet*(H[0][0]*H[2][2]-H[0][2]*H[2][0]);
            Hinv[1][2] = -invdet*(H[0][0]*H[1][2]-H[0][2]*H[1][0]);

            Hinv[2][0] =  invdet*(H[1][0]*H[2][1]-H[1][1]*H[2][0]);
            Hinv[2][1] = -invdet*(H[0][0]*H[2][1]-H[0][1]*H[2][0]);
            Hinv[2][2] =  invdet*(H[0][0]*H[1][1]-H[0][1]*H[1][0]);

            __shared__ float x[3];

            x[0] = -(Hinv[0][0]*(dD[0]) + Hinv[0][1]*(dD[1]) + Hinv[0][2]*(dD[2]));
            x[1] = -(Hinv[1][0]*(dD[0]) + Hinv[1][1]*(dD[1]) + Hinv[1][2]*(dD[2]));
            x[2] = -(Hinv[2][0]*(dD[0]) + Hinv[2][1]*(dD[1]) + Hinv[2][2]*(dD[2]));

            if (fabs(x[0]) < 1.f && fabs(x[1]) < 1.f && fabs(x[2]) < 1.f) 
            { 
                // if the step is within the interpolation region, perform it
	
	            // Get a new feature index.
	            unsigned int i = atomicInc(featureCounter, (unsigned int)-1);

 	            if (i < c_max_features) 
                {	  
	                p.x = ((float)maxPosBuffer[blockIdx.x].x + x[1]) * (float)c_step + c_border;
	                p.y = ((float)maxPosBuffer[blockIdx.x].y + x[0]) * (float)c_step + c_border;

 	                if (x[2] > 0)
 	                {
                        float a = calcScale(maxPosBuffer[blockIdx.x].z);
                        float b = calcScale(maxPosBuffer[blockIdx.x].z + 1);

	                    p.size = (1.f - x[2]) * a + x[2] * b;
 	                } 
 	                else
 	                {
                        float a = calcScale(maxPosBuffer[blockIdx.x].z);
                        float b = calcScale(maxPosBuffer[blockIdx.x].z - 1);

	                    p.size = (1.f + x[2]) * a - x[2] * b;
 	                }

	                p.octave = c_octave;
			
	                p.response = fh_vals[MID_IDX][MID_IDX][MID_IDX];

	                // Should we split up this transfer over many threads?
	                featuresBuffer[i] = p;
	            }
            } // If the subpixel interpolation worked
        } // If this is thread 0.

        #endif
    }
    #undef MID_IDX

    void fh_interp_extremum_gpu(PtrStepf hessianBuffer, const int4* maxPosBuffer, unsigned int maxCounter, 
        KeyPoint_GPU* featuresBuffer, unsigned int& featureCounter)
    {
        dim3 threads;
        threads.x = 3;
        threads.y = 3;
        threads.z = 3;
    
        dim3 grid;
        grid.x = maxCounter;
        grid.y = 1; 
        grid.z = 1;

        DeviceReference<unsigned int> featureCounterWrapper(featureCounter);
    
        fh_interp_extremum<<<grid, threads>>>(hessianBuffer, maxPosBuffer, featuresBuffer, featureCounterWrapper);

        cudaSafeCall( hipDeviceSynchronize() );
    }

    ////////////////////////////////////////////////////////////////////////
    // Orientation

    // precomputed values for a Gaussian with a standard deviation of 2
    __constant__ float c_gauss1D[13] = 
    {
        0.002215924206f, 0.008764150247f, 0.026995483257f, 0.064758797833f, 
        0.120985362260f, 0.176032663382f, 0.199471140201f, 0.176032663382f, 
        0.120985362260f, 0.064758797833f, 0.026995483257f, 0.008764150247f, 
        0.002215924206f
    };

    __global__ void find_orientation(KeyPoint_GPU* features)
    {
        int tid = threadIdx.y * 17 + threadIdx.x;
        int tid2 = numeric_limits_gpu<int>::max();

        if (threadIdx.x < 13 && threadIdx.y < 13) 
        {
            tid2 = threadIdx.y * 13 + threadIdx.x;
        }

        __shared__ float texLookups[17][17];
    
        __shared__ float Edx[13*13];
        __shared__ float Edy[13*13];
        __shared__ float xys[3];

        // Read my x, y, size.
        if (tid < 3)
        {
	        xys[tid] = ((float*)(&features[blockIdx.x]))[tid];
        }
        __syncthreads();

        // Read all texture locations into memory
        // Maybe I should use __mul24 here?
        texLookups[threadIdx.x][threadIdx.y] = tex2D(sumTex, xys[SF_X] + ((int)threadIdx.x - 8) * xys[SF_SIZE], 
                  xys[SF_Y] + ((int)threadIdx.y - 8) * xys[SF_SIZE]);

        __syncthreads();

        float dx = 0.f;
        float dy = 0.f;
	 
	    // Computes lookups for all points in a 13x13 lattice.
	    // - SURF says to only use a circle, but the branching logic would slow it down
	    // - Gaussian weighting should reduce the effects of the outer points anyway
        if (tid2 < 169)
        {
	        dx -=     texLookups[threadIdx.x    ][threadIdx.y    ];
	        dx += 2.f*texLookups[threadIdx.x + 2][threadIdx.y    ];
	        dx -=     texLookups[threadIdx.x + 4][threadIdx.y    ];
	        dx +=     texLookups[threadIdx.x    ][threadIdx.y + 4];
	        dx -= 2.f*texLookups[threadIdx.x + 2][threadIdx.y + 4];
	        dx +=     texLookups[threadIdx.x + 4][threadIdx.y + 4];

	        dy -=     texLookups[threadIdx.x    ][threadIdx.y    ];
	        dy += 2.f*texLookups[threadIdx.x    ][threadIdx.y + 2];
	        dy -=     texLookups[threadIdx.x    ][threadIdx.y + 4];
	        dy +=     texLookups[threadIdx.x + 4][threadIdx.y    ];
	        dy -= 2.f*texLookups[threadIdx.x + 4][threadIdx.y + 2];
	        dy +=     texLookups[threadIdx.x + 4][threadIdx.y + 4];

	        float g = c_gauss1D[threadIdx.x] * c_gauss1D[threadIdx.y];

	        Edx[tid2] = dx * g;
	        Edy[tid2] = dy * g;
        }

        __syncthreads();

        // This is a scan to get the summed dx, dy values.
        // Gets 128-168
        if (tid < 41)
        {
            Edx[tid] += Edx[tid + 128]; 
        } 
        __syncthreads(); 
        if (tid < 64) 
        {
            Edx[tid] += Edx[tid + 64]; 
        } 
        __syncthreads(); 
        if (tid < 32) 
        {
            volatile float* smem = Edx;

            smem[tid] += smem[tid + 32];
            smem[tid] += smem[tid + 16];
            smem[tid] += smem[tid + 8];
            smem[tid] += smem[tid + 4];
            smem[tid] += smem[tid + 2];
            smem[tid] += smem[tid + 1];
        }

        // Gets 128-168
        if (tid < 41) 
        {
            Edy[tid] += Edy[tid + 128]; 
        } 
        __syncthreads(); 
        if (tid < 64) 
        {
            Edy[tid] += Edy[tid + 64]; 
        } 
        __syncthreads(); 
        if (tid < 32) 
        {
            volatile float* smem = Edy;

            smem[tid] += smem[tid + 32];
            smem[tid] += smem[tid + 16];
            smem[tid] += smem[tid + 8];
            smem[tid] += smem[tid + 4];
            smem[tid] += smem[tid + 2];
            smem[tid] += smem[tid + 1];
        }
 
        // Thread 0 saves back the result.
        if (tid == 0)
        {
	        features[blockIdx.x].angle = -atan2(Edy[0], Edx[0]) * (180.0f / CV_PI);
        }
    }

    void find_orientation_gpu(KeyPoint_GPU* features, int nFeatures) 
    {
        dim3 threads;
        threads.x = 17;
        threads.y = 17;

        dim3 grid;
        grid.x = nFeatures;
        grid.y = 1;
        grid.z = 1;

        find_orientation<<<grid, threads>>>(features);
        cudaSafeCall( hipDeviceSynchronize() );
    }

    ////////////////////////////////////////////////////////////////////////
    // Descriptors

    // precomputed values for a Gaussian with a standard deviation of 3.3
    // - it appears SURF uses a different value, but not sure what it is
    __constant__ float c_3p3gauss1D[20] = 
    {
        0.001917811039f, 0.004382549939f, 0.009136246641f, 0.017375153068f, 0.030144587513f,
		0.047710056854f, 0.068885910797f, 0.090734146446f, 0.109026229640f, 0.119511889092f,
		0.119511889092f, 0.109026229640f, 0.090734146446f, 0.068885910797f, 0.047710056854f,
		0.030144587513f, 0.017375153068f, 0.009136246641f, 0.004382549939f, 0.001917811039f
    };   

    template <int BLOCK_DIM_X>
    __global__ void normalize_descriptors(PtrStepf descriptors)
    {
        // no need for thread ID
        float* descriptor_base = descriptors.ptr(blockIdx.x);

        // read in the unnormalized descriptor values (squared)
        __shared__ float sqDesc[BLOCK_DIM_X];
        const float lookup = descriptor_base[threadIdx.x];
        sqDesc[threadIdx.x] = lookup * lookup;
        __syncthreads();

        if (BLOCK_DIM_X >= 128)
        {
            if (threadIdx.x < 64)
	            sqDesc[threadIdx.x] += sqDesc[threadIdx.x + 64];
            __syncthreads();
        }

        // reduction to get total
        if (threadIdx.x < 32)
        {
            volatile float* smem = sqDesc;

	        smem[threadIdx.x] += smem[threadIdx.x + 32];
	        smem[threadIdx.x] += smem[threadIdx.x + 16];
	        smem[threadIdx.x] += smem[threadIdx.x + 8];
	        smem[threadIdx.x] += smem[threadIdx.x + 4];
	        smem[threadIdx.x] += smem[threadIdx.x + 2];
	        smem[threadIdx.x] += smem[threadIdx.x + 1];
        }

        // compute length (square root)
        __shared__ float len;
        if (threadIdx.x == 0)
        {
	        len = sqrtf(sqDesc[0]);
        }
        __syncthreads();

        // normalize and store in output
        descriptor_base[threadIdx.x] = lookup / len;	
    }

    __device__ void calc_dx_dy(float sdx[4][4][25], float sdy[4][4][25], const KeyPoint_GPU* features)
    {
        // get the interest point parameters (x, y, size, response, angle)
        __shared__ float ipt[5];
        if (threadIdx.x < 5 && threadIdx.y == 0 && threadIdx.z == 0)
        {
	        ipt[threadIdx.x] = ((float*)(&features[blockIdx.x]))[threadIdx.x];
        }
        __syncthreads();

        float sin_theta, cos_theta;
        sincosf(ipt[SF_ANGLE] * (CV_PI / 180.0f), &sin_theta, &cos_theta);

        // Compute sampling points
        // since grids are 2D, need to compute xBlock and yBlock indices
        const int xIndex = threadIdx.y * 5 + threadIdx.x % 5;
        const int yIndex = threadIdx.z * 5 + threadIdx.x / 5;

        // Compute rotated sampling points
        // (clockwise rotation since we are rotating the lattice)
        // (subtract 9.5f to start sampling at the top left of the lattice, 0.5f is to space points out properly - there is no center pixel)
        const float sample_x = ipt[SF_X] + (cos_theta * ((float) (xIndex-9.5f)) * ipt[SF_SIZE] 
            + sin_theta * ((float) (yIndex-9.5f)) * ipt[SF_SIZE]);
        const float sample_y = ipt[SF_Y] + (-sin_theta * ((float) (xIndex-9.5f)) * ipt[SF_SIZE] 
            + cos_theta * ((float) (yIndex-9.5f)) * ipt[SF_SIZE]);

        // gather integral image lookups for Haar wavelets at each point (some lookups are shared between dx and dy)
        //	a b c
        //	d	f
        //	g h i

        const float a = tex2D(sumTex, sample_x - ipt[SF_SIZE], sample_y - ipt[SF_SIZE]);
        const float b = tex2D(sumTex, sample_x,                sample_y - ipt[SF_SIZE]);
        const float c = tex2D(sumTex, sample_x + ipt[SF_SIZE], sample_y - ipt[SF_SIZE]);
        const float d = tex2D(sumTex, sample_x - ipt[SF_SIZE], sample_y);
        const float f = tex2D(sumTex, sample_x + ipt[SF_SIZE], sample_y);
        const float g = tex2D(sumTex, sample_x - ipt[SF_SIZE], sample_y + ipt[SF_SIZE]);
        const float h = tex2D(sumTex, sample_x,                sample_y + ipt[SF_SIZE]);
        const float i = tex2D(sumTex, sample_x + ipt[SF_SIZE], sample_y + ipt[SF_SIZE]);

        // compute axis-aligned HaarX, HaarY
        // (could group the additions together into multiplications)
        const float gauss = c_3p3gauss1D[xIndex] * c_3p3gauss1D[yIndex]; // separable because independent (circular)
        const float aa_dx = gauss * (-(a-b-g+h) + (b-c-h+i));            // unrotated dx
        const float aa_dy = gauss * (-(a-c-d+f) + (d-f-g+i));            // unrotated dy

        // rotate responses (store all dxs then all dys)
        // - counterclockwise rotation to rotate back to zero orientation
        sdx[threadIdx.z][threadIdx.y][threadIdx.x] = aa_dx * cos_theta - aa_dy * sin_theta; // rotated dx
        sdy[threadIdx.z][threadIdx.y][threadIdx.x] = aa_dx * sin_theta + aa_dy * cos_theta; // rotated dy
    }

    __device__ void reduce_sum(float sdata1[4][4][25], float sdata2[4][4][25], float sdata3[4][4][25],
        float sdata4[4][4][25])
    {
        // first step is to reduce from 25 to 16
        if (threadIdx.x < 9) // use 9 threads
        {
	        sdata1[threadIdx.z][threadIdx.y][threadIdx.x] += sdata1[threadIdx.z][threadIdx.y][threadIdx.x + 16];
	        sdata2[threadIdx.z][threadIdx.y][threadIdx.x] += sdata2[threadIdx.z][threadIdx.y][threadIdx.x + 16];
	        sdata3[threadIdx.z][threadIdx.y][threadIdx.x] += sdata3[threadIdx.z][threadIdx.y][threadIdx.x + 16];
	        sdata4[threadIdx.z][threadIdx.y][threadIdx.x] += sdata4[threadIdx.z][threadIdx.y][threadIdx.x + 16];
        }
        __syncthreads();

        // sum (reduce) from 16 to 1 (unrolled - aligned to a half-warp)
        if (threadIdx.x < 16)
        {
            volatile float* smem = sdata1[threadIdx.z][threadIdx.y];

	        smem[threadIdx.x] += smem[threadIdx.x + 8];
	        smem[threadIdx.x] += smem[threadIdx.x + 4];
	        smem[threadIdx.x] += smem[threadIdx.x + 2];
	        smem[threadIdx.x] += smem[threadIdx.x + 1];

            smem = sdata2[threadIdx.z][threadIdx.y];

	        smem[threadIdx.x] += smem[threadIdx.x + 8];
	        smem[threadIdx.x] += smem[threadIdx.x + 4];
	        smem[threadIdx.x] += smem[threadIdx.x + 2];
	        smem[threadIdx.x] += smem[threadIdx.x + 1];

            smem = sdata3[threadIdx.z][threadIdx.y];

	        smem[threadIdx.x] += smem[threadIdx.x + 8];
	        smem[threadIdx.x] += smem[threadIdx.x + 4];
	        smem[threadIdx.x] += smem[threadIdx.x + 2];
	        smem[threadIdx.x] += smem[threadIdx.x + 1];

            smem = sdata4[threadIdx.z][threadIdx.y];

	        smem[threadIdx.x] += smem[threadIdx.x + 8];
	        smem[threadIdx.x] += smem[threadIdx.x + 4];
	        smem[threadIdx.x] += smem[threadIdx.x + 2];
	        smem[threadIdx.x] += smem[threadIdx.x + 1];
        }
    }

    // Spawn 16 blocks per interest point
    // - computes unnormalized 64 dimensional descriptor, puts it into d_descriptors in the correct location
    __global__ void compute_descriptors64(PtrStepf descriptors, const KeyPoint_GPU* features)
    {        
        // 2 floats (dx, dy) for each thread (5x5 sample points in each sub-region)
        __shared__ float sdx[4][4][25]; 
        __shared__ float sdy[4][4][25];

        calc_dx_dy(sdx, sdy, features);
        __syncthreads();

        __shared__ float sdxabs[4][4][25];
        __shared__ float sdyabs[4][4][25];
        
        sdxabs[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdx[threadIdx.z][threadIdx.y][threadIdx.x]); // |dx| array
        sdyabs[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdy[threadIdx.z][threadIdx.y][threadIdx.x]); // |dy| array
        __syncthreads();

        reduce_sum(sdx, sdy, sdxabs, sdyabs);

        float* descriptors_block = descriptors.ptr(blockIdx.x) + threadIdx.z * 16 + threadIdx.y * 4;

        // write dx, dy, |dx|, |dy|
        if (threadIdx.x == 0)
        {
            descriptors_block[0] = sdx[threadIdx.z][threadIdx.y][0];
            descriptors_block[1] = sdy[threadIdx.z][threadIdx.y][0];
            descriptors_block[2] = sdxabs[threadIdx.z][threadIdx.y][0];
            descriptors_block[3] = sdyabs[threadIdx.z][threadIdx.y][0];
        }
    }    

    // Spawn 16 blocks per interest point
    // - computes unnormalized 128 dimensional descriptor, puts it into d_descriptors in the correct location
    __global__ void compute_descriptors128(PtrStepf descriptors, const KeyPoint_GPU* features)
    {        
        // 2 floats (dx,dy) for each thread (5x5 sample points in each sub-region)
        __shared__ float sdx[4][4][25]; 
        __shared__ float sdy[4][4][25];
        
        calc_dx_dy(sdx, sdy, features);
        __syncthreads();

        // sum (reduce) 5x5 area response
        __shared__ float sd1[4][4][25];
        __shared__ float sd2[4][4][25];
        __shared__ float sdabs1[4][4][25]; 
        __shared__ float sdabs2[4][4][25];

        if (sdy[threadIdx.z][threadIdx.y][threadIdx.x] >= 0)
        {
            sd1[threadIdx.z][threadIdx.y][threadIdx.x] = sdx[threadIdx.z][threadIdx.y][threadIdx.x];
            sdabs1[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdx[threadIdx.z][threadIdx.y][threadIdx.x]);
            sd2[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sdabs2[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            sd1[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sdabs1[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sd2[threadIdx.z][threadIdx.y][threadIdx.x] = sdx[threadIdx.z][threadIdx.y][threadIdx.x];
            sdabs2[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdx[threadIdx.z][threadIdx.y][threadIdx.x]);
        }
        __syncthreads();
        
        reduce_sum(sd1, sd2, sdabs1, sdabs2);
        
        float* descriptors_block = descriptors.ptr(blockIdx.x) + threadIdx.z * 32 + threadIdx.y * 8;

        // write dx (dy >= 0), |dx| (dy >= 0), dx (dy < 0), |dx| (dy < 0)
        if (threadIdx.x == 0)
        {
	        descriptors_block[0] = sd1[threadIdx.z][threadIdx.y][0];
	        descriptors_block[1] = sdabs1[threadIdx.z][threadIdx.y][0];
	        descriptors_block[2] = sd2[threadIdx.z][threadIdx.y][0];
	        descriptors_block[3] = sdabs2[threadIdx.z][threadIdx.y][0];
        }
        __syncthreads();

        if (sdx[threadIdx.z][threadIdx.y][threadIdx.x] >= 0)
        {
            sd1[threadIdx.z][threadIdx.y][threadIdx.x] = sdy[threadIdx.z][threadIdx.y][threadIdx.x];
            sdabs1[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdy[threadIdx.z][threadIdx.y][threadIdx.x]);
            sd2[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sdabs2[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            sd1[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sdabs1[threadIdx.z][threadIdx.y][threadIdx.x] = 0;
            sd2[threadIdx.z][threadIdx.y][threadIdx.x] = sdy[threadIdx.z][threadIdx.y][threadIdx.x];
            sdabs2[threadIdx.z][threadIdx.y][threadIdx.x] = fabs(sdy[threadIdx.z][threadIdx.y][threadIdx.x]);
        }
        __syncthreads();

        reduce_sum(sd1, sd2, sdabs1, sdabs2);

        // write dy (dx >= 0), |dy| (dx >= 0), dy (dx < 0), |dy| (dx < 0)
        if (threadIdx.x == 0)
        {
	        descriptors_block[4] = sd1[threadIdx.z][threadIdx.y][0];
	        descriptors_block[5] = sdabs1[threadIdx.z][threadIdx.y][0];
	        descriptors_block[6] = sd2[threadIdx.z][threadIdx.y][0];
	        descriptors_block[7] = sdabs2[threadIdx.z][threadIdx.y][0];
        }
    }

    void compute_descriptors_gpu(const DevMem2Df& descriptors, const KeyPoint_GPU* features, int nFeatures)
    {
        // compute unnormalized descriptors, then normalize them - odd indexing since grid must be 2D
        
        if (descriptors.cols == 64)
        {
            compute_descriptors64<<<dim3(nFeatures, 1, 1), dim3(25, 4, 4)>>>(descriptors, features);
            cudaSafeCall( hipDeviceSynchronize() );

            normalize_descriptors<64><<<dim3(nFeatures, 1, 1), dim3(64, 1, 1)>>>(descriptors);
            cudaSafeCall( hipDeviceSynchronize() );
        }
        else
        {
            compute_descriptors128<<<dim3(nFeatures, 1, 1), dim3(25, 4, 4)>>>(descriptors, features);
            cudaSafeCall( hipDeviceSynchronize() );

            normalize_descriptors<128><<<dim3(nFeatures, 1, 1), dim3(128, 1, 1)>>>(descriptors);
            cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}
