#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/transform.hpp"

namespace cv { namespace gpu
{
    namespace transform_points
    {
        __constant__ float3 crot0;
        __constant__ float3 crot1;
        __constant__ float3 crot2;
        __constant__ float3 ctransl;

        struct TransformOp
        {
            __device__ float3 operator()(float3 p) const
            {
                return make_float3(
                        crot0.x * p.x + crot0.y * p.y + crot0.z * p.z + ctransl.x,
                        crot1.x * p.x + crot1.y * p.y + crot1.z * p.z + ctransl.y,
                        crot2.x * p.x + crot2.y * p.y + crot2.z * p.z + ctransl.z);
            }
        };

        void call(const DevMem2D_<float3> src, const float* rot,
                  const float* transl, DevMem2D_<float3> dst)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot0), rot, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot1), rot + 3, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot2), rot + 6, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctransl), transl, sizeof(float) * 3));
            transform(src, dst, TransformOp());
        }
    } // namespace transform_points

    namespace project_points
    {
        __constant__ float3 crot0;
        __constant__ float3 crot1;
        __constant__ float3 crot2;
        __constant__ float3 ctransl;
        __constant__ float3 cproj0;
        __constant__ float3 cproj1;

        struct ProjectOp
        {
            __device__ float2 operator()(float3 p) const
            {
                // Rotate and translate in 3D
                float3 t = make_float3(
                        crot0.x * p.x + crot0.y * p.y + crot0.z * p.z + ctransl.x,
                        crot1.x * p.x + crot1.y * p.y + crot1.z * p.z + ctransl.y,
                        crot2.x * p.x + crot2.y * p.y + crot2.z * p.z + ctransl.z);
                // Project on 2D plane
                return make_float2(
                        (cproj0.x * t.x + cproj0.y * t.y) / t.z + cproj0.z,
                        (cproj1.x * t.x + cproj1.y * t.y) / t.z + cproj1.z);
            }
        };

        void call(const DevMem2D_<float3> src, const float* rot,
                  const float* transl, const float* proj, DevMem2D_<float2> dst)
        {
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot0), rot, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot1), rot + 3, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(crot2), rot + 6, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctransl), transl, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cproj0), proj, sizeof(float) * 3));
            cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cproj1), proj + 3, sizeof(float) * 3));
            transform(src, dst, ProjectOp());
        }
    } // namespace project_points

}} // namespace cv { namespace gpu
