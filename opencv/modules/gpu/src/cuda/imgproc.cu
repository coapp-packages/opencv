#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"

using namespace cv::gpu;


/////////////////////////////////// Remap ///////////////////////////////////////////////
namespace imgproc
{
    texture<unsigned char, 2, hipReadModeNormalizedFloat> tex_remap;

    __global__ void kernel_remap(const float *mapx, const float *mapy, size_t map_step, unsigned char* out, size_t out_step, int width, int height)
    {    
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            float xcoo = mapx[idx];
            float ycoo = mapy[idx];

            out[y * out_step + x] = (unsigned char)(255.f * tex2D(tex_remap, xcoo, ycoo));            
        }
    }

}

namespace cv { namespace gpu { namespace impl 
{
    extern "C" void remap_gpu(const DevMem2D& src, const DevMem2D_<float>& xmap, const DevMem2D_<float>& ymap, DevMem2D dst)
    {
        dim3 block(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, block.x);
        grid.y = divUp(dst.rows, block.y);

        imgproc::tex_remap.filterMode = hipFilterModeLinear;	    
        imgproc::tex_remap.addressMode[0] = imgproc::tex_remap.addressMode[1] = hipAddressModeWrap;
        hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
        cudaSafeCall( hipBindTexture2D(0, imgproc::tex_remap, src.ptr, desc, dst.cols, dst.rows, src.step) );

        imgproc::kernel_remap<<<grid, block>>>(xmap.ptr, ymap.ptr, xmap.step, dst.ptr, dst.step, dst.cols, dst.rows);

        cudaSafeCall( hipDeviceSynchronize() );  
        cudaSafeCall( hipUnbindTexture(imgproc::tex_remap) );
    }
}}}


/////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

namespace imgproc
{
    texture<uchar4, 2> tex_meanshift;

    extern "C" __global__ void meanshift_kernel( unsigned char* out, int out_step, int cols, int rows, int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
        {
            int isr2 = sr*sr;
            uchar4 c = tex2D(tex_meanshift, x0, y0 );
            // iterate meanshift procedure
            for( int iter = 0; iter < maxIter; iter++ )
            {
                int count = 0;
                int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
                float icount;

                //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
                int minx = x0-sp;
                int miny = y0-sp;
                int maxx = x0+sp;
                int maxy = y0+sp;

                for( int y = miny; y <= maxy; y++)
                {
                    int rowCount = 0;
                    for( int x = minx; x <= maxx; x++ )
                    {                    
                        uchar4 t = tex2D( tex_meanshift, x, y );

                        int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                        if( norm2 <= isr2 )
                        {
                            s0 += t.x; s1 += t.y; s2 += t.z;
                            sx += x; rowCount++;
                        }
                    }
                    count += rowCount;
                    sy += y*rowCount;
                }

                if( count == 0 )
                    break;

                icount = 1.f/count;
                int x1 = __float2int_rz(sx*icount);
                int y1 = __float2int_rz(sy*icount);
                s0 = __float2int_rz(s0*icount);
                s1 = __float2int_rz(s1*icount);
                s2 = __float2int_rz(s2*icount);

                int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

                bool stopFlag = (x0 == x1 && y0 == y1) || (abs(x1-x0) + abs(y1-y0) + norm2 <= eps);

                x0 = x1; y0 = y1;
                c.x = s0; c.y = s1; c.z = s2;

                if( stopFlag )
                    break;
            }

            int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 3 * sizeof(uchar);
            out[base+0] = c.x;
            out[base+1] = c.y;
            out[base+2] = c.z;
        }
    }
}

namespace cv { namespace gpu { namespace impl 
{
    extern "C" void meanShiftFiltering_gpu(const DevMem2D& src, DevMem2D dst, int sp, int sr, int maxIter, float eps)
    {                        
        dim3  grid(1, 1, 1);
        dim3 threads(32, 16, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, imgproc::tex_meanshift, src.ptr, desc, src.cols, src.rows, src.step ) );

        imgproc::meanshift_kernel<<< grid, threads >>>( dst.ptr, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( imgproc::tex_meanshift ) );        
    }
}}}


