#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"

using namespace cv::gpu;

namespace imgproc
{
    texture<unsigned char, 2, hipReadModeNormalizedFloat> tex;

    __global__ void kernel_remap(const float *mapx, const float *mapy, size_t map_step, unsigned char* out, size_t out_step, int width, int height)
    {    
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            float xcoo = mapx[idx];
            float ycoo = mapy[idx];
            
            out[y * out_step + x] = (unsigned char)(255.f * tex2D(tex, xcoo, ycoo));            
        }
    }
}

namespace cv { namespace gpu { namespace impl {
    extern "C" void remap_gpu(const DevMem2D& src, const DevMem2D_<float>& xmap, const DevMem2D_<float>& ymap, DevMem2D dst, size_t width, size_t height)
    {
        dim3 block(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp( width, block.x);
        grid.y = divUp(height, block.y);

        ::imgproc::tex.filterMode = hipFilterModeLinear;	    
        ::imgproc::tex.addressMode[0] = ::imgproc::tex.addressMode[1] = hipAddressModeWrap;
        hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
        cudaSafeCall( hipBindTexture2D(0, ::imgproc::tex, src.ptr, desc, width, height, src.step) );

        ::imgproc::kernel_remap<<<grid, block>>>(xmap.ptr, ymap.ptr, xmap.step, dst.ptr, dst.step, width, height);

        cudaSafeCall( hipDeviceSynchronize() );  
        cudaSafeCall( hipUnbindTexture(::imgproc::tex) );
    }
}}}