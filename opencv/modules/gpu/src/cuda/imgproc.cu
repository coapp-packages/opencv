#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"

using namespace cv::gpu;


/////////////////////////////////// Remap ///////////////////////////////////////////////
namespace imgproc
{
    texture<unsigned char, 2, hipReadModeNormalizedFloat> tex_remap;

    __global__ void kernel_remap(const float *mapx, const float *mapy, size_t map_step, unsigned char* out, size_t out_step, int width, int height)
    {    
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            float xcoo = mapx[idx];
            float ycoo = mapy[idx];

            out[y * out_step + x] = (unsigned char)(255.f * tex2D(tex_remap, xcoo, ycoo));            
        }
    }

}

namespace cv { namespace gpu { namespace impl 
{
    extern "C" void remap_gpu(const DevMem2D& src, const DevMem2D_<float>& xmap, const DevMem2D_<float>& ymap, DevMem2D dst)
    {
        dim3 block(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, block.x);
        grid.y = divUp(dst.rows, block.y);

        imgproc::tex_remap.filterMode = hipFilterModeLinear;	    
        imgproc::tex_remap.addressMode[0] = imgproc::tex_remap.addressMode[1] = hipAddressModeWrap;
        hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
        cudaSafeCall( hipBindTexture2D(0, imgproc::tex_remap, src.ptr, desc, dst.cols, dst.rows, src.step) );

        imgproc::kernel_remap<<<grid, block>>>(xmap.ptr, ymap.ptr, xmap.step, dst.ptr, dst.step, dst.cols, dst.rows);

        cudaSafeCall( hipDeviceSynchronize() );  
        cudaSafeCall( hipUnbindTexture(imgproc::tex_remap) );
    }
}}}


/////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

namespace imgproc
{
    texture<uchar4, 2> tex_meanshift;

    extern "C" __global__ void meanshift_kernel( unsigned char* out, int out_step, int cols, int rows, int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
        {
            int isr2 = sr*sr;
            uchar4 c = tex2D(tex_meanshift, x0, y0 );
            // iterate meanshift procedure
            for( int iter = 0; iter < maxIter; iter++ )
            {
                int count = 0;
                int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
                float icount;

                //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
                int minx = x0-sp;
                int miny = y0-sp;
                int maxx = x0+sp;
                int maxy = y0+sp;

                for( int y = miny; y <= maxy; y++)
                {
                    int rowCount = 0;
                    for( int x = minx; x <= maxx; x++ )
                    {                    
                        uchar4 t = tex2D( tex_meanshift, x, y );

                        int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                        if( norm2 <= isr2 )
                        {
                            s0 += t.x; s1 += t.y; s2 += t.z;
                            sx += x; rowCount++;
                        }
                    }
                    count += rowCount;
                    sy += y*rowCount;
                }

                if( count == 0 )
                    break;

                icount = 1.f/count;
                int x1 = __float2int_rz(sx*icount);
                int y1 = __float2int_rz(sy*icount);
                s0 = __float2int_rz(s0*icount);
                s1 = __float2int_rz(s1*icount);
                s2 = __float2int_rz(s2*icount);

                int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

                bool stopFlag = (x0 == x1 && y0 == y1) || (abs(x1-x0) + abs(y1-y0) + norm2 <= eps);

                x0 = x1; y0 = y1;
                c.x = s0; c.y = s1; c.z = s2;

                if( stopFlag )
                    break;
            }

            int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 3 * sizeof(uchar);
            out[base+0] = c.x;
            out[base+1] = c.y;
            out[base+2] = c.z;
        }
    }
}

namespace cv { namespace gpu { namespace impl 
{
    extern "C" void meanShiftFiltering_gpu(const DevMem2D& src, DevMem2D dst, int sp, int sr, int maxIter, float eps)
    {                        
        dim3  grid(1, 1, 1);
        dim3 threads(32, 16, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, imgproc::tex_meanshift, src.ptr, desc, src.cols, src.rows, src.step ) );

        imgproc::meanshift_kernel<<< grid, threads >>>( dst.ptr, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( imgproc::tex_meanshift ) );        
    }
}}}

/////////////////////////////////// colorizeDisp ///////////////////////////////////////////////

namespace imgproc
{
    template <typename T>
    __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
    {        
        unsigned int H = ((ndisp-d) * 240)/ndisp;

        unsigned int hi = (H/60) % 6;
        float f = H/60.f - H/60;
        float p = V * (1 - S);
        float q = V * (1 - f * S);
        float t = V * (1 - (1 - f) * S);

        float3 res;
        
        if (hi == 0) //R = V,	G = t,	B = p
        {
            res.x = p;
            res.y = t;
            res.z = V;
        }

        if (hi == 1) // R = q,	G = V,	B = p
        {
            res.x = p;
            res.y = V;
            res.z = q;
        }        
        
        if (hi == 2) // R = p,	G = V,	B = t
        {
            res.x = t;
            res.y = V;
            res.z = p;
        }
            
        if (hi == 3) // R = p,	G = q,	B = V
        {
            res.x = V;
            res.y = q;
            res.z = p;
        }

        if (hi == 4) // R = t,	G = p,	B = V
        {
            res.x = V;
            res.y = p;
            res.z = t;
        }

        if (hi == 5) // R = V,	G = p,	B = q
        {
            res.x = q;
            res.y = p;
            res.z = V;
        }
        unsigned int b = (unsigned int)(max(0.f, min (res.x, 1.f)) * 255.f);
        unsigned int g = (unsigned int)(max(0.f, min (res.y, 1.f)) * 255.f);
        unsigned int r = (unsigned int)(max(0.f, min (res.z, 1.f)) * 255.f);

        return (r << 16) + (g << 8) + b;    
    } 

    __global__ void colorizeDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

            uint4 res;
            res.x = cvtPixel(d4.x, ndisp);
            res.y = cvtPixel(d4.y, ndisp);
            res.z = cvtPixel(d4.z, ndisp);
            res.w = cvtPixel(d4.w, ndisp);
                    
            uint4* line = (uint4*)(out_image + y * out_step);
            line[x >> 2] = res;
        }
    }

    __global__ void colorizeDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            short2 d2 = *(short2*)(disp + y * disp_step + x);

            uint2 res;
            res.x = cvtPixel(d2.x, ndisp);            
            res.y = cvtPixel(d2.y, ndisp);

            uint2* line = (uint2*)(out_image + y * out_step);
            line[x >> 1] = res;
        }
    }
}

namespace cv { namespace gpu { namespace impl 
{
    void colorizeDisp_gpu(const DevMem2D& src, const DevMem2D& dst, int ndisp)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 2);
        grid.y = divUp(src.rows, threads.y);
         
        imgproc::colorizeDisp<<<grid, threads>>>(src.ptr, src.step, dst.ptr, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipDeviceSynchronize() ); 
    }

    void colorizeDisp_gpu(const DevMem2D_<short>& src, const DevMem2D& dst, int ndisp)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 1);
        grid.y = divUp(src.rows, threads.y);
         
        imgproc::colorizeDisp<<<grid, threads>>>(src.ptr, src.step / sizeof(short), dst.ptr, dst.step, src.cols, src.rows, ndisp);
        cudaSafeCall( hipDeviceSynchronize() );
    }
}}}

/////////////////////////////////// colorizeDisp ///////////////////////////////////////////////

namespace imgproc
{
    __constant__ float cq[16];

    template <typename T>
    __global__ void reprojectImageTo3D(const T* disp, size_t disp_step, float* xyzw, size_t xyzw_step, int rows, int cols)
    {        
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y < rows && x < cols)
        {

            float qx = cq[1] * y + cq[3], qy = cq[5] * y + cq[7];
            float qz = cq[9] * y + cq[11], qw = cq[13] * y + cq[15];

            qx += x * cq[0]; 
            qy += x * cq[4];
            qz += x * cq[8];
            qw += x * cq[12];

            T d = *(disp + disp_step * y + x);

            float iW = 1.f / (qw + cq[14] * d);
            float4 v;
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;
            v.w = 1.f;

            *(float4*)(xyzw + xyzw_step * y + (x * 4)) = v;
        }
    }
}

namespace cv { namespace gpu { namespace impl 
{
    template <typename T>
    inline void reprojectImageTo3D_caller(const DevMem2D_<T>& disp, const DevMem2Df& xyzw, const float* q)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(imgproc::cq), q, 16 * sizeof(float)) );

        imgproc::reprojectImageTo3D<<<grid, threads>>>(disp.ptr, disp.step / sizeof(T), xyzw.ptr, xyzw.step / sizeof(float), disp.rows, disp.cols);

        cudaSafeCall( hipDeviceSynchronize() );
    }

    void reprojectImageTo3D_gpu(const DevMem2D& disp, const DevMem2Df& xyzw, const float* q)
    {
        reprojectImageTo3D_caller(disp, xyzw, q);
    }

    void reprojectImageTo3D_gpu(const DevMem2D_<short>& disp, const DevMem2Df& xyzw, const float* q)
    {
        reprojectImageTo3D_caller(disp, xyzw, q);
    }
}}}
