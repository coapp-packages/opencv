#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "opencv2/gpu/device/vecmath.hpp"
#include "opencv2/gpu/device/limits_gpu.hpp"
#include "opencv2/gpu/device/transform.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

#ifndef CV_DESCALE
#define CV_DESCALE(x, n) (((x) + (1 << ((n)-1))) >> (n))
#endif

namespace cv { namespace gpu { namespace color
{
    template<typename T> struct ColorChannel
    {
        typedef float worktype_f;
        static __device__ __forceinline__ T max() { return numeric_limits_gpu<T>::max(); }
        static __device__ __forceinline__ T half() { return (T)(max()/2 + 1); }
    };
    template<> struct ColorChannel<float>
    {
        typedef float worktype_f;
        static __device__ __forceinline__ float max() { return 1.f; }
        static __device__ __forceinline__ float half() { return 0.5f; }
    };

    template <typename T>
    __device__ __forceinline__ void setAlpha(typename TypeVec<T, 3>::vec_t& vec, T val)
    {
    }
    template <typename T>
    __device__ __forceinline__ void setAlpha(typename TypeVec<T, 4>::vec_t& vec, T val)
    {
        vec.w = val;
    }
    template <typename T>
    __device__ __forceinline__ T getAlpha(const typename TypeVec<T, 3>::vec_t& vec)
    {
        return ColorChannel<T>::max();
    }
    template <typename T>
    __device__ __forceinline__ T getAlpha(const typename TypeVec<T, 4>::vec_t& vec)
    {
        return vec.w;
    }

    template <typename Cvt>
    void callConvert(const DevMem2D& src, const DevMem2D& dst, const Cvt& cvt, hipStream_t stream)
    {
        typedef typename Cvt::src_t src_t;
        typedef typename Cvt::dst_t dst_t;

        transform((DevMem2D_<src_t>)src, (DevMem2D_<dst_t>)dst, cvt, stream);
    }

////////////////// Various 3/4-channel to 3/4-channel RGB transformations /////////////////

    template <typename T, int SRCCN, int DSTCN>
    struct RGB2RGB
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        explicit RGB2RGB(int bidx) : bidx(bidx) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;

            dst.x = (&src.x)[bidx];
            dst.y = src.y;
            dst.z = (&src.x)[bidx ^ 2];
            setAlpha(dst, getAlpha<T>(src));

            return dst;
        }

    private:
        int bidx;
    };

    template <typename T, int SRCCN, int DSTCN>
    void RGB2RGB_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        RGB2RGB<T, SRCCN, DSTCN> cvt(bidx);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2RGB_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        static const RGB2RGB_caller_t RGB2RGB_callers[2][2] =
        {
            {RGB2RGB_caller<uchar, 3, 3>, RGB2RGB_caller<uchar, 3, 4>},
            {RGB2RGB_caller<uchar, 4, 3>, RGB2RGB_caller<uchar, 4, 4>}
        };

        RGB2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, stream);
    }

    void RGB2RGB_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        static const RGB2RGB_caller_t RGB2RGB_callers[2][2] =
        {
            {RGB2RGB_caller<ushort, 3, 3>, RGB2RGB_caller<ushort, 3, 4>},
            {RGB2RGB_caller<ushort, 4, 3>, RGB2RGB_caller<ushort, 4, 4>}
        };

        RGB2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, stream);
    }

    void RGB2RGB_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        static const RGB2RGB_caller_t RGB2RGB_callers[2][2] =
        {
            {RGB2RGB_caller<float, 3, 3>, RGB2RGB_caller<float, 3, 4>},
            {RGB2RGB_caller<float, 4, 3>, RGB2RGB_caller<float, 4, 4>}
        };

        RGB2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, stream);
    }

/////////// Transforming 16-bit (565 or 555) RGB to/from 24/32-bit (888[8]) RGB //////////

    template <int GREEN_BITS> struct RGB5x52RGBConverter;
    template <> struct RGB5x52RGBConverter<5>
    {
        template <typename D>
        static __device__ __forceinline__ void cvt(uint src, D& dst, int bidx)
        {
            (&dst.x)[bidx] = (uchar)(src << 3);
            dst.y = (uchar)((src >> 2) & ~7);
            (&dst.x)[bidx ^ 2] = (uchar)((src >> 7) & ~7);
            setAlpha(dst, (uchar)(src & 0x8000 ? 255 : 0));
        }
    };
    template <> struct RGB5x52RGBConverter<6>
    {
        template <typename D>
        static __device__ __forceinline__ void cvt(uint src, D& dst, int bidx)
        {
            (&dst.x)[bidx] = (uchar)(src << 3);
            dst.y = (uchar)((src >> 3) & ~3);
            (&dst.x)[bidx ^ 2] = (uchar)((src >> 8) & ~7);
            setAlpha(dst, (uchar)(255));
        }
    };

    template <int GREEN_BITS, int DSTCN> struct RGB5x52RGB
    {
        typedef ushort src_t;
        typedef typename TypeVec<uchar, DSTCN>::vec_t dst_t;

        explicit RGB5x52RGB(int bidx) : bidx(bidx) {}

        __device__ __forceinline__ dst_t operator()(ushort src) const
        {
            dst_t dst;
            RGB5x52RGBConverter<GREEN_BITS>::cvt((uint)src, dst, bidx);
            return dst;
        }

    private:
        int bidx;
    };

    template <int GREEN_BITS> struct RGB2RGB5x5Converter;
    template<> struct RGB2RGB5x5Converter<6>
    {
        template <typename T>
        static __device__ __forceinline__ ushort cvt(const T& src, int bidx)
        {
            return (ushort)(((&src.x)[bidx] >> 3) | ((src.y & ~3) << 3) | (((&src.x)[bidx^2] & ~7) << 8));
        }
    };
    template<> struct RGB2RGB5x5Converter<5>
    {
        static __device__ __forceinline__ ushort cvt(const uchar3& src, int bidx)
        {
            return (ushort)(((&src.x)[bidx] >> 3) | ((src.y & ~7) << 2) | (((&src.x)[bidx^2] & ~7) << 7));
        }
        static __device__ __forceinline__ ushort cvt(const uchar4& src, int bidx)
        {
            return (ushort)(((&src.x)[bidx] >> 3) | ((src.y & ~7) << 2) | (((&src.x)[bidx^2] & ~7) << 7) | (src.w ? 0x8000 : 0));
        }
    };

    template<int SRCCN, int GREEN_BITS> struct RGB2RGB5x5
    {
        typedef typename TypeVec<uchar, SRCCN>::vec_t src_t;
        typedef ushort dst_t;

        explicit RGB2RGB5x5(int bidx) : bidx(bidx) {}

        __device__ __forceinline__ ushort operator()(const src_t& src)
        {
            return RGB2RGB5x5Converter<GREEN_BITS>::cvt(src, bidx);
        }

    private:
        int bidx;
    };

    template <int GREEN_BITS, int DSTCN>
    void RGB5x52RGB_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        RGB5x52RGB<GREEN_BITS, DSTCN> cvt(bidx);
        callConvert(src, dst, cvt, stream);
    }

    void RGB5x52RGB_gpu(const DevMem2D& src, int green_bits, const DevMem2D& dst, int dstcn, int bidx, hipStream_t stream)
    {
        typedef void (*RGB5x52RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        static const RGB5x52RGB_caller_t RGB5x52RGB_callers[2][2] =
        {
            {RGB5x52RGB_caller<5, 3>, RGB5x52RGB_caller<5, 4>},
            {RGB5x52RGB_caller<6, 3>, RGB5x52RGB_caller<6, 4>}
        };

        RGB5x52RGB_callers[green_bits - 5][dstcn - 3](src, dst, bidx, stream);
    }

    template <int SRCCN, int GREEN_BITS>
    void RGB2RGB5x5_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        RGB2RGB5x5<SRCCN, GREEN_BITS> cvt(bidx);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2RGB5x5_gpu(const DevMem2D& src, int srccn, const DevMem2D& dst, int green_bits, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2RGB5x5_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        static const RGB2RGB5x5_caller_t RGB2RGB5x5_callers[2][2] =
        {
            {RGB2RGB5x5_caller<3, 5>, RGB2RGB5x5_caller<3, 6>},
            {RGB2RGB5x5_caller<4, 5>, RGB2RGB5x5_caller<4, 6>}
        };

        RGB2RGB5x5_callers[srccn - 3][green_bits - 5](src, dst, bidx, stream);
    }

///////////////////////////////// Grayscale to Color ////////////////////////////////

    template <int DSTCN, typename T> struct Gray2RGB
    {
        typedef T src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        __device__ __forceinline__ dst_t operator()(const T& src) const
        {
            dst_t dst;

            dst.z = dst.y = dst.x = src;
            setAlpha(dst, ColorChannel<T>::max());

            return dst;
        }
    };

    template <int GREEN_BITS> struct Gray2RGB5x5Converter;
    template<> struct Gray2RGB5x5Converter<6>
    {
        static __device__ __forceinline__ ushort cvt(uint t)
        {
            return (ushort)((t >> 3) | ((t & ~3) << 3) | ((t & ~7) << 8));
        }
    };
    template<> struct Gray2RGB5x5Converter<5>
    {
        static __device__ __forceinline__ ushort cvt(uint t)
        {
            t >>= 3;
            return (ushort)(t | (t << 5) | (t << 10));
        }
    };

    template<int GREEN_BITS> struct Gray2RGB5x5
    {
        typedef uchar src_t;
        typedef ushort dst_t;

        __device__ __forceinline__ ushort operator()(uchar src) const
        {
            return Gray2RGB5x5Converter<GREEN_BITS>::cvt((uint)src);
        }
    };

    template <typename T, int DSTCN>
    void Gray2RGB_caller(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream)
    {
        Gray2RGB<DSTCN, T> cvt;
        callConvert(src, dst, cvt, stream);
    }

    void Gray2RGB_gpu_8u(const DevMem2D& src, const DevMem2D& dst, int dstcn, hipStream_t stream)
    {
        typedef void (*Gray2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream);
        static const Gray2RGB_caller_t Gray2RGB_callers[] = {Gray2RGB_caller<uchar, 3>, Gray2RGB_caller<uchar, 4>};

        Gray2RGB_callers[dstcn - 3](src, dst, stream);
    }

    void Gray2RGB_gpu_16u(const DevMem2D& src, const DevMem2D& dst, int dstcn, hipStream_t stream)
    {
        typedef void (*Gray2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream);
        static const Gray2RGB_caller_t Gray2RGB_callers[] = {Gray2RGB_caller<ushort, 3>, Gray2RGB_caller<ushort, 4>};

        Gray2RGB_callers[dstcn - 3](src, dst, stream);
    }

    void Gray2RGB_gpu_32f(const DevMem2D& src, const DevMem2D& dst, int dstcn, hipStream_t stream)
    {
        typedef void (*Gray2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream);
        static const Gray2RGB_caller_t Gray2RGB_callers[] = {Gray2RGB_caller<float, 3>, Gray2RGB_caller<float, 4>};

        Gray2RGB_callers[dstcn - 3](src, dst, stream);
    }

    template <int GREEN_BITS>
    void Gray2RGB5x5_caller(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream)
    {
        Gray2RGB5x5<GREEN_BITS> cvt;
        callConvert(src, dst, cvt, stream);
    }

    void Gray2RGB5x5_gpu(const DevMem2D& src, const DevMem2D& dst, int green_bits, hipStream_t stream)
    {
        typedef void (*Gray2RGB5x5_caller_t)(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream);
        static const Gray2RGB5x5_caller_t Gray2RGB5x5_callers[2] =
        {
            Gray2RGB5x5_caller<5>, Gray2RGB5x5_caller<6>
        };

        Gray2RGB5x5_callers[green_bits - 5](src, dst, stream);
    }

///////////////////////////////// Color to Grayscale ////////////////////////////////

    #undef R2Y
    #undef G2Y
    #undef B2Y

    enum
    {
        yuv_shift  = 14,
        xyz_shift  = 12,
        R2Y        = 4899,
        G2Y        = 9617,
        B2Y        = 1868,
        BLOCK_SIZE = 256
    };

    template <int GREEN_BITS> struct RGB5x52GrayConverter;
    template<> struct RGB5x52GrayConverter<6>
    {
        static __device__ __forceinline__ uchar cvt(uint t)
        {
            return (uchar)CV_DESCALE(((t << 3) & 0xf8) * B2Y + ((t >> 3) & 0xfc) * G2Y + ((t >> 8) & 0xf8) * R2Y, yuv_shift);
        }
    };
    template<> struct RGB5x52GrayConverter<5>
    {
        static __device__ __forceinline__ uchar cvt(uint t)
        {
            return (uchar)CV_DESCALE(((t << 3) & 0xf8) * B2Y + ((t >> 2) & 0xf8) * G2Y + ((t >> 7) & 0xf8) * R2Y, yuv_shift);
        }
    };

    template<int GREEN_BITS> struct RGB5x52Gray
    {
        typedef ushort src_t;
        typedef uchar dst_t;

        __device__ __forceinline__ uchar operator()(ushort src) const
        {
            return RGB5x52GrayConverter<GREEN_BITS>::cvt((uint)src);
        }
    };

    template <typename T>
    __device__ __forceinline__ T RGB2GrayConvert(const T* src, int bidx)
    {
        return (T)CV_DESCALE((unsigned)(src[bidx] * B2Y + src[1] * G2Y + src[bidx^2] * R2Y), yuv_shift);
    }
     __device__ __forceinline__ float RGB2GrayConvert(const float* src, int bidx)
    {
        const float cr = 0.299f;
        const float cg = 0.587f;
        const float cb = 0.114f;

        return src[bidx] * cb + src[1] * cg + src[bidx^2] * cr;
    }

    template <int SRCCN, typename T> struct RGB2Gray
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef T dst_t;

        explicit RGB2Gray(int bidx) : bidx(bidx) {}

        __device__ __forceinline__ T operator()(const src_t& src)
        {
            return RGB2GrayConvert(&src.x, bidx);
        }

    private:
        int bidx;
    };

    template <typename T, int SRCCN>
    void RGB2Gray_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        RGB2Gray<SRCCN, T> cvt(bidx);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2Gray_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2Gray_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        RGB2Gray_caller_t RGB2Gray_callers[] = {RGB2Gray_caller<uchar, 3>, RGB2Gray_caller<uchar, 4>};

        RGB2Gray_callers[srccn - 3](src, dst, bidx, stream);
    }

    void RGB2Gray_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2Gray_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        RGB2Gray_caller_t RGB2Gray_callers[] = {RGB2Gray_caller<ushort, 3>, RGB2Gray_caller<ushort, 4>};

        RGB2Gray_callers[srccn - 3](src, dst, bidx, stream);
    }

    void RGB2Gray_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int bidx, hipStream_t stream)
    {
        typedef void (*RGB2Gray_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, hipStream_t stream);
        RGB2Gray_caller_t RGB2Gray_callers[] = {RGB2Gray_caller<float, 3>, RGB2Gray_caller<float, 4>};

        RGB2Gray_callers[srccn - 3](src, dst, bidx, stream);
    }

    template <int GREEN_BITS>
    void RGB5x52Gray_caller(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream)
    {
        RGB5x52Gray<GREEN_BITS> cvt;
        callConvert(src, dst, cvt, stream);
    }

    void RGB5x52Gray_gpu(const DevMem2D& src, int green_bits, const DevMem2D& dst, hipStream_t stream)
    {
        typedef void (*RGB5x52Gray_caller_t)(const DevMem2D& src, const DevMem2D& dst, hipStream_t stream);
        static const RGB5x52Gray_caller_t RGB5x52Gray_callers[2] =
        {
            RGB5x52Gray_caller<5>, RGB5x52Gray_caller<6>
        };

        RGB5x52Gray_callers[green_bits - 5](src, dst, stream);
    }

///////////////////////////////////// RGB <-> YCrCb //////////////////////////////////////

    __constant__ int cYCrCbCoeffs_i[5];
    __constant__ float cYCrCbCoeffs_f[5];

    template <typename T, typename D>
    __device__ __forceinline__ void RGB2YCrCbConvert(const T* src, D& dst, int bidx)
    {
        const int delta = ColorChannel<T>::half() * (1 << yuv_shift);

        const int Y = CV_DESCALE(src[0] * cYCrCbCoeffs_i[0] + src[1] * cYCrCbCoeffs_i[1] + src[2] * cYCrCbCoeffs_i[2], yuv_shift);
        const int Cr = CV_DESCALE((src[bidx^2] - Y) * cYCrCbCoeffs_i[3] + delta, yuv_shift);
        const int Cb = CV_DESCALE((src[bidx] - Y) * cYCrCbCoeffs_i[4] + delta, yuv_shift);

        dst.x = saturate_cast<T>(Y);
        dst.y = saturate_cast<T>(Cr);
        dst.z = saturate_cast<T>(Cb);
    }
    template <typename D>
    static __device__ __forceinline__ void RGB2YCrCbConvert(const float* src, D& dst, int bidx)
    {
        dst.x = src[0] * cYCrCbCoeffs_f[0] + src[1] * cYCrCbCoeffs_f[1] + src[2] * cYCrCbCoeffs_f[2];
        dst.y = (src[bidx^2] - dst.x) * cYCrCbCoeffs_f[3] + ColorChannel<float>::half();
        dst.z = (src[bidx] - dst.x) * cYCrCbCoeffs_f[4] + ColorChannel<float>::half();
    }

    template<typename T> struct RGB2YCrCbBase
    {
        typedef int coeff_t;

        explicit RGB2YCrCbBase(const coeff_t coeffs[5])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cYCrCbCoeffs_i), coeffs, 5 * sizeof(int)) );
        }
    };
    template<> struct RGB2YCrCbBase<float>
    {
        typedef float coeff_t;

        explicit RGB2YCrCbBase(const coeff_t coeffs[5])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cYCrCbCoeffs_f), coeffs, 5 * sizeof(float)) );
        }
    };
    template <int SRCCN, int DSTCN, typename T> struct RGB2YCrCb : RGB2YCrCbBase<T>
    {
        typedef typename RGB2YCrCbBase<T>::coeff_t coeff_t;
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        RGB2YCrCb(int bidx, const coeff_t coeffs[5]) : RGB2YCrCbBase<T>(coeffs), bidx(bidx) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            RGB2YCrCbConvert(&src.x, dst, bidx);
            return dst;
        }

    private:
        int bidx;
    };

    template <typename T, typename D>
    __device__ __forceinline__ void YCrCb2RGBConvert(const T& src, D* dst, int bidx)
    {
        const int b = src.x + CV_DESCALE((src.z - ColorChannel<D>::half()) * cYCrCbCoeffs_i[3], yuv_shift);
        const int g = src.x + CV_DESCALE((src.z - ColorChannel<D>::half()) * cYCrCbCoeffs_i[2] + (src.y - ColorChannel<D>::half()) * cYCrCbCoeffs_i[1], yuv_shift);
        const int r = src.x + CV_DESCALE((src.y - ColorChannel<D>::half()) * cYCrCbCoeffs_i[0], yuv_shift);

        dst[bidx] = saturate_cast<D>(b);
        dst[1] = saturate_cast<D>(g);
        dst[bidx^2] = saturate_cast<D>(r);
    }
    template <typename T>
    __device__ __forceinline__ void YCrCb2RGBConvert(const T& src, float* dst, int bidx)
    {
        dst[bidx] = src.x + (src.z - ColorChannel<float>::half()) * cYCrCbCoeffs_f[3];
        dst[1] = src.x + (src.z - ColorChannel<float>::half()) * cYCrCbCoeffs_f[2] + (src.y - ColorChannel<float>::half()) * cYCrCbCoeffs_f[1];
        dst[bidx^2] = src.x + (src.y - ColorChannel<float>::half()) * cYCrCbCoeffs_f[0];
    }

    template<typename T> struct YCrCb2RGBBase
    {
        typedef int coeff_t;

        explicit YCrCb2RGBBase(const coeff_t coeffs[4])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cYCrCbCoeffs_i), coeffs, 4 * sizeof(int)) );
        }
    };
    template<> struct YCrCb2RGBBase<float>
    {
        typedef float coeff_t;

        explicit YCrCb2RGBBase(const coeff_t coeffs[4])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cYCrCbCoeffs_f), coeffs, 4 * sizeof(float)) );
        }
    };
    template <int SRCCN, int DSTCN, typename T> struct YCrCb2RGB : YCrCb2RGBBase<T>
    {
        typedef typename YCrCb2RGBBase<T>::coeff_t coeff_t;
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        YCrCb2RGB(int bidx, const coeff_t coeffs[4]) : YCrCb2RGBBase<T>(coeffs), bidx(bidx) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;

            YCrCb2RGBConvert(src, &dst.x, bidx);
            setAlpha(dst, ColorChannel<T>::max());

            return dst;
        }

    private:
        int bidx;
    };

    template <typename T, int SRCCN, int DSTCN>
    void RGB2YCrCb_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef typename RGB2YCrCb<SRCCN, DSTCN, T>::coeff_t coeff_t;
        RGB2YCrCb<SRCCN, DSTCN, T> cvt(bidx, (const coeff_t*)coeffs);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2YCrCb_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2YCrCb_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const RGB2YCrCb_caller_t RGB2YCrCb_callers[2][2] =
        {
            {RGB2YCrCb_caller<uchar, 3, 3>, RGB2YCrCb_caller<uchar, 3, 4>},
            {RGB2YCrCb_caller<uchar, 4, 3>, RGB2YCrCb_caller<uchar, 4, 4>}
        };

        RGB2YCrCb_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

    void RGB2YCrCb_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2YCrCb_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const RGB2YCrCb_caller_t RGB2YCrCb_callers[2][2] =
        {
            {RGB2YCrCb_caller<ushort, 3, 3>, RGB2YCrCb_caller<ushort, 3, 4>},
            {RGB2YCrCb_caller<ushort, 4, 3>, RGB2YCrCb_caller<ushort, 4, 4>}
        };

        RGB2YCrCb_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

    void RGB2YCrCb_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2YCrCb_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const RGB2YCrCb_caller_t RGB2YCrCb_callers[2][2] =
        {
            {RGB2YCrCb_caller<float, 3, 3>, RGB2YCrCb_caller<float, 3, 4>},
            {RGB2YCrCb_caller<float, 4, 3>, RGB2YCrCb_caller<float, 4, 4>}
        };

        RGB2YCrCb_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

    template <typename T, int SRCCN, int DSTCN>
    void YCrCb2RGB_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef typename YCrCb2RGB<SRCCN, DSTCN, T>::coeff_t coeff_t;
        YCrCb2RGB<SRCCN, DSTCN, T> cvt(bidx, (const coeff_t*)coeffs);
        callConvert(src, dst, cvt, stream);
    }

    void YCrCb2RGB_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*YCrCb2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const YCrCb2RGB_caller_t YCrCb2RGB_callers[2][2] =
        {
            {YCrCb2RGB_caller<uchar, 3, 3>, YCrCb2RGB_caller<uchar, 3, 4>},
            {YCrCb2RGB_caller<uchar, 4, 3>, YCrCb2RGB_caller<uchar, 4, 4>}
        };

        YCrCb2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

    void YCrCb2RGB_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*YCrCb2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const YCrCb2RGB_caller_t YCrCb2RGB_callers[2][2] =
        {
            {YCrCb2RGB_caller<ushort, 3, 3>, YCrCb2RGB_caller<ushort, 3, 4>},
            {YCrCb2RGB_caller<ushort, 4, 3>, YCrCb2RGB_caller<ushort, 4, 4>}
        };

        YCrCb2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

    void YCrCb2RGB_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, const void* coeffs, hipStream_t stream)
    {
        typedef void (*YCrCb2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, const void* coeffs, hipStream_t stream);
        static const YCrCb2RGB_caller_t YCrCb2RGB_callers[2][2] =
        {
            {YCrCb2RGB_caller<float, 3, 3>, YCrCb2RGB_caller<float, 3, 4>},
            {YCrCb2RGB_caller<float, 4, 3>, YCrCb2RGB_caller<float, 4, 4>}
        };

        YCrCb2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, coeffs, stream);
    }

////////////////////////////////////// RGB <-> XYZ ///////////////////////////////////////

    __constant__ int cXYZ_D65i[9];
    __constant__ float cXYZ_D65f[9];

    template <typename T, typename D>
    __device__ __forceinline__ void RGB2XYZConvert(const T* src, D& dst)
    {
        dst.x = saturate_cast<T>(CV_DESCALE(src[0] * cXYZ_D65i[0] + src[1] * cXYZ_D65i[1] + src[2] * cXYZ_D65i[2], xyz_shift));
        dst.y = saturate_cast<T>(CV_DESCALE(src[0] * cXYZ_D65i[3] + src[1] * cXYZ_D65i[4] + src[2] * cXYZ_D65i[5], xyz_shift));
        dst.z = saturate_cast<T>(CV_DESCALE(src[0] * cXYZ_D65i[6] + src[1] * cXYZ_D65i[7] + src[2] * cXYZ_D65i[8], xyz_shift));
    }
    template <typename D>
    __device__ __forceinline__ void RGB2XYZConvert(const float* src, D& dst)
    {
        dst.x = src[0] * cXYZ_D65f[0] + src[1] * cXYZ_D65f[1] + src[2] * cXYZ_D65f[2];
        dst.y = src[0] * cXYZ_D65f[3] + src[1] * cXYZ_D65f[4] + src[2] * cXYZ_D65f[5];
        dst.z = src[0] * cXYZ_D65f[6] + src[1] * cXYZ_D65f[7] + src[2] * cXYZ_D65f[8];
    }

    template <typename T> struct RGB2XYZBase
    {
        typedef int coeff_t;

        explicit RGB2XYZBase(const coeff_t coeffs[9])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cXYZ_D65i), coeffs, 9 * sizeof(int)) );
        }
    };
    template <> struct RGB2XYZBase<float>
    {
        typedef float coeff_t;

        explicit RGB2XYZBase(const coeff_t coeffs[9])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cXYZ_D65f), coeffs, 9 * sizeof(float)) );
        }
    };
    template <int SRCCN, int DSTCN, typename T> struct RGB2XYZ : RGB2XYZBase<T>
    {
        typedef typename RGB2XYZBase<T>::coeff_t coeff_t;
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        explicit RGB2XYZ(const coeff_t coeffs[9]) : RGB2XYZBase<T>(coeffs) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            RGB2XYZConvert(&src.x, dst);
            return dst;
        }
    };

    template <typename T, typename D>
    __device__ __forceinline__ void XYZ2RGBConvert(const T& src, D* dst)
    {
        dst[0] = saturate_cast<D>(CV_DESCALE(src.x * cXYZ_D65i[0] + src.y * cXYZ_D65i[1] + src.z * cXYZ_D65i[2], xyz_shift));
            dst[1] = saturate_cast<D>(CV_DESCALE(src.x * cXYZ_D65i[3] + src.y * cXYZ_D65i[4] + src.z * cXYZ_D65i[5], xyz_shift));
            dst[2] = saturate_cast<D>(CV_DESCALE(src.x * cXYZ_D65i[6] + src.y * cXYZ_D65i[7] + src.z * cXYZ_D65i[8], xyz_shift));
    }
    template <typename T>
    __device__ __forceinline__ void XYZ2RGBConvert(const T& src, float* dst)
    {
        dst[0] = src.x * cXYZ_D65f[0] + src.y * cXYZ_D65f[1] + src.z * cXYZ_D65f[2];
            dst[1] = src.x * cXYZ_D65f[3] + src.y * cXYZ_D65f[4] + src.z * cXYZ_D65f[5];
            dst[2] = src.x * cXYZ_D65f[6] + src.y * cXYZ_D65f[7] + src.z * cXYZ_D65f[8];
    }

    template <typename T> struct XYZ2RGBBase
    {
        typedef int coeff_t;

        explicit XYZ2RGBBase(const coeff_t coeffs[9])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cXYZ_D65i), coeffs, 9 * sizeof(int)) );
        }
    };
    template <> struct XYZ2RGBBase<float>
    {
        typedef float coeff_t;

        explicit XYZ2RGBBase(const coeff_t coeffs[9])
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cXYZ_D65f), coeffs, 9 * sizeof(float)) );
        }
    };
    template <int SRCCN, int DSTCN, typename T> struct XYZ2RGB : XYZ2RGBBase<T>
    {
        typedef typename RGB2XYZBase<T>::coeff_t coeff_t;
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        explicit XYZ2RGB(const coeff_t coeffs[9]) : XYZ2RGBBase<T>(coeffs) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            XYZ2RGBConvert(src, &dst.x);
            setAlpha(dst, ColorChannel<T>::max());
            return dst;
        }
    };

    template <typename T, int SRCCN, int DSTCN>
    void RGB2XYZ_caller(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream)
    {
        typedef typename RGB2XYZ<SRCCN, DSTCN, T>::coeff_t coeff_t;
        RGB2XYZ<SRCCN, DSTCN, T> cvt((const coeff_t*)coeffs);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2XYZ_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2XYZ_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const RGB2XYZ_caller_t RGB2XYZ_callers[2][2] =
        {
            {RGB2XYZ_caller<uchar, 3, 3>, RGB2XYZ_caller<uchar, 3, 4>},
            {RGB2XYZ_caller<uchar, 4, 3>, RGB2XYZ_caller<uchar, 4, 4>}
        };

        RGB2XYZ_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

    void RGB2XYZ_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2XYZ_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const RGB2XYZ_caller_t RGB2XYZ_callers[2][2] =
        {
            {RGB2XYZ_caller<ushort, 3, 3>, RGB2XYZ_caller<ushort, 3, 4>},
            {RGB2XYZ_caller<ushort, 4, 3>, RGB2XYZ_caller<ushort, 4, 4>}
        };

        RGB2XYZ_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

    void RGB2XYZ_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*RGB2XYZ_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const RGB2XYZ_caller_t RGB2XYZ_callers[2][2] =
        {
            {RGB2XYZ_caller<float, 3, 3>, RGB2XYZ_caller<float, 3, 4>},
            {RGB2XYZ_caller<float, 4, 3>, RGB2XYZ_caller<float, 4, 4>}
        };

        RGB2XYZ_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

    template <typename T, int SRCCN, int DSTCN>
    void XYZ2RGB_caller(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream)
    {
        typedef typename XYZ2RGB<SRCCN, DSTCN, T>::coeff_t coeff_t;
        XYZ2RGB<SRCCN, DSTCN, T> cvt((const coeff_t*)coeffs);
        callConvert(src, dst, cvt, stream);
    }

    void XYZ2RGB_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*XYZ2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const XYZ2RGB_caller_t XYZ2RGB_callers[2][2] =
        {
            {XYZ2RGB_caller<uchar, 3, 3>, XYZ2RGB_caller<uchar, 3, 4>},
            {XYZ2RGB_caller<uchar, 4, 3>, XYZ2RGB_caller<uchar, 4, 4>}
        };

        XYZ2RGB_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

    void XYZ2RGB_gpu_16u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*XYZ2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const XYZ2RGB_caller_t XYZ2RGB_callers[2][2] =
        {
            {XYZ2RGB_caller<ushort, 3, 3>, XYZ2RGB_caller<ushort, 3, 4>},
            {XYZ2RGB_caller<ushort, 4, 3>, XYZ2RGB_caller<ushort, 4, 4>}
        };

        XYZ2RGB_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

    void XYZ2RGB_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, const void* coeffs, hipStream_t stream)
    {
        typedef void (*XYZ2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, const void* coeffs, hipStream_t stream);
        static const XYZ2RGB_caller_t XYZ2RGB_callers[2][2] =
        {
            {XYZ2RGB_caller<float, 3, 3>, XYZ2RGB_caller<float, 3, 4>},
            {XYZ2RGB_caller<float, 4, 3>, XYZ2RGB_caller<float, 4, 4>}
        };

        XYZ2RGB_callers[srccn-3][dstcn-3](src, dst, coeffs, stream);
    }

////////////////////////////////////// RGB <-> HSV ///////////////////////////////////////

    __constant__ int cHsvDivTable   [256] = {0, 1044480, 522240, 348160, 261120, 208896, 174080, 149211, 130560, 116053, 104448, 94953, 87040, 80345, 74606, 69632, 65280, 61440, 58027, 54973, 52224, 49737, 47476, 45412, 43520, 41779, 40172, 38684, 37303, 36017, 34816, 33693, 32640, 31651, 30720, 29842, 29013, 28229, 27486, 26782, 26112, 25475, 24869, 24290, 23738, 23211, 22706, 22223, 21760, 21316, 20890, 20480, 20086, 19707, 19342, 18991, 18651, 18324, 18008, 17703, 17408, 17123, 16846, 16579, 16320, 16069, 15825, 15589, 15360, 15137, 14921, 14711, 14507, 14308, 14115, 13926, 13743, 13565, 13391, 13221, 13056, 12895, 12738, 12584, 12434, 12288, 12145, 12006, 11869, 11736, 11605, 11478, 11353, 11231, 11111, 10995, 10880, 10768, 10658, 10550, 10445, 10341, 10240, 10141, 10043, 9947, 9854, 9761, 9671, 9582, 9495, 9410, 9326, 9243, 9162, 9082, 9004, 8927, 8852, 8777, 8704, 8632, 8561, 8492, 8423, 8356, 8290, 8224, 8160, 8097, 8034, 7973, 7913, 7853, 7795, 7737, 7680, 7624, 7569, 7514, 7461, 7408, 7355, 7304, 7253, 7203, 7154, 7105, 7057, 7010, 6963, 6917, 6872, 6827, 6782, 6739, 6695, 6653, 6611, 6569, 6528, 6487, 6447, 6408, 6369, 6330, 6292, 6254, 6217, 6180, 6144, 6108, 6073, 6037, 6003, 5968, 5935, 5901, 5868, 5835, 5803, 5771, 5739, 5708, 5677, 5646, 5615, 5585, 5556, 5526, 5497, 5468, 5440, 5412, 5384, 5356, 5329, 5302, 5275, 5249, 5222, 5196, 5171, 5145, 5120, 5095, 5070, 5046, 5022, 4998, 4974, 4950, 4927, 4904, 4881, 4858, 4836, 4813, 4791, 4769, 4748, 4726, 4705, 4684, 4663, 4642, 4622, 4601, 4581, 4561, 4541, 4522, 4502, 4483, 4464, 4445, 4426, 4407, 4389, 4370, 4352, 4334, 4316, 4298, 4281, 4263, 4246, 4229, 4212, 4195, 4178, 4161, 4145, 4128, 4112, 4096};
    __constant__ int cHsvDivTable180[256] = {0, 122880, 61440, 40960, 30720, 24576, 20480, 17554, 15360, 13653, 12288, 11171, 10240, 9452, 8777, 8192, 7680, 7228, 6827, 6467, 6144, 5851, 5585, 5343, 5120, 4915, 4726, 4551, 4389, 4237, 4096, 3964, 3840, 3724, 3614, 3511, 3413, 3321, 3234, 3151, 3072, 2997, 2926, 2858, 2793, 2731, 2671, 2614, 2560, 2508, 2458, 2409, 2363, 2318, 2276, 2234, 2194, 2156, 2119, 2083, 2048, 2014, 1982, 1950, 1920, 1890, 1862, 1834, 1807, 1781, 1755, 1731, 1707, 1683, 1661, 1638, 1617, 1596, 1575, 1555, 1536, 1517, 1499, 1480, 1463, 1446, 1429, 1412, 1396, 1381, 1365, 1350, 1336, 1321, 1307, 1293, 1280, 1267, 1254, 1241, 1229, 1217, 1205, 1193, 1182, 1170, 1159, 1148, 1138, 1127, 1117, 1107, 1097, 1087, 1078, 1069, 1059, 1050, 1041, 1033, 1024, 1016, 1007, 999, 991, 983, 975, 968, 960, 953, 945, 938, 931, 924, 917, 910, 904, 897, 890, 884, 878, 871, 865, 859, 853, 847, 842, 836, 830, 825, 819, 814, 808, 803, 798, 793, 788, 783, 778, 773, 768, 763, 759, 754, 749, 745, 740, 736, 731, 727, 723, 719, 714, 710, 706, 702, 698, 694, 690, 686, 683, 679, 675, 671, 668, 664, 661, 657, 654, 650, 647, 643, 640, 637, 633, 630, 627, 624, 621, 617, 614, 611, 608, 605, 602, 599, 597, 594, 591, 588, 585, 582, 580, 577, 574, 572, 569, 566, 564, 561, 559, 556, 554, 551, 549, 546, 544, 541, 539, 537, 534, 532, 530, 527, 525, 523, 521, 518, 516, 514, 512, 510, 508, 506, 504, 502, 500, 497, 495, 493, 492, 490, 488, 486, 484, 482};
    __constant__ int cHsvDivTable256[256] = {0, 174763, 87381, 58254, 43691, 34953, 29127, 24966, 21845, 19418, 17476, 15888, 14564, 13443, 12483, 11651, 10923, 10280, 9709, 9198, 8738, 8322, 7944, 7598, 7282, 6991, 6722, 6473, 6242, 6026, 5825, 5638, 5461, 5296, 5140, 4993, 4855, 4723, 4599, 4481, 4369, 4263, 4161, 4064, 3972, 3884, 3799, 3718, 3641, 3567, 3495, 3427, 3361, 3297, 3236, 3178, 3121, 3066, 3013, 2962, 2913, 2865, 2819, 2774, 2731, 2689, 2648, 2608, 2570, 2533, 2497, 2461, 2427, 2394, 2362, 2330, 2300, 2270, 2241, 2212, 2185, 2158, 2131, 2106, 2081, 2056, 2032, 2009, 1986, 1964, 1942, 1920, 1900, 1879, 1859, 1840, 1820, 1802, 1783, 1765, 1748, 1730, 1713, 1697, 1680, 1664, 1649, 1633, 1618, 1603, 1589, 1574, 1560, 1547, 1533, 1520, 1507, 1494, 1481, 1469, 1456, 1444, 1432, 1421, 1409, 1398, 1387, 1376, 1365, 1355, 1344, 1334, 1324, 1314, 1304, 1295, 1285, 1276, 1266, 1257, 1248, 1239, 1231, 1222, 1214, 1205, 1197, 1189, 1181, 1173, 1165, 1157, 1150, 1142, 1135, 1128, 1120, 1113, 1106, 1099, 1092, 1085, 1079, 1072, 1066, 1059, 1053, 1046, 1040, 1034, 1028, 1022, 1016, 1010, 1004, 999, 993, 987, 982, 976, 971, 966, 960, 955, 950, 945, 940, 935, 930, 925, 920, 915, 910, 906, 901, 896, 892, 887, 883, 878, 874, 869, 865, 861, 857, 853, 848, 844, 840, 836, 832, 828, 824, 820, 817, 813, 809, 805, 802, 798, 794, 791, 787, 784, 780, 777, 773, 770, 767, 763, 760, 757, 753, 750, 747, 744, 741, 737, 734, 731, 728, 725, 722, 719, 716, 713, 710, 708, 705, 702, 699, 696, 694, 691, 688, 685};

    template <typename D> __device__ void RGB2HSVConvert(const uchar* src, D& dst, int bidx, int hr)
    {
        const int hsv_shift = 12;
        const int* hdiv_table = hr == 180 ? cHsvDivTable180 : cHsvDivTable256;

        int b = src[bidx], g = src[1], r = src[bidx^2];
        int h, s, v = b;
        int vmin = b, diff;
        int vr, vg;

        v = max(v, g);
        v = max(v, r);
        vmin = min(vmin, g);
        vmin = min(vmin, r);

        diff = v - vmin;
        vr = v == r ? -1 : 0;
        vg = v == g ? -1 : 0;

        s = (diff * cHsvDivTable[v] + (1 << (hsv_shift-1))) >> hsv_shift;
        h = (vr & (g - b)) + (~vr & ((vg & (b - r + 2 * diff)) + ((~vg) & (r - g + 4 * diff))));
        h = (h * hdiv_table[diff] + (1 << (hsv_shift-1))) >> hsv_shift;
        h += h < 0 ? hr : 0;

        dst.x = saturate_cast<uchar>(h);
        dst.y = (uchar)s;
        dst.z = (uchar)v;
    }
    template<typename D> __device__ void RGB2HSVConvert(const float* src, D& dst, int bidx, int hr)
    {
        const float hscale = hr * (1.f / 360.f);

        float b = src[bidx], g = src[1], r = src[bidx^2];
        float h, s, v;

        float vmin, diff;

        v = vmin = r;
        v = fmax(v, g);
        v = fmax(v, b);
        vmin = fmin(vmin, g);
        vmin = fmin(vmin, b);

        diff = v - vmin;
        s = diff / (float)(fabs(v) + numeric_limits_gpu<float>::epsilon());
        diff = (float)(60. / (diff + numeric_limits_gpu<float>::epsilon()));

        if (v == r)
            h = (g - b) * diff;
        else if (v == g)
            h = (b - r) * diff + 120.f;
        else
            h = (r - g) * diff + 240.f;

        if (h < 0) h += 360.f;

        dst.x = h * hscale;
        dst.y = s;
        dst.z = v;
    }

    template <int SRCCN, int DSTCN, typename T> struct RGB2HSV
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        RGB2HSV(int bidx, int hr) : bidx(bidx), hr(hr) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            RGB2HSVConvert(&src.x, dst, bidx, hr);
            return dst;
        }

    private:
        int bidx;
        int hr;
    };

    __constant__ int cHsvSectorData[6][3] =
    {
        {1,3,0}, {1,0,2}, {3,0,1}, {0,2,1}, {0,1,3}, {2,1,0}
    };

    template <typename T> __device__ void HSV2RGBConvert(const T& src, float* dst, int bidx, int hr)
    {
        const float hscale = 6.f / hr;

        float h = src.x, s = src.y, v = src.z;
        float b, g, r;

        if( s == 0 )
            b = g = r = v;
        else
        {
            float tab[4];
            int sector;
            h *= hscale;
            if( h < 0 )
                do h += 6; while( h < 0 );
            else if( h >= 6 )
                do h -= 6; while( h >= 6 );
            sector = __float2int_rd(h);
            h -= sector;

            tab[0] = v;
            tab[1] = v*(1.f - s);
            tab[2] = v*(1.f - s*h);
            tab[3] = v*(1.f - s*(1.f - h));

            b = tab[cHsvSectorData[sector][0]];
            g = tab[cHsvSectorData[sector][1]];
            r = tab[cHsvSectorData[sector][2]];
        }

        dst[bidx] = b;
        dst[1] = g;
        dst[bidx^2] = r;
    }
    template <typename T> __device__ void HSV2RGBConvert(const T& src, uchar* dst, int bidx, int hr)
    {
        float3 buf;

        buf.x = src.x;
        buf.y = src.y * (1.f/255.f);
        buf.z = src.z * (1.f/255.f);

        HSV2RGBConvert(buf, &buf.x, bidx, hr);

        dst[0] = saturate_cast<uchar>(buf.x * 255.f);
        dst[1] = saturate_cast<uchar>(buf.y * 255.f);
        dst[2] = saturate_cast<uchar>(buf.z * 255.f);
    }

    template <int SRCCN, int DSTCN, typename T> struct HSV2RGB
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        HSV2RGB(int bidx, int hr) : bidx(bidx), hr(hr) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            HSV2RGBConvert(src, &dst.x, bidx, hr);
            setAlpha(dst, ColorChannel<T>::max());
            return dst;
        }

    private:
        int bidx;
        int hr;
    };

    template <typename T, int SRCCN, int DSTCN>
    void RGB2HSV_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream)
    {
        RGB2HSV<SRCCN, DSTCN, T> cvt(bidx, hrange);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2HSV_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*RGB2HSV_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const RGB2HSV_caller_t RGB2HSV_callers[2][2] =
        {
            {RGB2HSV_caller<uchar, 3, 3>, RGB2HSV_caller<uchar, 3, 4>},
            {RGB2HSV_caller<uchar, 4, 3>, RGB2HSV_caller<uchar, 4, 4>}
        };

        RGB2HSV_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

    void RGB2HSV_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*RGB2HSV_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const RGB2HSV_caller_t RGB2HSV_callers[2][2] =
        {
            {RGB2HSV_caller<float, 3, 3>, RGB2HSV_caller<float, 3, 4>},
            {RGB2HSV_caller<float, 4, 3>, RGB2HSV_caller<float, 4, 4>}
        };

        RGB2HSV_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

    template <typename T, int SRCCN, int DSTCN>
    void HSV2RGB_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream)
    {
        HSV2RGB<SRCCN, DSTCN, T> cvt(bidx, hrange);
        callConvert(src, dst, cvt, stream);
    }

    void HSV2RGB_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*HSV2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const HSV2RGB_caller_t HSV2RGB_callers[2][2] =
        {
            {HSV2RGB_caller<uchar, 3, 3>, HSV2RGB_caller<uchar, 3, 4>},
            {HSV2RGB_caller<uchar, 4, 3>, HSV2RGB_caller<uchar, 4, 4>}
        };

        HSV2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

    void HSV2RGB_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*HSV2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const HSV2RGB_caller_t HSV2RGB_callers[2][2] =
        {
            {HSV2RGB_caller<float, 3, 3>, HSV2RGB_caller<float, 3, 4>},
            {HSV2RGB_caller<float, 4, 3>, HSV2RGB_caller<float, 4, 4>}
        };

        HSV2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

/////////////////////////////////////// RGB <-> HLS ////////////////////////////////////////

    template <typename D> __device__ void RGB2HLSConvert(const float* src, D& dst, int bidx, int hr)
    {
        const float hscale = hr * (1.f / 360.f);

        float b = src[bidx], g = src[1], r = src[bidx^2];
        float h = 0.f, s = 0.f, l;
        float vmin, vmax, diff;

        vmax = vmin = r;
        vmax = fmax(vmax, g);
        vmax = fmax(vmax, b);
        vmin = fmin(vmin, g);
        vmin = fmin(vmin, b);

        diff = vmax - vmin;
        l = (vmax + vmin) * 0.5f;

        if (diff > numeric_limits_gpu<float>::epsilon())
        {
            s = l < 0.5f ? diff / (vmax + vmin) : diff / (2.0f - vmax - vmin);
            diff = 60.f / diff;

            if (vmax == r)
                h = (g - b)*diff;
            else if (vmax == g)
                h = (b - r)*diff + 120.f;
            else
                h = (r - g)*diff + 240.f;

            if (h < 0.f) h += 360.f;
        }

        dst.x = h * hscale;
        dst.y = l;
        dst.z = s;
    }
    template <typename D> __device__ void RGB2HLSConvert(const uchar* src, D& dst, int bidx, int hr)
    {
        float3 buf;

        buf.x = src[0]*(1.f/255.f);
        buf.y = src[1]*(1.f/255.f);
        buf.z = src[2]*(1.f/255.f);

        RGB2HLSConvert(&buf.x, buf, bidx, hr);

        dst.x = saturate_cast<uchar>(buf.x);
        dst.y = saturate_cast<uchar>(buf.y*255.f);
        dst.z = saturate_cast<uchar>(buf.z*255.f);
    }

    template <int SRCCN, int DSTCN, typename T> struct RGB2HLS
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        RGB2HLS(int bidx, int hr) : bidx(bidx), hr(hr) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            RGB2HLSConvert(&src.x, dst, bidx, hr);
            return dst;
        }

    private:
        int bidx;
        int hr;
    };

    __constant__ int cHlsSectorData[6][3] =
    {
        {1,3,0}, {1,0,2}, {3,0,1}, {0,2,1}, {0,1,3}, {2,1,0}
    };

    template <typename T> __device__ void HLS2RGBConvert(const T& src, float* dst, int bidx, int hr)
    {
        const float hscale = 6.0f / hr;

        float h = src.x, l = src.y, s = src.z;
        float b, g, r;

        if (s == 0)
            b = g = r = l;
        else
        {
            float tab[4];
            int sector;

            float p2 = l <= 0.5f ? l * (1 + s) : l + s - l * s;
            float p1 = 2 * l - p2;

            h *= hscale;

            if( h < 0 )
                do h += 6; while( h < 0 );
            else if( h >= 6 )
                do h -= 6; while( h >= 6 );

            sector = __float2int_rd(h);
            h -= sector;

            tab[0] = p2;
            tab[1] = p1;
            tab[2] = p1 + (p2 - p1) * (1 - h);
            tab[3] = p1 + (p2 - p1) * h;

            b = tab[cHlsSectorData[sector][0]];
            g = tab[cHlsSectorData[sector][1]];
            r = tab[cHlsSectorData[sector][2]];
        }

        dst[bidx] = b;
        dst[1] = g;
        dst[bidx^2] = r;
    }
    template <typename T> __device__ void HLS2RGBConvert(const T& src, uchar* dst, int bidx, int hr)
    {
        float3 buf;

        buf.x = src.x;
        buf.y = src.y*(1.f/255.f);
        buf.z = src.z*(1.f/255.f);

        HLS2RGBConvert(buf, &buf.x, bidx, hr);

        dst[0] = saturate_cast<uchar>(buf.x*255.f);
        dst[1] = saturate_cast<uchar>(buf.y*255.f);
        dst[2] = saturate_cast<uchar>(buf.z*255.f);
    }

    template <int SRCCN, int DSTCN, typename T> struct HLS2RGB
    {
        typedef typename TypeVec<T, SRCCN>::vec_t src_t;
        typedef typename TypeVec<T, DSTCN>::vec_t dst_t;

        HLS2RGB(int bidx, int hr) : bidx(bidx), hr(hr) {}

        __device__ __forceinline__ dst_t operator()(const src_t& src) const
        {
            dst_t dst;
            HLS2RGBConvert(src, &dst.x, bidx, hr);
            setAlpha(dst, ColorChannel<T>::max());
            return dst;
        }

    private:
        int bidx;
        int hr;
    };

    template <typename T, int SRCCN, int DSTCN>
    void RGB2HLS_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream)
    {
        RGB2HLS<SRCCN, DSTCN, T> cvt(bidx, hrange);
        callConvert(src, dst, cvt, stream);
    }

    void RGB2HLS_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*RGB2HLS_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const RGB2HLS_caller_t RGB2HLS_callers[2][2] =
        {
            {RGB2HLS_caller<uchar, 3, 3>, RGB2HLS_caller<uchar, 3, 4>},
            {RGB2HLS_caller<uchar, 4, 3>, RGB2HLS_caller<uchar, 4, 4>}
        };

        RGB2HLS_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

    void RGB2HLS_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*RGB2HLS_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const RGB2HLS_caller_t RGB2HLS_callers[2][2] =
        {
            {RGB2HLS_caller<float, 3, 3>, RGB2HLS_caller<float, 3, 4>},
            {RGB2HLS_caller<float, 4, 3>, RGB2HLS_caller<float, 4, 4>}
        };

        RGB2HLS_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }


    template <typename T, int SRCCN, int DSTCN>
    void HLS2RGB_caller(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream)
    {
        HLS2RGB<SRCCN, DSTCN, T> cvt(bidx, hrange);
        callConvert(src, dst, cvt, stream);
    }

    void HLS2RGB_gpu_8u(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*HLS2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const HLS2RGB_caller_t HLS2RGB_callers[2][2] =
        {
            {HLS2RGB_caller<uchar, 3, 3>, HLS2RGB_caller<uchar, 3, 4>},
            {HLS2RGB_caller<uchar, 4, 3>, HLS2RGB_caller<uchar, 4, 4>}
        };

        HLS2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }

    void HLS2RGB_gpu_32f(const DevMem2D& src, int srccn, const DevMem2D& dst, int dstcn, int bidx, int hrange, hipStream_t stream)
    {
        typedef void (*HLS2RGB_caller_t)(const DevMem2D& src, const DevMem2D& dst, int bidx, int hrange, hipStream_t stream);
        static const HLS2RGB_caller_t HLS2RGB_callers[2][2] =
        {
            {HLS2RGB_caller<float, 3, 3>, HLS2RGB_caller<float, 3, 4>},
            {HLS2RGB_caller<float, 4, 3>, HLS2RGB_caller<float, 4, 4>}
        };

        HLS2RGB_callers[srccn-3][dstcn-3](src, dst, bidx, hrange, stream);
    }
}}}
