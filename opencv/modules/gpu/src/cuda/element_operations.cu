#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/vecmath.hpp"
#include "opencv2/gpu/device/transform.hpp"
#include "opencv2/gpu/device/saturate_cast.hpp"
#include "internal_shared.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace mathfunc
{

    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ __forceinline__ uchar operator()(const T1& src1, const T2& src2)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, stream);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<uint, uint>(src1, src2, dst, stream);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream)
    {
        compare_ne<float, float>(src1, src2, dst, stream);
    }


    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    { 
        static __device__ __forceinline__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStep src, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStep src, PtrStep dst, 
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), 
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStep src, 
                                      const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStep src, 
                     const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst); 
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseNotCaller(int rows, int cols, int elem_size1, int cn, 
                          const PtrStep src, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, cols * elem_size1 * cn, src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStep src, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a | b; } 
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a & b; } 
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    { 
        static __device__ __forceinline__ T call(T a, T b) { return a ^ b; } 
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStep src1, 
                                       const PtrStep src2, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStep src1, const PtrStep src2, 
                      PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
            const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                        const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0) 
            cudaSafeCall( hipDeviceSynchronize() );
    }


    void bitwiseOrCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                         const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                             const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // min/max

    struct MinOp
    {        
        template <typename T>
        __device__ __forceinline__ T operator()(T a, T b)
        {
            return min(a, b);
        }
        __device__ __forceinline__ float operator()(float a, float b)
        {
            return fmin(a, b);
        }
        __device__ __forceinline__ double operator()(double a, double b)
        {
            return fmin(a, b);
        }
    };

    struct MaxOp
    {        
        template <typename T>
        __device__ __forceinline__ T operator()(T a, T b)
        {
            return max(a, b);
        }
        __device__ __forceinline__ float operator()(float a, float b)
        {
            return fmax(a, b);
        }
        __device__ __forceinline__ double operator()(double a, double b)
        {
            return fmax(a, b);
        }
    };
    
    template <typename T> struct ScalarMinOp
    {
        T s;

        explicit ScalarMinOp(T s_) : s(s_) {}

        __device__ __forceinline__ T operator()(T a)
        {
            return min(a, s);
        }
    };
    template <> struct ScalarMinOp<float>
    {
        float s;

        explicit ScalarMinOp(float s_) : s(s_) {}

        __device__ __forceinline__ float operator()(float a)
        {
            return fmin(a, s);
        }
    };
    template <> struct ScalarMinOp<double>
    {
        double s;

        explicit ScalarMinOp(double s_) : s(s_) {}

        __device__ __forceinline__ double operator()(double a)
        {
            return fmin(a, s);
        }
    };
    
    template <typename T> struct ScalarMaxOp
    {
        T s;

        explicit ScalarMaxOp(T s_) : s(s_) {}

        __device__ __forceinline__ T operator()(T a)
        {
            return max(a, s);
        }
    };
    template <> struct ScalarMaxOp<float>
    {
        float s;

        explicit ScalarMaxOp(float s_) : s(s_) {}

        __device__ __forceinline__ float operator()(float a)
        {
            return fmax(a, s);
        }
    };
    template <> struct ScalarMaxOp<double>
    {
        double s;

        explicit ScalarMaxOp(double s_) : s(s_) {}

        __device__ __forceinline__ double operator()(double a)
        {
            return fmax(a, s);
        }
    };
    
    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        MinOp op;
        transform(src1, src2, dst, op, stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        MaxOp op;
        transform(src1, src2, dst, op, stream);    
    }
    
    template void max_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, const DevMem2D_<schar>& src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        ScalarMinOp<T> op(src2);
        transform(src1, dst, op, stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);
    
    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, T src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        ScalarMaxOp<T> op(src2);
        transform(src1, dst, op, stream);    
    }

    template void max_gpu<uchar >(const DevMem2D& src1, uchar src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<schar >(const DevMem2D_<schar>& src1, schar src2, const DevMem2D_<schar>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, ushort src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, short src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, int src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, float src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);

    
    //////////////////////////////////////////////////////////////////////////
    // threshold

    template <typename T> struct ThreshBinary
    {
        ThreshBinary(T thresh_, T maxVal_) : thresh(thresh_), maxVal(maxVal_) {}

        __device__ __forceinline__ T operator()(const T& src) const
        {
            return src > thresh ? maxVal : 0;
        }

    private:
        T thresh;
        T maxVal;
    };

    template <typename T> struct ThreshBinaryInv
    {
        ThreshBinaryInv(T thresh_, T maxVal_) : thresh(thresh_), maxVal(maxVal_) {}

        __device__ __forceinline__ T operator()(const T& src) const
        {
            return src > thresh ? 0 : maxVal;
        }

    private:
        T thresh;
        T maxVal;
    };

    template <typename T> struct ThreshTrunc
    {
        ThreshTrunc(T thresh_, T) : thresh(thresh_) {}

        __device__ __forceinline__ T operator()(const T& src) const
        {
            return min(src, thresh);
        }

    private:
        T thresh;
    };
    template <> struct  ThreshTrunc<float>
    {
        ThreshTrunc(float thresh_, float) : thresh(thresh_) {}

        __device__ __forceinline__ float operator()(const float& src) const
        {
            return fmin(src, thresh);
        }

    private:
        float thresh;
    };
    template <> struct  ThreshTrunc<double>
    {
        ThreshTrunc(double thresh_, double) : thresh(thresh_) {}

        __device__ __forceinline__ double operator()(const double& src) const
        {
            return fmin(src, thresh);
        }

    private:
        double thresh;
    };

    template <typename T> struct ThreshToZero
    {
    public:
        ThreshToZero(T thresh_, T) : thresh(thresh_) {}

        __device__ __forceinline__ T operator()(const T& src) const
        {
            return src > thresh ? src : 0;
        }

    private:
        T thresh;
    };

    template <typename T> struct ThreshToZeroInv
    {
    public:
        ThreshToZeroInv(T thresh_, T) : thresh(thresh_) {}

        __device__ __forceinline__ T operator()(const T& src) const
        {
            return src > thresh ? 0 : src;
        }

    private:
        T thresh;
    };

    template <template <typename> class Op, typename T>
    void threshold_caller(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
        hipStream_t stream)
    {
        Op<T> op(thresh, maxVal);
        transform(src, dst, op, stream);
    }

    template <typename T>
    void threshold_gpu(const DevMem2D& src, const DevMem2D& dst, T thresh, T maxVal, int type,
        hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2D_<T>& src, const DevMem2D_<T>& dst, T thresh, T maxVal, 
            hipStream_t stream);

        static const caller_t callers[] = 
        {
            threshold_caller<ThreshBinary, T>, 
            threshold_caller<ThreshBinaryInv, T>, 
            threshold_caller<ThreshTrunc, T>, 
            threshold_caller<ThreshToZero, T>, 
            threshold_caller<ThreshToZeroInv, T>
        };

        callers[type]((DevMem2D_<T>)src, (DevMem2D_<T>)dst, thresh, maxVal, stream);
    }

    template void threshold_gpu<uchar>(const DevMem2D& src, const DevMem2D& dst, uchar thresh, uchar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<schar>(const DevMem2D& src, const DevMem2D& dst, schar thresh, schar maxVal, int type, hipStream_t stream);
    template void threshold_gpu<ushort>(const DevMem2D& src, const DevMem2D& dst, ushort thresh, ushort maxVal, int type, hipStream_t stream);
    template void threshold_gpu<short>(const DevMem2D& src, const DevMem2D& dst, short thresh, short maxVal, int type, hipStream_t stream);
    template void threshold_gpu<int>(const DevMem2D& src, const DevMem2D& dst, int thresh, int maxVal, int type, hipStream_t stream);
    template void threshold_gpu<float>(const DevMem2D& src, const DevMem2D& dst, float thresh, float maxVal, int type, hipStream_t stream);
    template void threshold_gpu<double>(const DevMem2D& src, const DevMem2D& dst, double thresh, double maxVal, int type, hipStream_t stream);
}}}
