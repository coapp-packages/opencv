#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/device/vecmath.hpp"
#include "transform.hpp"
#include "internal_shared.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace mathfunc
{

    //////////////////////////////////////////////////////////////////////////////////////
    // Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ uchar operator()(const T1& src1, const T2& src2)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, 0);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<uint, uint>(src1, src2, dst);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<float, float>(src1, src2, dst);
    }


    //////////////////////////////////////////////////////////////////////////
    // Unary bitwise logical matrix operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp;

    template <typename T>
    struct UnOp<T, UN_OP_NOT>
    { 
        static __device__ T call(T v) { return ~v; }
    };


    template <int opid>
    __global__ void bitwiseUnOpKernel(int rows, int width, const PtrStep src, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src_ptr = src.ptr(y) + x;
            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = UnOp<uint, opid>::call(*(uint*)src_ptr);
            }
            else
            {
                const uchar* src_end = src.ptr(y) + width;
                while (src_ptr < src_end)
                {
                    *dst_ptr++ = UnOp<uchar, opid>::call(*src_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseUnOp(int rows, int width, const PtrStep src, PtrStep dst, 
                     hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), 
                  divUp(rows, threads.y));

        bitwiseUnOpKernel<opid><<<grid, threads>>>(rows, width, src, dst);

        if (stream == 0) 
            cudaSafeCall(hipDeviceSynchronize());
    }


    template <typename T, int opid>
    __global__ void bitwiseUnOpKernel(int rows, int cols, int cn, const PtrStep src, 
                                      const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src_row = (const T*)src.ptr(y);

            dst_row[x] = UnOp<T, opid>::call(src_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseUnOp(int rows, int cols, int cn, const PtrStep src, 
                     const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseUnOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src, mask, dst); 

        if (stream == 0) 
            cudaSafeCall(hipDeviceSynchronize());
    }


    void bitwiseNotCaller(int rows, int cols, int elem_size1, int cn, 
                          const PtrStep src, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<UN_OP_NOT>(rows, cols * elem_size1 * cn, src, dst, stream);
    }


    template <typename T>
    void bitwiseMaskNotCaller(int rows, int cols, int cn, const PtrStep src, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseUnOp<T, UN_OP_NOT>(rows, cols * cn, cn, src, mask, dst, stream);
    }

    template void bitwiseMaskNotCaller<uchar>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<ushort>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskNotCaller<uint>(int, int, int, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // Binary bitwise logical matrix operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp;

    template <typename T>
    struct BinOp<T, BIN_OP_OR>
    { 
        static __device__ T call(T a, T b) { return a | b; } 
    };


    template <typename T>
    struct BinOp<T, BIN_OP_AND>
    { 
        static __device__ T call(T a, T b) { return a & b; } 
    };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>
    { 
        static __device__ T call(T a, T b) { return a ^ b; } 
    };


    template <int opid>
    __global__ void bitwiseBinOpKernel(int rows, int width, const PtrStep src1, 
                                       const PtrStep src2, PtrStep dst)
    {
        const int x = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (y < rows) 
        {
            uchar* dst_ptr = dst.ptr(y) + x;
            const uchar* src1_ptr = src1.ptr(y) + x;
            const uchar* src2_ptr = src2.ptr(y) + x;

            if (x + sizeof(uint) - 1 < width)
            {
                *(uint*)dst_ptr = BinOp<uint, opid>::call(*(uint*)src1_ptr, *(uint*)src2_ptr);
            }
            else
            {
                const uchar* src1_end = src1.ptr(y) + width;
                while (src1_ptr < src1_end)
                {
                    *dst_ptr++ = BinOp<uchar, opid>::call(*src1_ptr++, *src2_ptr++);
                }
            }
        }
    }


    template <int opid>
    void bitwiseBinOp(int rows, int width, const PtrStep src1, const PtrStep src2, 
                      PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(width, threads.x * sizeof(uint)), divUp(rows, threads.y));

        bitwiseBinOpKernel<opid><<<grid, threads>>>(rows, width, src1, src2, dst);

        if (stream == 0) 
            cudaSafeCall(hipDeviceSynchronize());
    }


    template <typename T, int opid>
    __global__ void bitwiseBinOpKernel(
            int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
            const PtrStep mask, PtrStep dst)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask.ptr(y)[x / cn]) 
        {
            T* dst_row = (T*)dst.ptr(y);
            const T* src1_row = (const T*)src1.ptr(y);
            const T* src2_row = (const T*)src2.ptr(y);

            dst_row[x] = BinOp<T, opid>::call(src1_row[x], src2_row[x]);
        }
    }


    template <typename T, int opid>
    void bitwiseBinOp(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                        const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        bitwiseBinOpKernel<T, opid><<<grid, threads>>>(rows, cols, cn, src1, src2, mask, dst); 

        if (stream == 0) 
            cudaSafeCall(hipDeviceSynchronize());
    }


    void bitwiseOrCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                         const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_OR>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskOrCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                             const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_OR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskOrCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskOrCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseAndCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_AND>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskAndCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_AND>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskAndCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskAndCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    void bitwiseXorCaller(int rows, int cols, int elem_size1, int cn, const PtrStep src1, 
                          const PtrStep src2, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<BIN_OP_XOR>(rows, cols * elem_size1 * cn, src1, src2, dst, stream);
    }


    template <typename T>
    void bitwiseMaskXorCaller(int rows, int cols, int cn, const PtrStep src1, const PtrStep src2, 
                              const PtrStep mask, PtrStep dst, hipStream_t stream)
    {
        bitwiseBinOp<T, BIN_OP_XOR>(rows, cols * cn, cn, src1, src2, mask, dst, stream);
    }

    template void bitwiseMaskXorCaller<uchar>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<ushort>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);
    template void bitwiseMaskXorCaller<uint>(int, int, int, const PtrStep, const PtrStep, const PtrStep, PtrStep, hipStream_t);


    //////////////////////////////////////////////////////////////////////////
    // min/max

    struct MinOp
    {        
        template <typename T>
        __device__ T operator()(T a, T b)
        {
            return min(a, b);
        }
        __device__ float operator()(float a, float b)
        {
            return fmin(a, b);
        }
        __device__ double operator()(double a, double b)
        {
            return fmin(a, b);
        }
    };

    struct MaxOp
    {        
        template <typename T>
        __device__ T operator()(T a, T b)
        {
            return max(a, b);
        }
        __device__ float operator()(float a, float b)
        {
            return fmax(a, b);
        }
        __device__ double operator()(double a, double b)
        {
            return fmax(a, b);
        }
    };
    
    struct ScalarMinOp
    {
        double s;

        explicit ScalarMinOp(double s_) : s(s_) {}

        template <typename T>
        __device__ T operator()(T a)
        {
            return saturate_cast<T>(fmin((double)a, s));
        }
    };
    
    struct ScalarMaxOp
    {
        double s;

        explicit ScalarMaxOp(double s_) : s(s_) {}

        template <typename T>
        __device__ T operator()(T a)
        {
            return saturate_cast<T>(fmax((double)a, s));
        }
    };
    
    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        MinOp op;
        transform(src1, src2, dst, op, stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<char  >(const DevMem2D_<char>& src1, const DevMem2D_<char>& src2, const DevMem2D_<char>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, const DevMem2D_<T>& src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        MaxOp op;
        transform(src1, src2, dst, op, stream);    
    }
    
    template void max_gpu<uchar >(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<char  >(const DevMem2D_<char>& src1, const DevMem2D_<char>& src2, const DevMem2D_<char>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, const DevMem2D_<ushort>& src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, const DevMem2D_<short>& src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, const DevMem2D_<int>& src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, const DevMem2D_<float>& src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, const DevMem2D_<double>& src2, const DevMem2D_<double>& dst, hipStream_t stream);

    template <typename T>
    void min_gpu(const DevMem2D_<T>& src1, double src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        ScalarMinOp op(src2);
        transform(src1, dst, op, stream);    
    }

    template void min_gpu<uchar >(const DevMem2D& src1, double src2, const DevMem2D& dst, hipStream_t stream);
    template void min_gpu<char  >(const DevMem2D_<char>& src1, double src2, const DevMem2D_<char>& dst, hipStream_t stream);
    template void min_gpu<ushort>(const DevMem2D_<ushort>& src1, double src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void min_gpu<short >(const DevMem2D_<short>& src1, double src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void min_gpu<int   >(const DevMem2D_<int>& src1, double src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void min_gpu<float >(const DevMem2D_<float>& src1, double src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void min_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);
    
    template <typename T>
    void max_gpu(const DevMem2D_<T>& src1, double src2, const DevMem2D_<T>& dst, hipStream_t stream)
    {
        ScalarMaxOp op(src2);
        transform(src1, dst, op, stream);    
    }

    template void max_gpu<uchar >(const DevMem2D& src1, double src2, const DevMem2D& dst, hipStream_t stream);
    template void max_gpu<char  >(const DevMem2D_<char>& src1, double src2, const DevMem2D_<char>& dst, hipStream_t stream);
    template void max_gpu<ushort>(const DevMem2D_<ushort>& src1, double src2, const DevMem2D_<ushort>& dst, hipStream_t stream);
    template void max_gpu<short >(const DevMem2D_<short>& src1, double src2, const DevMem2D_<short>& dst, hipStream_t stream);
    template void max_gpu<int   >(const DevMem2D_<int>& src1, double src2, const DevMem2D_<int>& dst, hipStream_t stream);
    template void max_gpu<float >(const DevMem2D_<float>& src1, double src2, const DevMem2D_<float>& dst, hipStream_t stream);
    template void max_gpu<double>(const DevMem2D_<double>& src1, double src2, const DevMem2D_<double>& dst, hipStream_t stream);
}}}
