#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"

#define ROWSperTHREAD 21     // the number of rows a thread will process
#define BLOCK_W 128          // the thread block width (464)
#define N_DISPARITIES 8

#define STEREO_MIND 0                    // The minimum d range to check 
#define STEREO_DISP_STEP N_DISPARITIES   // the d step, must be <= 1 to avoid aliasing
#define RADIUS 9                         // Kernel Radius 5V & 5H = 11x11 kernel

#define WINSZ (2 * RADIUS + 1)
#define N_DIRTY_PIXELS (2 * RADIUS)
#define COL_SSD_SIZE (BLOCK_W + N_DIRTY_PIXELS)
#define SHARED_MEM_SIZE (COL_SSD_SIZE) // amount of shared memory used

__constant__ unsigned int* cminSSDImage;
__constant__ size_t cminSSD_step;
__constant__ int cwidth;
__constant__ int cheight;

namespace device_code 
{

__device__ int SQ(int a)
{
    return a * a;    
}

__device__ unsigned int CalcSSD(unsigned int *col_ssd_cache, unsigned int *col_ssd)
{
    unsigned int cache = 0;
    unsigned int cache2 = 0;

    for(int i = 1; i <= RADIUS; i++)
        cache += col_ssd[i];
    
    col_ssd_cache[0] = cache;

    __syncthreads();

    if (threadIdx.x < BLOCK_W - RADIUS)
        cache2 = col_ssd_cache[RADIUS];
    else
        for(int i = RADIUS + 1; i < WINSZ; i++)
            cache2 += col_ssd[i];

    return col_ssd[0] + cache + cache2;
}

__device__ uint2 MinSSD(unsigned int *col_ssd_cache, unsigned int *col_ssd)
{
    unsigned int ssd[N_DISPARITIES];

    ssd[0] = CalcSSD(col_ssd_cache, col_ssd + 0 * SHARED_MEM_SIZE);
    ssd[1] = CalcSSD(col_ssd_cache, col_ssd + 1 * SHARED_MEM_SIZE);
    ssd[2] = CalcSSD(col_ssd_cache, col_ssd + 2 * SHARED_MEM_SIZE);
    ssd[3] = CalcSSD(col_ssd_cache, col_ssd + 3 * SHARED_MEM_SIZE);
    ssd[4] = CalcSSD(col_ssd_cache, col_ssd + 4 * SHARED_MEM_SIZE);
    ssd[5] = CalcSSD(col_ssd_cache, col_ssd + 5 * SHARED_MEM_SIZE);
    ssd[6] = CalcSSD(col_ssd_cache, col_ssd + 6 * SHARED_MEM_SIZE);
    ssd[7] = CalcSSD(col_ssd_cache, col_ssd + 7 * SHARED_MEM_SIZE);

    int mssd = min(min(min(ssd[0], ssd[1]), min(ssd[4], ssd[5])), min(min(ssd[2], ssd[3]), min(ssd[6], ssd[7])));

    int bestIdx = 0;
    for (int i = 0; i < N_DISPARITIES; i++)
    {
        if (mssd == ssd[i])
            bestIdx = i;
    }

    return make_uint2(mssd, bestIdx);
}

__device__ void StepDown(int idx1, int idx2, unsigned char* imageL, unsigned char* imageR, int d, unsigned int *col_ssd)
{
    unsigned char leftPixel1;
    unsigned char leftPixel2;
    unsigned char rightPixel1[8];
    unsigned char rightPixel2[8];
    unsigned int diff1, diff2;

    leftPixel1 = imageL[idx1];
    leftPixel2 = imageL[idx2];

    idx1 = idx1 - d;
    idx2 = idx2 - d;

    rightPixel1[7] = imageR[idx1 - 7];
    rightPixel1[0] = imageR[idx1 - 0];
    rightPixel1[1] = imageR[idx1 - 1];
    rightPixel1[2] = imageR[idx1 - 2];
    rightPixel1[3] = imageR[idx1 - 3];
    rightPixel1[4] = imageR[idx1 - 4];
    rightPixel1[5] = imageR[idx1 - 5];
    rightPixel1[6] = imageR[idx1 - 6];
    
    rightPixel2[7] = imageR[idx2 - 7];
    rightPixel2[0] = imageR[idx2 - 0];
    rightPixel2[1] = imageR[idx2 - 1];
    rightPixel2[2] = imageR[idx2 - 2];
    rightPixel2[3] = imageR[idx2 - 3];
    rightPixel2[4] = imageR[idx2 - 4];
    rightPixel2[5] = imageR[idx2 - 5];
    rightPixel2[6] = imageR[idx2 - 6];
    

    diff1 = leftPixel1 - rightPixel1[0];                
    diff2 = leftPixel2 - rightPixel2[0];    
    col_ssd[0 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);

    diff1 = leftPixel1 - rightPixel1[1];
    diff2 = leftPixel2 - rightPixel2[1];
    col_ssd[1 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
     
    diff1 = leftPixel1 - rightPixel1[2];
    diff2 = leftPixel2 - rightPixel2[2];
    col_ssd[2 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);

    diff1 = leftPixel1 - rightPixel1[3];
    diff2 = leftPixel2 - rightPixel2[3];
    col_ssd[3 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
    
    diff1 = leftPixel1 - rightPixel1[4]; 
    diff2 = leftPixel2 - rightPixel2[4];               
    col_ssd[4 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
    
    diff1 = leftPixel1 - rightPixel1[5];
    diff2 = leftPixel2 - rightPixel2[5];
    col_ssd[5 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
    
    diff1 = leftPixel1 - rightPixel1[6];
    diff2 = leftPixel2 - rightPixel2[6];
    col_ssd[6 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
        
    diff1 = leftPixel1 - rightPixel1[7];
    diff2 = leftPixel2 - rightPixel2[7];
    col_ssd[7 * SHARED_MEM_SIZE] += SQ(diff2) - SQ(diff1);
}

__device__ void InitColSSD(int x_tex, int y_tex, int im_pitch, unsigned char* imageL, unsigned char* imageR, int d, unsigned int *col_ssd)
{
    unsigned char leftPixel1;
    int idx;
    unsigned int diffa[] = {0, 0, 0, 0, 0, 0, 0, 0};

    for(int i = 0; i < WINSZ; i++)
    {
        idx = y_tex * im_pitch + x_tex;
        leftPixel1 = imageL[idx];
        idx = idx - d;

        diffa[0] += SQ(leftPixel1 - imageR[idx - 0]);        
        diffa[1] += SQ(leftPixel1 - imageR[idx - 1]);
        diffa[2] += SQ(leftPixel1 - imageR[idx - 2]);
        diffa[3] += SQ(leftPixel1 - imageR[idx - 3]);
        diffa[4] += SQ(leftPixel1 - imageR[idx - 4]);
        diffa[5] += SQ(leftPixel1 - imageR[idx - 5]);
        diffa[6] += SQ(leftPixel1 - imageR[idx - 6]);
        diffa[7] += SQ(leftPixel1 - imageR[idx - 7]);

        y_tex += 1;
    }

    col_ssd[0 * SHARED_MEM_SIZE] = diffa[0];
    col_ssd[1 * SHARED_MEM_SIZE] = diffa[1];
    col_ssd[2 * SHARED_MEM_SIZE] = diffa[2];
    col_ssd[3 * SHARED_MEM_SIZE] = diffa[3];
    col_ssd[4 * SHARED_MEM_SIZE] = diffa[4];
    col_ssd[5 * SHARED_MEM_SIZE] = diffa[5];
    col_ssd[6 * SHARED_MEM_SIZE] = diffa[6];
    col_ssd[7 * SHARED_MEM_SIZE] = diffa[7];
}

extern "C" __global__ void stereoKernel(unsigned char *left, unsigned char *right, size_t img_step, unsigned char* disp, size_t disp_pitch, int maxdisp)
{
    extern __shared__ unsigned int col_ssd_cache[];
    unsigned int *col_ssd = col_ssd_cache + BLOCK_W + threadIdx.x;    
    unsigned int *col_ssd_extra = threadIdx.x < N_DIRTY_PIXELS ? col_ssd + BLOCK_W : 0;

    //#define X (blockIdx.x * BLOCK_W + threadIdx.x + STEREO_MAXD)
    int X = (blockIdx.x * BLOCK_W + threadIdx.x + maxdisp);
    //#define Y (__mul24(blockIdx.y, ROWSperTHREAD) + RADIUS)
    #define Y (blockIdx.y * ROWSperTHREAD + RADIUS)
    //int Y = blockIdx.y * ROWSperTHREAD + RADIUS;

    unsigned int* minSSDImage = cminSSDImage + X + Y * cminSSD_step;
    unsigned char* disparImage = disp + X + Y * disp_pitch;
 /*   if (X < cwidth)
    {        
        unsigned int *minSSDImage_end = minSSDImage + min(ROWSperTHREAD, cheight - Y) * minssd_step;
        for(uint *ptr = minSSDImage; ptr != minSSDImage_end; ptr += minssd_step )
            *ptr = 0xFFFFFFFF;        
    }*/
    int end_row = min(ROWSperTHREAD, cheight - Y);
    int y_tex;    
    int x_tex = X - RADIUS;
    for(int d = STEREO_MIND; d < maxdisp; d += STEREO_DISP_STEP)
    {
        y_tex = Y - RADIUS;

        InitColSSD(x_tex, y_tex, img_step, left, right, d, col_ssd); 

        if (col_ssd_extra > 0)
            InitColSSD(x_tex + BLOCK_W, y_tex, img_step, left, right, d, col_ssd_extra);

        __syncthreads(); //before MinSSD function

        if (X < cwidth - RADIUS && Y < cheight - RADIUS)
        {
            uint2 minSSD = MinSSD(col_ssd_cache + threadIdx.x, col_ssd);
            if (minSSD.x < minSSDImage[0])
            {
                disparImage[0] = (unsigned char)(d + minSSD.y);
                minSSDImage[0] = minSSD.x;
            }
        }

        for(int row = 1; row < end_row; row++)
        {
            int idx1 = y_tex * img_step + x_tex;
            int idx2 = (y_tex + WINSZ) * img_step + x_tex;

            __syncthreads();

            StepDown(idx1, idx2, left, right, d, col_ssd);

            if (col_ssd_extra)
                StepDown(idx1, idx2, left + BLOCK_W, right + BLOCK_W, d, col_ssd_extra);

            y_tex += 1;
 
            __syncthreads(); //before MinSSD function

            if (X < cwidth - RADIUS && row < cheight - RADIUS - Y)
            {       
                int idx = row * cminSSD_step;         
                uint2 minSSD = MinSSD(col_ssd_cache + threadIdx.x, col_ssd);  
                if (minSSD.x < minSSDImage[idx])
                {
                    disparImage[disp_pitch * row] = (unsigned char)(d + minSSD.y);
                    minSSDImage[idx] = minSSD.x;
                }
            }
        } // for row loop
    } // for d loop
}

}

extern "C" void cv::gpu::impl::stereoBM_GPU(const DevMem2D& left, const DevMem2D& right, DevMem2D& disp, int maxdisp, DevMem2D_<unsigned int>& minSSD_buf)
{   
    //cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(&stereoKernel), hipFuncCachePreferL1) );
    //cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(&stereoKernel), hipFuncCachePreferShared) );
    
    size_t smem_size = (BLOCK_W + N_DISPARITIES * SHARED_MEM_SIZE) * sizeof(unsigned int);      

    cudaSafeCall( hipMemset2D(disp.ptr, disp.step, 0, disp.cols, disp. rows) );
    cudaSafeCall( hipMemset2D(minSSD_buf.ptr, minSSD_buf.step, 0xFF, minSSD_buf.cols * minSSD_buf.elemSize(), disp. rows) );        

    dim3 grid(1,1,1);
    dim3 threads(BLOCK_W, 1, 1);    
    
    grid.x = divUp(left.cols - maxdisp - 2 * RADIUS, BLOCK_W);
    grid.y = divUp(left.rows - 2 * RADIUS, ROWSperTHREAD);
    
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  cwidth), &left.cols, sizeof (left.cols) ) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cheight), &left.rows, sizeof (left.rows) ) );
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cminSSDImage),  &minSSD_buf.ptr, sizeof (minSSD_buf.ptr) ) );

    size_t minssd_step = minSSD_buf.step/minSSD_buf.elemSize();
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( cminSSD_step),  &minssd_step, sizeof (minssd_step) ) );
         
    device_code::stereoKernel<<<grid, threads, smem_size>>>(left.ptr, right.ptr, left.step, disp.ptr, disp.step, maxdisp);
    cudaSafeCall( hipDeviceSynchronize() );
}