#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <stddef.h>
#include <iostream>
#include "cuda_shared.hpp"
#include "hip/hip_runtime.h"

__constant__ float scalar_d[4];

namespace mat_operators
{

    template <typename T, int channels, int count = channels>
    struct unroll
    {
        __device__ static void unroll_set(T * mat, size_t i)
        {
            mat[i] = static_cast<T>(scalar_d[i % channels]);
            unroll<T, channels, count - 1>::unroll_set(mat, i+1);
        }

        __device__ static void unroll_set_with_mask(T * mat, float mask, size_t i)
        {
            mat[i] = mask * static_cast<T>(scalar_d[i % channels]);
            unroll<T, channels, count - 1>::unroll_set_with_mask(mat, mask, i+1);
        }
    };

    template <typename T, int channels>
    struct unroll<T,channels,0>
    {
        __device__ static void unroll_set(T * , size_t){}
        __device__ static void unroll_set_with_mask(T * , float, size_t){}
    };

    template <typename T, int channels>
    __global__ void kernel_set_to_without_mask(T * mat)
    {
        size_t i = (blockIdx.x * blockDim.x + threadIdx.x) * sizeof(T);
        unroll<T, channels>::unroll_set(mat, i);
    }

    template <typename T, int channels>
    __global__ void kernel_set_to_with_mask(T * mat, const float * mask)
    {
        size_t i = (blockIdx.x * blockDim.x + threadIdx.x) * sizeof(T);
        unroll<T, channels>::unroll_set_with_mask(mat, i, mask[i]);
    }
}


extern "C" void cv::gpu::impl::set_to_with_mask(const DevMem2D& mat, const double * scalar, const DevMem2D& mask, int elemSize1, int channels)
{
    // download scalar to constant memory
    float data[4];
    data[0] = scalar[0];
    data[1] = scalar[1];
    data[2] = scalar[2];
    data[3] = scalar[3];
    hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), data, sizeof(data));

    dim3 numBlocks(mat.rows * mat.step / 256, 1, 1);
    dim3 threadsPerBlock(256);

    if (channels == 1)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  1><<<numBlocks,threadsPerBlock>>>(mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 1><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<unsigned int,   1><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr, (float *)mask.ptr);
    }
    if (channels == 2)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  2><<<numBlocks,threadsPerBlock>>>(mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 2><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<unsigned int,   2><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr, (float *)mask.ptr);
    }
    if (channels == 3)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  3><<<numBlocks,threadsPerBlock>>>(mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 3><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (float *)mask.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<unsigned int,   3><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr, (float *)mask.ptr);
    }
}

extern "C" void cv::gpu::impl::set_to_without_mask(const DevMem2D& mat, const double * scalar, int elemSize1, int channels)
{
    float data[4];
    data[0] = scalar[0];
    data[1] = scalar[1];
    data[2] = scalar[2];
    data[3] = scalar[3];
    hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), data, sizeof(data));

    int numBlocks = mat.rows * mat.step / 256;

    dim3 threadsPerBlock(256);

    if (channels == 1)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  1><<<numBlocks,threadsPerBlock>>>(mat.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 1><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<unsigned int,   1><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr);
    }
    if (channels == 2)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  2><<<numBlocks,threadsPerBlock>>>(mat.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 2><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<unsigned int,   2><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr);
    }
    if (channels == 3)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  3><<<numBlocks,threadsPerBlock>>>(mat.ptr);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 3><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<unsigned int,   3><<<numBlocks,threadsPerBlock>>>((unsigned int *)mat.ptr);
    }
}
