#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <stddef.h>
#include <stdio.h>
#include <iostream>
#include "cuda_shared.hpp"
#include "hip/hip_runtime.h"

using namespace cv::gpu;
using namespace cv::gpu::impl;

__constant__ __align__(16) float scalar_d[4];

namespace mat_operators
{
    //////////////////////////////////////////////////////////
    // SetTo
    //////////////////////////////////////////////////////////
	
    template<typename T, int channels>
    __global__ void kernel_set_to_without_mask(T * mat, int cols, int rows, int step)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
        {
            size_t idx = y * (step / sizeof(T)) + x;
            mat[idx] = scalar_d[ x % channels ];
        }
    }

    template<typename T, int channels>
    __global__ void kernel_set_to_with_mask(T * mat, const unsigned char * mask, int cols, int rows, int step, int step_mask)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if (mask[y * step_mask + x] != 0)
            if ((x < cols * channels ) && (y < rows))
            {
                size_t idx = y * (step / sizeof(T)) + x;
                mat[idx] = scalar_d[ x % channels ];
            }
    }

	
    //////////////////////////////////////////////////////////
    // ConvertTo
    //////////////////////////////////////////////////////////

    template <typename T, typename DT, size_t src_elem_size, size_t dst_elem_size>
    struct Converter
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);

                dst[x] = (DT)__double2int_rn(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };
	
    template <typename T, typename DT>
    struct Converter<T, DT, 1, 1>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 2) + 3 < width)
                {
                    uchar4 src4b = ((const uchar4*)src)[x];
                    uchar4 dst4b;

                    const T* src1b = (const T*) &src4b.x;
                    DT* dst1b = (DT*) &dst4b.x;
                    
                    dst1b[0] = (DT)__double2int_rn(alpha * src1b[0] + beta);
                    dst1b[1] = (DT)__double2int_rn(alpha * src1b[1] + beta);
                    dst1b[2] = (DT)__double2int_rn(alpha * src1b[2] + beta);
                    dst1b[3] = (DT)__double2int_rn(alpha * src1b[3] + beta);

                    ((uchar4*)dst)[x] = dst4b;
                }
                else
                {
                    if ((x << 2) + 0 < width)
                        dst[(x << 2) + 0] = (DT)__double2int_rn(alpha * src[(x << 2) + 0] + beta);

                    if ((x << 2) + 1 < width)
                        dst[(x << 2) + 1] = (DT)__double2int_rn(alpha * src[(x << 2) + 1] + beta);

                    if ((x << 2) + 2 < width)
                        dst[(x << 2) + 2] = (DT)__double2int_rn(alpha * src[(x << 2) + 2] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 2), divUp(height, block.y));
        }
    };/**/
	
    template <typename T, typename DT>
    struct Converter<T, DT, 1, 2>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 1) + 1 < width)
                {
                    uchar2 src2b = ((const uchar2*)src)[x];
                    ushort2 dst2s;

                    const T* src1b = (const T*) &src2b;
                    DT* dst1s = (DT*) &dst2s;
                    dst1s[0] = (DT)__double2int_rn(alpha * src1b[0] + beta);
                    dst1s[1] = (DT)__double2int_rn(alpha * src1b[1] + beta);

                    ((ushort2*)(dst))[x] = dst2s;
                }
                else
                {
                    if ((x << 1) < width)
                        dst[(x << 1)] = (DT)__double2int_rn(alpha * src[(x << 1)] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 1), divUp(height, block.y));
        }
    };/**/
	
    template <typename T, typename DT>
    struct Converter<T, DT, 2, 1>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 2) + 3 < width)
                {                    
                    ushort4 src4s = ((const ushort4*)src)[x];
                    uchar4 dst4b;

                    const T* src1s = (const T*) &src4s.x;
                    DT* dst1b = (DT*) &dst4b.x;
                    dst1b[0] = (DT)__double2int_rn(alpha * src1s[0] + beta);
                    dst1b[1] = (DT)__double2int_rn(alpha * src1s[1] + beta);
                    dst1b[2] = (DT)__double2int_rn(alpha * src1s[2] + beta);
                    dst1b[3] = (DT)__double2int_rn(alpha * src1s[3] + beta);

                    ((uchar4*)(dst))[x] = dst4b;
                }
                else
                {
                    if ((x << 2) + 0 < width)
                        dst[(x << 2) + 0] = (DT)__double2int_rn(alpha * src[(x << 2) + 0] + beta);
                    if ((x << 2) + 1 < width)
                        dst[(x << 2) + 1] = (DT)__double2int_rn(alpha * src[(x << 2) + 1] + beta);
                    if ((x << 2) + 2 < width)
                        dst[(x << 2) + 2] = (DT)__double2int_rn(alpha * src[(x << 2) + 2] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 2), divUp(height, block.y));
        }
    };/**/
	
    template <typename T, typename DT>
    struct Converter<T, DT, 2, 2>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 1) + 1 < width)
                {
                    ushort2 src2s = ((const ushort2*)src)[x];
                    ushort2 dst2s;

                    const T* src1s = (const T*) &src2s.x;
                    DT* dst1s = (DT*) &dst2s.x;
                    dst1s[0] = (DT)__double2int_rn(alpha * src1s[0] + beta);
                    dst1s[1] = (DT)__double2int_rn(alpha * src1s[1] + beta);

                    ((ushort2*)dst)[x] = dst2s;
                }
                else
                {
                    if ((x << 1) < width)
                        dst[(x << 1)] = (DT)__double2int_rn(alpha * src[(x << 1)] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 1), divUp(height, block.y));
        }
    };/**/
	
    template <typename T, size_t src_elem_size, size_t dst_elem_size>
    struct Converter<T, float, src_elem_size, dst_elem_size>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                float* dst = (float*)(dstmat + dst_step * y);

                dst[x] = (float)(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };
	
    template <typename T, size_t src_elem_size, size_t dst_elem_size>
    struct Converter<T, double, src_elem_size, dst_elem_size>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                double* dst = (double*)(dstmat + dst_step * y);

                dst[x] = (double)(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };   
    
    template <typename T, typename DT> 
    __global__ static void kernel_convert_to(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
    {
        Converter<T, DT, sizeof(T), sizeof(DT)>::convert(srcmat, src_step, dstmat, dst_step, width, height, alpha, beta);
    }
	
} // namespace mat_operators

//////////////////////////////////////////////////////////////
// SetTo
//////////////////////////////////////////////////////////////

extern "C" void cv::gpu::impl::set_to_without_mask(const DevMem2D& mat, const double * scalar, int elemSize1, int channels)
{
    float data[4];
    data[0] = static_cast<float>(scalar[0]);
    data[1] = static_cast<float>(scalar[1]);
    data[2] = static_cast<float>(scalar[2]);
    data[3] = static_cast<float>(scalar[3]);
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

    if (channels == 1)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  1><<<numBlocks,threadsPerBlock>>>(mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 1><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<float,          1><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr, mat.cols, mat.rows, mat.step);
    }
    if (channels == 2)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  2><<<numBlocks,threadsPerBlock>>>(mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 2><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<float,          2><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr, mat.cols, mat.rows, mat.step);
    }
    if (channels == 3)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  3><<<numBlocks,threadsPerBlock>>>(mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 3><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<float,          3><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr, mat.cols, mat.rows, mat.step);
    }
    if (channels == 4)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_without_mask<unsigned char,  4><<<numBlocks,threadsPerBlock>>>(mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_without_mask<unsigned short, 4><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, mat.cols, mat.rows, mat.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_without_mask<float,          4><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr, mat.cols, mat.rows, mat.step);
    }

    cudaSafeCall ( hipDeviceSynchronize() );
}

extern "C" void cv::gpu::impl::set_to_with_mask(const DevMem2D& mat, const double * scalar, const DevMem2D& mask, int elemSize1, int channels)
{
    float data[4];
    data[0] = static_cast<float>(scalar[0]);
    data[1] = static_cast<float>(scalar[1]);
    data[2] = static_cast<float>(scalar[2]);
    data[3] = static_cast<float>(scalar[3]);
    cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);

    if (channels == 1)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  1><<<numBlocks,threadsPerBlock>>>(mat.ptr,                   (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 1><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<float,          1><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr,          (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
    }
    if (channels == 2)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  2><<<numBlocks,threadsPerBlock>>>(mat.ptr,                   (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 2><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<float,          2><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr,          (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
    }
    if (channels == 3)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  3><<<numBlocks,threadsPerBlock>>>(mat.ptr,                   (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 3><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<float,          3><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr,          (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
    }
    if (channels == 4)
    {
        if (elemSize1 == 1) ::mat_operators::kernel_set_to_with_mask<unsigned char,  4><<<numBlocks,threadsPerBlock>>>(mat.ptr,                   (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 2) ::mat_operators::kernel_set_to_with_mask<unsigned short, 4><<<numBlocks,threadsPerBlock>>>((unsigned short *)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
        if (elemSize1 == 4) ::mat_operators::kernel_set_to_with_mask<float,          4><<<numBlocks,threadsPerBlock>>>((float *)mat.ptr,          (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, mask.step);
    }

    cudaSafeCall ( hipDeviceSynchronize() );
}

//////////////////////////////////////////////////////////////
// ConvertTo
//////////////////////////////////////////////////////////////

namespace cv 
{ 
	namespace gpu 
	{
		namespace impl 
		{   

			typedef void (*CvtFunc)(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta);

			//#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 130)

			template<typename T, typename DT> 
			void cvt_(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta)
			{
				dim3 block(32, 8);
				dim3 grid = ::mat_operators::Converter<T, DT, sizeof(T), sizeof(DT)>::calcGrid(width, height, block);
				::mat_operators::kernel_convert_to<T, DT><<<grid, block>>>(src.ptr, src.step, dst.ptr, dst.step, width, height, alpha, beta);
				cudaSafeCall( hipDeviceSynchronize() );
			}
			//#endif

			extern "C" void convert_to(const DevMem2D& src, int sdepth, DevMem2D dst, int ddepth, size_t width, size_t height, double alpha, double beta)
			{
				static CvtFunc tab[8][8] =
				{
					{cvt_<uchar, uchar>, cvt_<uchar, schar>, cvt_<uchar, ushort>, cvt_<uchar, short>,
					cvt_<uchar, int>, cvt_<uchar, float>, cvt_<uchar, double>, 0},

					{cvt_<schar, uchar>, cvt_<schar, schar>, cvt_<schar, ushort>, cvt_<schar, short>,
					cvt_<schar, int>, cvt_<schar, float>, cvt_<schar, double>, 0},

					{cvt_<ushort, uchar>, cvt_<ushort, schar>, cvt_<ushort, ushort>, cvt_<ushort, short>,
					cvt_<ushort, int>, cvt_<ushort, float>, cvt_<ushort, double>, 0},

					{cvt_<short, uchar>, cvt_<short, schar>, cvt_<short, ushort>, cvt_<short, short>,
					cvt_<short, int>, cvt_<short, float>, cvt_<short, double>, 0},

					{cvt_<int, uchar>, cvt_<int, schar>, cvt_<int, ushort>,
					cvt_<int, short>, cvt_<int, int>, cvt_<int, float>, cvt_<int, double>, 0},

					{cvt_<float, uchar>, cvt_<float, schar>, cvt_<float, ushort>,
					cvt_<float, short>, cvt_<float, int>, cvt_<float, float>, cvt_<float, double>, 0},

					{cvt_<double, uchar>, cvt_<double, schar>, cvt_<double, ushort>,
					cvt_<double, short>, cvt_<double, int>, cvt_<double, float>, cvt_<double, double>, 0},

					{0,0,0,0,0,0,0,0}
				};

				CvtFunc func = tab[sdepth][ddepth];
				if (func == 0)
					error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);
				func(src, dst, width, height, alpha, beta);
			}
		}
		
		
	} 
}
