#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <stddef.h>
#include <stdio.h>
//#include <iostream>
#include "cuda_shared.hpp"
#include "hip/hip_runtime.h"

using namespace cv::gpu;
using namespace cv::gpu::impl;

__constant__ __align__(16) double scalar_d[4];

namespace mat_operators
{
    //////////////////////////////////////////////////////////
    // CopyTo
    //////////////////////////////////////////////////////////

    template<typename T>
    __global__ void kernel_copy_to_with_mask(T * mat_src, T * mat_dst, const unsigned char * mask, int cols, int rows, int step_mat, int step_mask, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * (step_mat / sizeof(T)) + x;
                mat_dst[idx] = mat_src[idx];
            }
    }

    //////////////////////////////////////////////////////////
    // SetTo
    //////////////////////////////////////////////////////////

    template<typename T>
    __global__ void kernel_set_to_without_mask(T * mat, int cols, int rows, int step, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
        {
            size_t idx = y * (step / sizeof(T)) + x;
            mat[idx] = scalar_d[ x % channels ];
        }
    }

    template<typename T>
    __global__ void kernel_set_to_with_mask(T * mat, const unsigned char * mask, int cols, int rows, int step, int channels, int step_mask)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * (step / sizeof(T)) + x;
                mat[idx] = scalar_d[ x % channels ];
            }
    }


    //////////////////////////////////////////////////////////
    // ConvertTo
    //////////////////////////////////////////////////////////

    template <typename T, typename DT, size_t src_elem_size, size_t dst_elem_size>
    struct Converter
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);

                dst[x] = (DT)__double2int_rn(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };

    template <typename T, typename DT>
    struct Converter<T, DT, 1, 1>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 2) + 3 < width)
                {
                    uchar4 src4b = ((const uchar4*)src)[x];
                    uchar4 dst4b;

                    const T* src1b = (const T*) &src4b.x;
                    DT* dst1b = (DT*) &dst4b.x;

                    dst1b[0] = (DT)__double2int_rn(alpha * src1b[0] + beta);
                    dst1b[1] = (DT)__double2int_rn(alpha * src1b[1] + beta);
                    dst1b[2] = (DT)__double2int_rn(alpha * src1b[2] + beta);
                    dst1b[3] = (DT)__double2int_rn(alpha * src1b[3] + beta);

                    ((uchar4*)dst)[x] = dst4b;
                }
                else
                {
                    if ((x << 2) + 0 < width)
                        dst[(x << 2) + 0] = (DT)__double2int_rn(alpha * src[(x << 2) + 0] + beta);

                    if ((x << 2) + 1 < width)
                        dst[(x << 2) + 1] = (DT)__double2int_rn(alpha * src[(x << 2) + 1] + beta);

                    if ((x << 2) + 2 < width)
                        dst[(x << 2) + 2] = (DT)__double2int_rn(alpha * src[(x << 2) + 2] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 2), divUp(height, block.y));
        }
    };/**/

    template <typename T, typename DT>
    struct Converter<T, DT, 1, 2>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 1) + 1 < width)
                {
                    uchar2 src2b = ((const uchar2*)src)[x];
                    ushort2 dst2s;

                    const T* src1b = (const T*) &src2b;
                    DT* dst1s = (DT*) &dst2s;
                    dst1s[0] = (DT)__double2int_rn(alpha * src1b[0] + beta);
                    dst1s[1] = (DT)__double2int_rn(alpha * src1b[1] + beta);

                    ((ushort2*)(dst))[x] = dst2s;
                }
                else
                {
                    if ((x << 1) < width)
                        dst[(x << 1)] = (DT)__double2int_rn(alpha * src[(x << 1)] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 1), divUp(height, block.y));
        }
    };/**/

    template <typename T, typename DT>
    struct Converter<T, DT, 2, 1>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 2) + 3 < width)
                {
                    ushort4 src4s = ((const ushort4*)src)[x];
                    uchar4 dst4b;

                    const T* src1s = (const T*) &src4s.x;
                    DT* dst1b = (DT*) &dst4b.x;
                    dst1b[0] = (DT)__double2int_rn(alpha * src1s[0] + beta);
                    dst1b[1] = (DT)__double2int_rn(alpha * src1s[1] + beta);
                    dst1b[2] = (DT)__double2int_rn(alpha * src1s[2] + beta);
                    dst1b[3] = (DT)__double2int_rn(alpha * src1s[3] + beta);

                    ((uchar4*)(dst))[x] = dst4b;
                }
                else
                {
                    if ((x << 2) + 0 < width)
                        dst[(x << 2) + 0] = (DT)__double2int_rn(alpha * src[(x << 2) + 0] + beta);
                    if ((x << 2) + 1 < width)
                        dst[(x << 2) + 1] = (DT)__double2int_rn(alpha * src[(x << 2) + 1] + beta);
                    if ((x << 2) + 2 < width)
                        dst[(x << 2) + 2] = (DT)__double2int_rn(alpha * src[(x << 2) + 2] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 2), divUp(height, block.y));
        }
    };/**/

    template <typename T, typename DT>
    struct Converter<T, DT, 2, 2>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x << 1) + 1 < width)
                {
                    ushort2 src2s = ((const ushort2*)src)[x];
                    ushort2 dst2s;

                    const T* src1s = (const T*) &src2s.x;
                    DT* dst1s = (DT*) &dst2s.x;
                    dst1s[0] = (DT)__double2int_rn(alpha * src1s[0] + beta);
                    dst1s[1] = (DT)__double2int_rn(alpha * src1s[1] + beta);

                    ((ushort2*)dst)[x] = dst2s;
                }
                else
                {
                    if ((x << 1) < width)
                        dst[(x << 1)] = (DT)__double2int_rn(alpha * src[(x << 1)] + beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x << 1), divUp(height, block.y));
        }
    };/**/

    template <typename T, size_t src_elem_size, size_t dst_elem_size>
    struct Converter<T, float, src_elem_size, dst_elem_size>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                float* dst = (float*)(dstmat + dst_step * y);

                dst[x] = (float)(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };

    template <typename T, size_t src_elem_size, size_t dst_elem_size>
    struct Converter<T, double, src_elem_size, dst_elem_size>
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (x < width && y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                double* dst = (double*)(dstmat + dst_step * y);

                dst[x] = (double)(alpha * src[x] + beta);
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x), divUp(height, block.y));
        }
    };

    template <typename T, typename DT>
    __global__ static void kernel_convert_to(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
    {
        Converter<T, DT, sizeof(T), sizeof(DT)>::convert(srcmat, src_step, dstmat, dst_step, width, height, alpha, beta);
    }

} // namespace mat_operators

namespace cv
{
	namespace gpu
	{
		namespace impl
		{

                        //////////////////////////////////////////////////////////////
                        // CopyTo
                        //////////////////////////////////////////////////////////////

                        typedef void (*CopyToFunc)(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels);

                        template<typename T>
                        void copy_to_with_mask_run(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels)
                        {
                            dim3 threadsPerBlock(16,16, 1);
                            dim3 numBlocks ( divUp(mat_src.cols * channels , threadsPerBlock.x) , divUp(mat_src.rows , threadsPerBlock.y), 1);
                            ::mat_operators::kernel_copy_to_with_mask<T><<<numBlocks,threadsPerBlock>>>
                            ((T*)mat_src.ptr, (T*)mat_dst.ptr, (unsigned char*)mask.ptr, mat_src.cols, mat_src.rows, mat_src.step, mask.step, channels);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        extern "C" void copy_to_with_mask(const DevMem2D& mat_src, const DevMem2D& mat_dst, int depth, const DevMem2D& mask, int channels)
                        {
                            static CopyToFunc tab[8] =
                            {
                                copy_to_with_mask_run<unsigned char>,
                                copy_to_with_mask_run<char>,
                                copy_to_with_mask_run<unsigned short>,
                                copy_to_with_mask_run<short>,
                                copy_to_with_mask_run<int>,
                                copy_to_with_mask_run<float>,
                                copy_to_with_mask_run<double>,
                                0
                            };

                            CopyToFunc func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat_src, mat_dst, mask, channels);
                        }


                        //////////////////////////////////////////////////////////////
                        // SetTo
                        //////////////////////////////////////////////////////////////

                        typedef void (*SetToFunc_with_mask)(const DevMem2D& mat, const DevMem2D& mask, int channels);
                        typedef void (*SetToFunc_without_mask)(const DevMem2D& mat, int channels);

                        template <typename T>
                        void set_to_with_mask_run(const DevMem2D& mat, const DevMem2D& mask, int channels)
                        {
                            dim3 threadsPerBlock(32, 8, 1);
                            dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);
                            ::mat_operators::kernel_set_to_with_mask<T><<<numBlocks,threadsPerBlock>>>((T*)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, channels, mask.step);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        template <typename T>
                        void set_to_without_mask_run(const DevMem2D& mat, int channels)
                        {
                            dim3 threadsPerBlock(32, 8, 1);
                            dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);
                            ::mat_operators::kernel_set_to_without_mask<T><<<numBlocks,threadsPerBlock>>>((T*)mat.ptr, mat.cols, mat.rows, mat.step, channels);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        extern "C" void set_to_without_mask(const DevMem2D& mat, int depth, const double * scalar, int channels)
                        {
                            double data[4];
                            data[0] = scalar[0];
                            data[1] = scalar[1];
                            data[2] = scalar[2];
                            data[3] = scalar[3];
                            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

                            static SetToFunc_without_mask tab[8] =
                            {
                                set_to_without_mask_run<unsigned char>,
                                set_to_without_mask_run<char>,
                                set_to_without_mask_run<unsigned short>,
                                set_to_without_mask_run<short>,
                                set_to_without_mask_run<int>,
                                set_to_without_mask_run<float>,
                                set_to_without_mask_run<double>,
                                0
                            };

                            SetToFunc_without_mask func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat, channels);
                        }


                        extern "C" void set_to_with_mask(const DevMem2D& mat, int depth, const double * scalar, const DevMem2D& mask, int channels)
                        {
                            double data[4];
                            data[0] = scalar[0];
                            data[1] = scalar[1];
                            data[2] = scalar[2];
                            data[3] = scalar[3];
                            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

                            static SetToFunc_with_mask tab[8] =
                            {
                                set_to_with_mask_run<unsigned char>,
                                set_to_with_mask_run<char>,
                                set_to_with_mask_run<unsigned short>,
                                set_to_with_mask_run<short>,
                                set_to_with_mask_run<int>,
                                set_to_with_mask_run<float>,
                                set_to_with_mask_run<double>,
                                0
                            };

                            SetToFunc_with_mask func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat, mask, channels);
                        }

                        //////////////////////////////////////////////////////////////
                        // ConvertTo
                        //////////////////////////////////////////////////////////////

                        typedef void (*CvtFunc)(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta);

                        //#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 130)

			template<typename T, typename DT>
			void cvt_(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta)
			{
				dim3 block(32, 8);
				dim3 grid = ::mat_operators::Converter<T, DT, sizeof(T), sizeof(DT)>::calcGrid(width, height, block);
				::mat_operators::kernel_convert_to<T, DT><<<grid, block>>>(src.ptr, src.step, dst.ptr, dst.step, width, height, alpha, beta);
				cudaSafeCall( hipDeviceSynchronize() );
			}
			//#endif

			extern "C" void convert_to(const DevMem2D& src, int sdepth, DevMem2D dst, int ddepth, size_t width, size_t height, double alpha, double beta)
			{
				static CvtFunc tab[8][8] =
				{
					{cvt_<uchar, uchar>, cvt_<uchar, schar>, cvt_<uchar, ushort>, cvt_<uchar, short>,
					cvt_<uchar, int>, cvt_<uchar, float>, cvt_<uchar, double>, 0},

					{cvt_<schar, uchar>, cvt_<schar, schar>, cvt_<schar, ushort>, cvt_<schar, short>,
					cvt_<schar, int>, cvt_<schar, float>, cvt_<schar, double>, 0},

					{cvt_<ushort, uchar>, cvt_<ushort, schar>, cvt_<ushort, ushort>, cvt_<ushort, short>,
					cvt_<ushort, int>, cvt_<ushort, float>, cvt_<ushort, double>, 0},

					{cvt_<short, uchar>, cvt_<short, schar>, cvt_<short, ushort>, cvt_<short, short>,
					cvt_<short, int>, cvt_<short, float>, cvt_<short, double>, 0},

					{cvt_<int, uchar>, cvt_<int, schar>, cvt_<int, ushort>,
					cvt_<int, short>, cvt_<int, int>, cvt_<int, float>, cvt_<int, double>, 0},

					{cvt_<float, uchar>, cvt_<float, schar>, cvt_<float, ushort>,
					cvt_<float, short>, cvt_<float, int>, cvt_<float, float>, cvt_<float, double>, 0},

					{cvt_<double, uchar>, cvt_<double, schar>, cvt_<double, ushort>,
					cvt_<double, short>, cvt_<double, int>, cvt_<double, float>, cvt_<double, double>, 0},

					{0,0,0,0,0,0,0,0}
				};

				CvtFunc func = tab[sdepth][ddepth];
				if (func == 0)
					error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);
				func(src, dst, width, height, alpha, beta);
			}
		}


	}
}
