#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <stddef.h>
#include <stdio.h>
//#include <iostream>
#include "cuda_shared.hpp"
#include "hip/hip_runtime.h"

using namespace cv::gpu;
using namespace cv::gpu::impl;

__constant__ __align__(16) double scalar_d[4];

namespace mat_operators
{
///////////////////////////////////////////////////////////////////////////
////////////////////////////////// CopyTo /////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    template<typename T>
    __global__ void kernel_copy_to_with_mask(T * mat_src, T * mat_dst, const unsigned char * mask, int cols, int rows, int step_mat, int step_mask, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * (step_mat / sizeof(T)) + x;
                mat_dst[idx] = mat_src[idx];
            }
    }

	
///////////////////////////////////////////////////////////////////////////
////////////////////////////////// SetTo //////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    template<typename T>
    __global__ void kernel_set_to_without_mask(T * mat, int cols, int rows, int step, int channels)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
        {
            size_t idx = y * (step / sizeof(T)) + x;
            mat[idx] = scalar_d[ x % channels ];
        }
    }

    template<typename T>
    __global__ void kernel_set_to_with_mask(T * mat, const unsigned char * mask, int cols, int rows, int step, int channels, int step_mask)
    {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;

        if ((x < cols * channels ) && (y < rows))
            if (mask[y * step_mask + x / channels] != 0)
            {
                size_t idx = y * (step / sizeof(T)) + x;
                mat[idx] = scalar_d[ x % channels ];
            }
    }


///////////////////////////////////////////////////////////////////////////
//////////////////////////////// ConvertTo ////////////////////////////////
///////////////////////////////////////////////////////////////////////////

    template <typename T, typename DT>
    struct CalcTraits
    {
        __device__ static DT calc(T src, double alpha, double beta)
        {
            return (DT)__double2int_rn(alpha * src + beta);
        }
    };
    template <typename T>
    struct CalcTraits<T, float>
    {
        __device__ static float calc(T src, double alpha, double beta)
        {
            return (float)(alpha * src + beta);
        }
    };
    template <typename T>
    struct CalcTraits<T, double>
    {
        __device__ static double calc(T src, double alpha, double beta)
        {
            return alpha * src + beta;
        }
    };

    template <typename T, typename DT, size_t src_elem_size, size_t dst_elem_size>
    struct ConverterTraits
    {
        enum {shift=1};

        typedef T read_type;
        typedef DT write_type;
    };
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 1, 1>
    {
        enum {shift=4};

        typedef char4 read_type;
        typedef char4 write_type;
    };    
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 2, 1>
    {
        enum {shift=4};

        typedef short4 read_type;
        typedef char4 write_type;
    };    
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 4, 1>
    {
        enum {shift=4};

        typedef int4 read_type;
        typedef char4 write_type;
    };    
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 1, 2>
    {
        enum {shift=2};

        typedef char2 read_type;
        typedef short2 write_type;
    };     
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 2, 2>
    {
        enum {shift=2};

        typedef short2 read_type;
        typedef short2 write_type;
    };     
    template <typename T, typename DT>
    struct ConverterTraits<T, DT, 4, 2>
    {
        enum {shift=2};

        typedef int2 read_type;
        typedef short2 write_type;
    };

    template <typename T, typename DT>
    struct Converter
    {
        __device__ static void convert(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
        {
            size_t x = threadIdx.x + blockIdx.x * blockDim.x;
            size_t y = threadIdx.y + blockIdx.y * blockDim.y;
            if (y < height)
            {
                const T* src = (const T*)(srcmat + src_step * y);
                DT* dst = (DT*)(dstmat + dst_step * y);
                if ((x * ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift) + ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift - 1 < width)
                {
                    typename ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::read_type srcn_el = ((const typename ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::read_type*)src)[x];
                    typename ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::write_type dstn_el;

                    const T* src1_el = (const T*) &srcn_el;
                    DT* dst1_el = (DT*) &dstn_el;

                    for (int i = 0; i < ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift; ++i)
                        dst1_el[i] = CalcTraits<T, DT>::calc(src1_el[i], alpha, beta);

                    ((typename ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::write_type*)dst)[x] = dstn_el;
                }
                else
                {                    
                    for (int i = 0; i < ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift - 1; ++i)
                        if ((x * ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift) + i < width)
                            dst[(x * ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift) + i] = CalcTraits<T, DT>::calc(src[(x * ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift) + i], alpha, beta);
                }
            }
        }
        __host__ static inline dim3 calcGrid(size_t width, size_t height, dim3 block)
        {
            return dim3(divUp(width, block.x * ConverterTraits<T, DT, sizeof(T), sizeof(DT)>::shift), divUp(height, block.y));
        }
    };
    
    template <typename T, typename DT> 
    __global__ static void kernel_convert_to(uchar* srcmat, size_t src_step, uchar* dstmat, size_t dst_step, size_t width, size_t height, double alpha, double beta)
    {
        Converter<T, DT>::convert(srcmat, src_step, dstmat, dst_step, width, height, alpha, beta);
    }

} // namespace mat_operators

namespace cv
{
	namespace gpu
	{
		namespace impl
		{

///////////////////////////////////////////////////////////////////////////
////////////////////////////////// CopyTo /////////////////////////////////
///////////////////////////////////////////////////////////////////////////

                        typedef void (*CopyToFunc)(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels);

                        template<typename T>
                        void copy_to_with_mask_run(const DevMem2D& mat_src, const DevMem2D& mat_dst, const DevMem2D& mask, int channels)
                        {
                            dim3 threadsPerBlock(16,16, 1);
                            dim3 numBlocks ( divUp(mat_src.cols * channels , threadsPerBlock.x) , divUp(mat_src.rows , threadsPerBlock.y), 1);
                            ::mat_operators::kernel_copy_to_with_mask<T><<<numBlocks,threadsPerBlock>>>
                            ((T*)mat_src.ptr, (T*)mat_dst.ptr, (unsigned char*)mask.ptr, mat_src.cols, mat_src.rows, mat_src.step, mask.step, channels);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        extern "C" void copy_to_with_mask(const DevMem2D& mat_src, const DevMem2D& mat_dst, int depth, const DevMem2D& mask, int channels)
                        {
                            static CopyToFunc tab[8] =
                            {
                                copy_to_with_mask_run<unsigned char>,
                                copy_to_with_mask_run<char>,
                                copy_to_with_mask_run<unsigned short>,
                                copy_to_with_mask_run<short>,
                                copy_to_with_mask_run<int>,
                                copy_to_with_mask_run<float>,
                                copy_to_with_mask_run<double>,
                                0
                            };

                            CopyToFunc func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat_src, mat_dst, mask, channels);
                        }


///////////////////////////////////////////////////////////////////////////
////////////////////////////////// SetTo //////////////////////////////////
///////////////////////////////////////////////////////////////////////////

                        typedef void (*SetToFunc_with_mask)(const DevMem2D& mat, const DevMem2D& mask, int channels);
                        typedef void (*SetToFunc_without_mask)(const DevMem2D& mat, int channels);

                        template <typename T>
                        void set_to_with_mask_run(const DevMem2D& mat, const DevMem2D& mask, int channels)
                        {
                            dim3 threadsPerBlock(32, 8, 1);
                            dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);
                            ::mat_operators::kernel_set_to_with_mask<T><<<numBlocks,threadsPerBlock>>>((T*)mat.ptr, (unsigned char *)mask.ptr, mat.cols, mat.rows, mat.step, channels, mask.step);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        template <typename T>
                        void set_to_without_mask_run(const DevMem2D& mat, int channels)
                        {
                            dim3 threadsPerBlock(32, 8, 1);
                            dim3 numBlocks (mat.cols * channels / threadsPerBlock.x + 1, mat.rows / threadsPerBlock.y + 1, 1);
                            ::mat_operators::kernel_set_to_without_mask<T><<<numBlocks,threadsPerBlock>>>((T*)mat.ptr, mat.cols, mat.rows, mat.step, channels);
                            cudaSafeCall ( hipDeviceSynchronize() );
                        }

                        extern "C" void set_to_without_mask(const DevMem2D& mat, int depth, const double * scalar, int channels)
                        {
                            double data[4];
                            data[0] = scalar[0];
                            data[1] = scalar[1];
                            data[2] = scalar[2];
                            data[3] = scalar[3];
                            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

                            static SetToFunc_without_mask tab[8] =
                            {
                                set_to_without_mask_run<unsigned char>,
                                set_to_without_mask_run<char>,
                                set_to_without_mask_run<unsigned short>,
                                set_to_without_mask_run<short>,
                                set_to_without_mask_run<int>,
                                set_to_without_mask_run<float>,
                                set_to_without_mask_run<double>,
                                0
                            };

                            SetToFunc_without_mask func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat, channels);
                        }


                        extern "C" void set_to_with_mask(const DevMem2D& mat, int depth, const double * scalar, const DevMem2D& mask, int channels)
                        {
                            double data[4];
                            data[0] = scalar[0];
                            data[1] = scalar[1];
                            data[2] = scalar[2];
                            data[3] = scalar[3];
                            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(scalar_d), &data, sizeof(data)));

                            static SetToFunc_with_mask tab[8] =
                            {
                                set_to_with_mask_run<unsigned char>,
                                set_to_with_mask_run<char>,
                                set_to_with_mask_run<unsigned short>,
                                set_to_with_mask_run<short>,
                                set_to_with_mask_run<int>,
                                set_to_with_mask_run<float>,
                                set_to_with_mask_run<double>,
                                0
                            };

                            SetToFunc_with_mask func = tab[depth];

                            if (func == 0) error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);

                            func(mat, mask, channels);
                        }

						
///////////////////////////////////////////////////////////////////////////
//////////////////////////////// ConvertTo ////////////////////////////////
///////////////////////////////////////////////////////////////////////////

			            typedef void (*CvtFunc)(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta);

			            template<typename T, typename DT> 
			            void cvt_(const DevMem2D& src, DevMem2D& dst, size_t width, size_t height, double alpha, double beta)
			            {
				            dim3 block(32, 8);
				            dim3 grid = ::mat_operators::Converter<T, DT>::calcGrid(width, height, block);
				            ::mat_operators::kernel_convert_to<T, DT><<<grid, block>>>(src.ptr, src.step, dst.ptr, dst.step, width, height, alpha, beta);
				            cudaSafeCall( hipDeviceSynchronize() );
			            }

			            extern "C" void convert_to(const DevMem2D& src, int sdepth, DevMem2D dst, int ddepth, size_t width, size_t height, double alpha, double beta)
			            {
				            static CvtFunc tab[8][8] =
				            {
					            {cvt_<uchar, uchar>, cvt_<uchar, schar>, cvt_<uchar, ushort>, cvt_<uchar, short>,
					            cvt_<uchar, int>, cvt_<uchar, float>, cvt_<uchar, double>, 0},

					            {cvt_<schar, uchar>, cvt_<schar, schar>, cvt_<schar, ushort>, cvt_<schar, short>,
					            cvt_<schar, int>, cvt_<schar, float>, cvt_<schar, double>, 0},

					            {cvt_<ushort, uchar>, cvt_<ushort, schar>, cvt_<ushort, ushort>, cvt_<ushort, short>,
					            cvt_<ushort, int>, cvt_<ushort, float>, cvt_<ushort, double>, 0},

					            {cvt_<short, uchar>, cvt_<short, schar>, cvt_<short, ushort>, cvt_<short, short>,
					            cvt_<short, int>, cvt_<short, float>, cvt_<short, double>, 0},

					            {cvt_<int, uchar>, cvt_<int, schar>, cvt_<int, ushort>,
					            cvt_<int, short>, cvt_<int, int>, cvt_<int, float>, cvt_<int, double>, 0},

					            {cvt_<float, uchar>, cvt_<float, schar>, cvt_<float, ushort>,
					            cvt_<float, short>, cvt_<float, int>, cvt_<float, float>, cvt_<float, double>, 0},

					            {cvt_<double, uchar>, cvt_<double, schar>, cvt_<double, ushort>,
					            cvt_<double, short>, cvt_<double, int>, cvt_<double, float>, cvt_<double, double>, 0},

					            {0,0,0,0,0,0,0,0}
				            };

				            CvtFunc func = tab[sdepth][ddepth];
				            if (func == 0)
                                cv::gpu::error("Operation \'ConvertTo\' doesn't supported on your GPU model", __FILE__, __LINE__);
				            func(src, dst, width, height, alpha, beta);
			            }
		} // namespace impl		
	} // namespace gpu
} // namespace cv
