#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <hipfft/hipfft.h>
#include "internal_shared.hpp"

#include <iostream>
using namespace std;

using namespace cv::gpu;

namespace cv { namespace gpu { namespace imgproc {


texture<unsigned char, 2> imageTex_8U;
texture<unsigned char, 2> templTex_8U;


__global__ void matchTemplateNaiveKernel_8U_SQDIFF(int w, int h, 
                                                DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;
        float delta;

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                delta = (float)tex2D(imageTex_8U, x + j, y + i) - 
                        (float)tex2D(templTex_8U, j, i);
                sum += delta * delta;
            }
        }

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_8U_SQDIFF(const DevMem2D image, const DevMem2D templ,
                                DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
            divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
    hipBindTexture2D(0, imageTex_8U, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_8U, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_8U.filterMode = hipFilterModePoint;
    templTex_8U.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_8U_SQDIFF<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_8U));
    cudaSafeCall(hipUnbindTexture(templTex_8U));
}


texture<float, 2> imageTex_32F;
texture<float, 2> templTex_32F;


__global__ void matchTemplateNaiveKernel_32F_SQDIFF(int w, int h, 
                                                    DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;
        float delta;

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                delta = tex2D(imageTex_32F, x + j, y + i) - 
                        tex2D(templTex_32F, j, i);
                sum += delta * delta;
            }
        }

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_32F_SQDIFF(const DevMem2D image, const DevMem2D templ,
                                DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
            divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(0, imageTex_32F, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_32F, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_8U.filterMode = hipFilterModePoint;
    templTex_8U.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_32F_SQDIFF<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_32F));
    cudaSafeCall(hipUnbindTexture(templTex_32F));
}


__global__ void multiplyAndNormalizeSpectsKernel(
        int n, float scale, const hipfftComplex* a, 
        const hipfftComplex* b, hipfftComplex* c)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;    
    if (x < n) 
    {
        hipfftComplex v = hipCmulf(a[x], hipConjf(b[x]));
        c[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
    }
}


void multiplyAndNormalizeSpects(int n, float scale, const hipfftComplex* a, 
                                const hipfftComplex* b, hipfftComplex* c)
{
    dim3 threads(256);
    dim3 grid(divUp(n, threads.x));
    multiplyAndNormalizeSpectsKernel<<<grid, threads>>>(n, scale, a, b, c);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_SQDIFF(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sq = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = image_sq - 2.f * ccorr + templ_sqsum;
    }
}


void matchTemplatePrepared_8U_SQDIFF(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_SQDIFF<<<grid, threads>>>(
            w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_SQDIFF_NORMED(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sq = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = (image_sq - 2.f * ccorr + templ_sqsum) * 
                           rsqrtf(image_sq * templ_sqsum);
    }
}


void matchTemplatePrepared_8U_SQDIFF_NORMED(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_SQDIFF_NORMED<<<grid, threads>>>(
            w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_CCOEFF(
        int w, int h, float templ_sum_scale, 
        const PtrStep_<unsigned int> image_sum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float ccorr = result.ptr(y)[x];
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        result.ptr(y)[x] = ccorr - image_sum_ * templ_sum_scale;
    }
}


void matchTemplatePrepared_8U_CCOEFF(
        int w, int h, const DevMem2D_<unsigned int> image_sum,
        unsigned int templ_sum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_CCOEFF<<<grid, threads>>>(
            w, h, (float)templ_sum / (w * h), image_sum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_CCOEFF_NORMED(
        int w, int h, float weight, 
        float templ_sum_scale, float templ_sqsum_scale,
        const PtrStep_<unsigned int> image_sum, 
        const PtrStep_<unsigned long long> image_sqsum,
        DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float ccorr = result.ptr(y)[x];
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        result.ptr(y)[x] = (ccorr - image_sum_ * templ_sum_scale) * 
                           rsqrtf(templ_sqsum_scale * (image_sqsum_ - weight * image_sum_ * image_sum_));
    }
}


void matchTemplatePrepared_8U_CCOEFF_NORMED(
            int w, int h, const DevMem2D_<unsigned int> image_sum, 
            const DevMem2D_<unsigned long long> image_sqsum,
            unsigned int templ_sum, unsigned int templ_sqsum,
            DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    float weight = 1.f / (w * h);
    float templ_sum_scale = templ_sum * weight;
    float templ_sqsum_scale = templ_sqsum - templ_sum * templ_sum * weight;
    matchTemplatePreparedKernel_8U_CCOEFF_NORMED<<<grid, threads>>>(
            w, h, weight, templ_sum_scale, templ_sqsum_scale, 
            image_sum, image_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void normalizeKernel_8U(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sq = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        result.ptr(y)[x] *= rsqrtf(image_sq * templ_sqsum);
    }
}


void normalize_8U(int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
                  unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    normalizeKernel_8U<<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


}}}
