#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <hipfft/hipfft.h>
#include "internal_shared.hpp"
#include "opencv2/gpu/device/vecmath.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace imgproc {


__device__ float sum(float v) { return v; }
__device__ float sum(float2 v) { return v.x + v.y; }
__device__ float sum(float3 v) { return v.x + v.y + v.z; }
__device__ float sum(float4 v) { return v.x + v.y + v.z + v.w; }

__device__ float first(float v) { return v; }
__device__ float first(float2 v) { return v.x; }
__device__ float first(float3 v) { return v.x; }
__device__ float first(float4 v) { return v.x; }

__device__ float mul(float a, float b) { return a * b; }
__device__ float2 mul(float2 a, float2 b) { return make_float2(a.x * b.x, a.y * b.y); }
__device__ float3 mul(float3 a, float3 b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
__device__ float4 mul(float4 a, float4 b) { return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w); }

__device__ float mul(uchar a, uchar b) { return a * b; }
__device__ float2 mul(uchar2 a, uchar2 b) { return make_float2(a.x * b.x, a.y * b.y); }
__device__ float3 mul(uchar3 a, uchar3 b) { return make_float3(a.x * b.x, a.y * b.y, a.z * b.z); }
__device__ float4 mul(uchar4 a, uchar4 b) { return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w); }

__device__ float sub(float a, float b) { return a - b; }
__device__ float2 sub(float2 a, float2 b) { return make_float2(a.x - b.x, a.y - b.y); }
__device__ float3 sub(float3 a, float3 b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
__device__ float4 sub(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }

__device__ float sub(uchar a, uchar b) { return a - b; }
__device__ float2 sub(uchar2 a, uchar2 b) { return make_float2(a.x - b.x, a.y - b.y); }
__device__ float3 sub(uchar3 a, uchar3 b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
__device__ float4 sub(uchar4 a, uchar4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }


template <typename T, int cn>
__global__ void matchTemplateNaiveKernel_CCORR(
        int w, int h, const PtrStep image, const PtrStep templ, 
        DevMem2Df result)
{
    typedef typename TypeVec<T, cn>::vec_t Type;
    typedef typename TypeVec<float, cn>::vec_t Typef;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        Typef res = VecTraits<Typef>::all(0);

        for (int i = 0; i < h; ++i)
        {
            const Type* image_ptr = (const Type*)image.ptr(y + i);
            const Type* templ_ptr = (const Type*)templ.ptr(i);
            for (int j = 0; j < w; ++j)
                res = res + mul(image_ptr[x + j], templ_ptr[j]);
        }

        result.ptr(y)[x] = sum(res);
    }
}


void matchTemplateNaive_CCORR_32F(const DevMem2D image, const DevMem2D templ,
                                  DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    switch (cn)
    {
    case 1:
        matchTemplateNaiveKernel_CCORR<float, 1><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 2:
        matchTemplateNaiveKernel_CCORR<float, 2><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 3:
        matchTemplateNaiveKernel_CCORR<float, 3><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 4:
        matchTemplateNaiveKernel_CCORR<float, 4><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


void matchTemplateNaive_CCORR_8U(const DevMem2D image, const DevMem2D templ,
                                 DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    switch (cn)
    {
    case 1:
        matchTemplateNaiveKernel_CCORR<uchar, 1><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 2:
        matchTemplateNaiveKernel_CCORR<uchar, 2><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 3:
        matchTemplateNaiveKernel_CCORR<uchar, 3><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 4:
        matchTemplateNaiveKernel_CCORR<uchar, 4><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


template <typename T, int cn>
__global__ void matchTemplateNaiveKernel_SQDIFF(
        int w, int h, const PtrStep image, const PtrStep templ, 
        DevMem2Df result)
{
    typedef typename TypeVec<T, cn>::vec_t Type;
    typedef typename TypeVec<float, cn>::vec_t Typef;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        Typef res = VecTraits<Typef>::all(0);
        Typef delta;

        for (int i = 0; i < h; ++i)
        {
            const Type* image_ptr = (const Type*)image.ptr(y + i);
            const Type* templ_ptr = (const Type*)templ.ptr(i);
            for (int j = 0; j < w; ++j)
            {
                delta = sub(image_ptr[x + j], templ_ptr[j]);
                res = res + delta * delta;
            }
        }

        result.ptr(y)[x] = sum(res);
    }
}


void matchTemplateNaive_SQDIFF_32F(const DevMem2D image, const DevMem2D templ,
                                   DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    switch (cn)
    {
    case 1:
        matchTemplateNaiveKernel_SQDIFF<float, 1><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 2:
        matchTemplateNaiveKernel_SQDIFF<float, 2><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 3:
        matchTemplateNaiveKernel_SQDIFF<float, 3><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 4:
        matchTemplateNaiveKernel_SQDIFF<float, 4><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


void matchTemplateNaive_SQDIFF_8U(const DevMem2D image, const DevMem2D templ,
                                  DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    switch (cn)
    {
    case 1:
        matchTemplateNaiveKernel_SQDIFF<uchar, 1><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 2:
        matchTemplateNaiveKernel_SQDIFF<uchar, 2><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 3:
        matchTemplateNaiveKernel_SQDIFF<uchar, 3><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    case 4:
        matchTemplateNaiveKernel_SQDIFF<uchar, 4><<<grid, threads>>>(
                templ.cols, templ.rows, image, templ, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void multiplyAndNormalizeSpectsKernel(
        int n, float scale, const hipfftComplex* a, 
        const hipfftComplex* b, hipfftComplex* c)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;    
    if (x < n) 
    {
        hipfftComplex v = hipCmulf(a[x], hipConjf(b[x]));
        c[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
    }
}


void multiplyAndNormalizeSpects(int n, float scale, const hipfftComplex* a, 
                                const hipfftComplex* b, hipfftComplex* c)
{
    dim3 threads(256);
    dim3 grid(divUp(n, threads.x));
    multiplyAndNormalizeSpectsKernel<<<grid, threads>>>(n, scale, a, b, c);
    cudaSafeCall(hipDeviceSynchronize());
}


template <int cn>
__global__ void matchTemplatePreparedKernel_SQDIFF_8U(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = image_sqsum_ - 2.f * ccorr + templ_sqsum;
    }
}


void matchTemplatePrepared_SQDIFF_8U(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    switch (cn)
    {
    case 1:
        matchTemplatePreparedKernel_SQDIFF_8U<1><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 2:
        matchTemplatePreparedKernel_SQDIFF_8U<2><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 3:
        matchTemplatePreparedKernel_SQDIFF_8U<3><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 4:
        matchTemplatePreparedKernel_SQDIFF_8U<4><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


template <int cn>
__global__ void matchTemplatePreparedKernel_SQDIFF_NORMED_8U(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = min(1.f, (image_sqsum_ - 2.f * ccorr + templ_sqsum) * 
                           rsqrtf(image_sqsum_ * templ_sqsum));
    }
}


void matchTemplatePrepared_SQDIFF_NORMED_8U(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    switch (cn)
    {
    case 1:
        matchTemplatePreparedKernel_SQDIFF_NORMED_8U<1><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 2:
        matchTemplatePreparedKernel_SQDIFF_NORMED_8U<2><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 3:
        matchTemplatePreparedKernel_SQDIFF_NORMED_8U<3><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    case 4:
        matchTemplatePreparedKernel_SQDIFF_NORMED_8U<4><<<grid, threads>>>(
                w, h, image_sqsum, templ_sqsum, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_CCOFF_8U(
        int w, int h, float templ_sum_scale, 
        const PtrStep_<unsigned int> image_sum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = ccorr - image_sum_ * templ_sum_scale;
    }
}


void matchTemplatePrepared_CCOFF_8U(
        int w, int h, const DevMem2D_<unsigned int> image_sum,
        unsigned int templ_sum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_CCOFF_8U<<<grid, threads>>>(
            w, h, (float)templ_sum / (w * h), image_sum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_CCOFF_8UC2(
        int w, int h, float templ_sum_scale_r, float templ_sum_scale_g,
        const PtrStep_<unsigned int> image_sum_r,
        const PtrStep_<unsigned int> image_sum_g,
        DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sum_r_ = (float)(
                (image_sum_r.ptr(y + h)[x + w] - image_sum_r.ptr(y)[x + w]) -
                (image_sum_r.ptr(y + h)[x] - image_sum_r.ptr(y)[x]));
        float image_sum_g_ = (float)(
                (image_sum_g.ptr(y + h)[x + w] - image_sum_g.ptr(y)[x + w]) -
                (image_sum_g.ptr(y + h)[x] - image_sum_g.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = ccorr - image_sum_r_ * templ_sum_scale_r 
                                 - image_sum_g_ * templ_sum_scale_g;
    }
}


void matchTemplatePrepared_CCOFF_8UC2(
        int w, int h, 
        const DevMem2D_<unsigned int> image_sum_r, 
        const DevMem2D_<unsigned int> image_sum_g,
        unsigned int templ_sum_r, unsigned int templ_sum_g, 
        DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_CCOFF_8UC2<<<grid, threads>>>(
            w, h, (float)templ_sum_r / (w * h), (float)templ_sum_g / (w * h),
            image_sum_r, image_sum_g, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_CCOFF_NORMED_8U(
        int w, int h, float weight, 
        float templ_sum_scale, float templ_sqsum_scale,
        const PtrStep_<unsigned int> image_sum, 
        const PtrStep_<unsigned long long> image_sqsum,
        DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float ccorr = result.ptr(y)[x];
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        result.ptr(y)[x] = min(1.f, (ccorr - image_sum_ * templ_sum_scale) * 
                           rsqrtf(templ_sqsum_scale * (image_sqsum_ - weight * image_sum_ * image_sum_)));
    }
}


void matchTemplatePrepared_CCOFF_NORMED_8U(
            int w, int h, const DevMem2D_<unsigned int> image_sum, 
            const DevMem2D_<unsigned long long> image_sqsum,
            unsigned int templ_sum, unsigned int templ_sqsum,
            DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    float weight = 1.f / (w * h);
    float templ_sum_scale = templ_sum * weight;
    float templ_sqsum_scale = templ_sqsum - templ_sum * templ_sum * weight;
    matchTemplatePreparedKernel_CCOFF_NORMED_8U<<<grid, threads>>>(
            w, h, weight, templ_sum_scale, templ_sqsum_scale, 
            image_sum, image_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


template <int cn>
__global__ void normalizeKernel_8U(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[(x + w) * cn] - image_sqsum.ptr(y)[(x + w) * cn]) -
                (image_sqsum.ptr(y + h)[x * cn] - image_sqsum.ptr(y)[x * cn]));
        result.ptr(y)[x] = min(1.f, result.ptr(y)[x] * rsqrtf(image_sqsum_ * templ_sqsum));
    }
}


void normalize_8U(int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
                  unsigned int templ_sqsum, DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    switch (cn)
    {
    case 1:
        normalizeKernel_8U<1><<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
        break;
    case 2:
        normalizeKernel_8U<2><<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
        break;
    case 3:
        normalizeKernel_8U<3><<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
        break;
    case 4:
        normalizeKernel_8U<4><<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


template <int cn>
__global__ void extractFirstChannel_32F(const PtrStep image, DevMem2Df result)
{
    typedef typename TypeVec<float, cn>::vec_t Typef;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        Typef val = ((const Typef*)image.ptr(y))[x];
        result.ptr(y)[x] = first(val);
    }
}


void extractFirstChannel_32F(const DevMem2D image, DevMem2Df result, int cn)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    switch (cn)
    {
    case 1:
        extractFirstChannel_32F<1><<<grid, threads>>>(image, result);
        break;
    case 2:
        extractFirstChannel_32F<2><<<grid, threads>>>(image, result);
        break;
    case 3:
        extractFirstChannel_32F<3><<<grid, threads>>>(image, result);
        break;
    case 4:
        extractFirstChannel_32F<4><<<grid, threads>>>(image, result);
        break;
    }
    cudaSafeCall(hipDeviceSynchronize());
}


}}}

