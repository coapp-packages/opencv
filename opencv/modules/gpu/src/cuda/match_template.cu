#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <hipfft/hipfft.h>
#include "internal_shared.hpp"

using namespace cv::gpu;

namespace cv { namespace gpu { namespace imgproc {

texture<unsigned char, 2> imageTex_8U_CCORR;
texture<unsigned char, 2> templTex_8U_CCORR;


__global__ void matchTemplateNaiveKernel_8U_CCORR(int w, int h, 
                                                  DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;

        for (int i = 0; i < h; ++i)
            for (int j = 0; j < w; ++j)
                sum += (float)tex2D(imageTex_8U_CCORR, x + j, y + i) * 
                       (float)tex2D(templTex_8U_CCORR, j, i);

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_8U_CCORR(const DevMem2D image, const DevMem2D templ,
                                 DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
              divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
    hipBindTexture2D(0, imageTex_8U_CCORR, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_8U_CCORR, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_8U_CCORR.filterMode = hipFilterModePoint;
    templTex_8U_CCORR.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_8U_CCORR<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_8U_CCORR));
    cudaSafeCall(hipUnbindTexture(templTex_8U_CCORR));
}


texture<float, 2> imageTex_32F_CCORR;
texture<float, 2> templTex_32F_CCORR;


__global__ void matchTemplateNaiveKernel_32F_CCORR(int w, int h, 
                                                   DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;

        for (int i = 0; i < h; ++i)
            for (int j = 0; j < w; ++j)
                sum += tex2D(imageTex_32F_CCORR, x + j, y + i) * 
                       tex2D(templTex_32F_CCORR, j, i);

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_32F_CCORR(const DevMem2D image, const DevMem2D templ,
                                  DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
              divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(0, imageTex_32F_CCORR, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_32F_CCORR, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_32F_CCORR.filterMode = hipFilterModePoint;
    templTex_32F_CCORR.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_32F_CCORR<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_32F_CCORR));
    cudaSafeCall(hipUnbindTexture(templTex_32F_CCORR));
}


texture<unsigned char, 2> imageTex_8U_SQDIFF;
texture<unsigned char, 2> templTex_8U_SQDIFF;


__global__ void matchTemplateNaiveKernel_8U_SQDIFF(int w, int h, 
                                                   DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;
        float delta;

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                delta = (float)tex2D(imageTex_8U_SQDIFF, x + j, y + i) - 
                        (float)tex2D(templTex_8U_SQDIFF, j, i);
                sum += delta * delta;
            }
        }

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_8U_SQDIFF(const DevMem2D image, const DevMem2D templ,
                                  DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
              divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
    hipBindTexture2D(0, imageTex_8U_SQDIFF, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_8U_SQDIFF, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_8U_SQDIFF.filterMode = hipFilterModePoint;
    templTex_8U_SQDIFF.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_8U_SQDIFF<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_8U_SQDIFF));
    cudaSafeCall(hipUnbindTexture(templTex_8U_SQDIFF));
}


texture<float, 2> imageTex_32F_SQDIFF;
texture<float, 2> templTex_32F_SQDIFF;


__global__ void matchTemplateNaiveKernel_32F_SQDIFF(int w, int h, 
                                                    DevMem2Df result)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float sum = 0.f;
        float delta;

        for (int i = 0; i < h; ++i)
        {
            for (int j = 0; j < w; ++j)
            {
                delta = tex2D(imageTex_32F_SQDIFF, x + j, y + i) - 
                        tex2D(templTex_32F_SQDIFF, j, i);
                sum += delta * delta;
            }
        }

        result.ptr(y)[x] = sum;
    }
}


void matchTemplateNaive_32F_SQDIFF(const DevMem2D image, const DevMem2D templ,
                                   DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(image.cols - templ.cols + 1, threads.x), 
              divUp(image.rows - templ.rows + 1, threads.y));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(0, imageTex_32F_SQDIFF, image.data, desc, image.cols, image.rows, image.step);
    hipBindTexture2D(0, templTex_32F_SQDIFF, templ.data, desc, templ.cols, templ.rows, templ.step);
    imageTex_8U_SQDIFF.filterMode = hipFilterModePoint;
    templTex_8U_SQDIFF.filterMode = hipFilterModePoint;

    matchTemplateNaiveKernel_32F_SQDIFF<<<grid, threads>>>(templ.cols, templ.rows, result);
    cudaSafeCall(hipDeviceSynchronize());
    cudaSafeCall(hipUnbindTexture(imageTex_32F_SQDIFF));
    cudaSafeCall(hipUnbindTexture(templTex_32F_SQDIFF));
}


__global__ void multiplyAndNormalizeSpectsKernel(
        int n, float scale, const hipfftComplex* a, 
        const hipfftComplex* b, hipfftComplex* c)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;    
    if (x < n) 
    {
        hipfftComplex v = hipCmulf(a[x], hipConjf(b[x]));
        c[x] = make_hipFloatComplex(hipCrealf(v) * scale, hipCimagf(v) * scale);
    }
}


void multiplyAndNormalizeSpects(int n, float scale, const hipfftComplex* a, 
                                const hipfftComplex* b, hipfftComplex* c)
{
    dim3 threads(256);
    dim3 grid(divUp(n, threads.x));
    multiplyAndNormalizeSpectsKernel<<<grid, threads>>>(n, scale, a, b, c);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_SQDIFF(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = image_sqsum_ - 2.f * ccorr + templ_sqsum;
    }
}


void matchTemplatePrepared_8U_SQDIFF(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_SQDIFF<<<grid, threads>>>(
            w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_SQDIFF_NORMED(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        float ccorr = result.ptr(y)[x];
        result.ptr(y)[x] = (image_sqsum_ - 2.f * ccorr + templ_sqsum) * 
                           rsqrtf(image_sqsum_ * templ_sqsum);
    }
}


void matchTemplatePrepared_8U_SQDIFF_NORMED(
        int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_SQDIFF_NORMED<<<grid, threads>>>(
            w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_CCOEFF(
        int w, int h, float templ_sum_scale, 
        const PtrStep_<unsigned int> image_sum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float ccorr = result.ptr(y)[x];
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        result.ptr(y)[x] = ccorr - image_sum_ * templ_sum_scale;
    }
}


void matchTemplatePrepared_8U_CCOEFF(
        int w, int h, const DevMem2D_<unsigned int> image_sum,
        unsigned int templ_sum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    matchTemplatePreparedKernel_8U_CCOEFF<<<grid, threads>>>(
            w, h, (float)templ_sum / (w * h), image_sum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void matchTemplatePreparedKernel_8U_CCOEFF_NORMED(
        int w, int h, float weight, 
        float templ_sum_scale, float templ_sqsum_scale,
        const PtrStep_<unsigned int> image_sum, 
        const PtrStep_<unsigned long long> image_sqsum,
        DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float ccorr = result.ptr(y)[x];
        float image_sum_ = (float)(
                (image_sum.ptr(y + h)[x + w] - image_sum.ptr(y)[x + w]) -
                (image_sum.ptr(y + h)[x] - image_sum.ptr(y)[x]));
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        result.ptr(y)[x] = (ccorr - image_sum_ * templ_sum_scale) * 
                           rsqrtf(templ_sqsum_scale * (image_sqsum_ - weight * image_sum_ * image_sum_));
    }
}


void matchTemplatePrepared_8U_CCOEFF_NORMED(
            int w, int h, const DevMem2D_<unsigned int> image_sum, 
            const DevMem2D_<unsigned long long> image_sqsum,
            unsigned int templ_sum, unsigned int templ_sqsum,
            DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));

    float weight = 1.f / (w * h);
    float templ_sum_scale = templ_sum * weight;
    float templ_sqsum_scale = templ_sqsum - templ_sum * templ_sum * weight;
    matchTemplatePreparedKernel_8U_CCOEFF_NORMED<<<grid, threads>>>(
            w, h, weight, templ_sum_scale, templ_sqsum_scale, 
            image_sum, image_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void normalizeKernel_8U(
        int w, int h, const PtrStep_<unsigned long long> image_sqsum, 
        unsigned int templ_sqsum, DevMem2Df result)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < result.cols && y < result.rows)
    {
        float image_sqsum_ = (float)(
                (image_sqsum.ptr(y + h)[x + w] - image_sqsum.ptr(y)[x + w]) -
                (image_sqsum.ptr(y + h)[x] - image_sqsum.ptr(y)[x]));
        result.ptr(y)[x] *= rsqrtf(image_sqsum_ * templ_sqsum);
    }
}


void normalize_8U(int w, int h, const DevMem2D_<unsigned long long> image_sqsum, 
                  unsigned int templ_sqsum, DevMem2Df result)
{
    dim3 threads(32, 8);
    dim3 grid(divUp(result.cols, threads.x), divUp(result.rows, threads.y));
    normalizeKernel_8U<<<grid, threads>>>(w, h, image_sqsum, templ_sqsum, result);
    cudaSafeCall(hipDeviceSynchronize());
}


}}}
