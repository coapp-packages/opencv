#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "internal_shared.hpp"
#include "opencv2/gpu/device/limits_gpu.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

namespace cv { namespace gpu { namespace bfmatcher
{
///////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// General funcs //////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////

    ///////////////////////////////////////////////////////////////////////////////
    // Mask strategy

    class SingleMask
    {
    public:
        explicit SingleMask(const PtrStep& mask_) : mask(mask_) {}
        
        __device__ bool operator()(int queryIdx, int trainIdx) const
        {            
            return mask.ptr(queryIdx)[trainIdx] != 0;
        }

    private:
        PtrStep mask;
    };

    class MaskCollection
    {
    public:
        explicit MaskCollection(PtrStep* maskCollection_) : maskCollection(maskCollection_) {}

        __device__ void nextMask()
        {
            curMask = *maskCollection++;
        }
        
        __device__ bool operator()(int queryIdx, int trainIdx) const
        {            
            return curMask.data == 0 || curMask.ptr(queryIdx)[trainIdx] != 0;
        }

    private:
        PtrStep* maskCollection;
        PtrStep curMask;
    };

    class WithOutMask
    {
    public:
        __device__ void nextMask()
        {
        }
        __device__ bool operator()(int queryIdx, int trainIdx) const
        {
            return true;
        }
    };

    ///////////////////////////////////////////////////////////////////////////////
    // Reduce Sum
    
    template <int BLOCK_DIM_X> __device__ void reduceSum(float* sdiff_row, float& mySum);

    template <> __device__ void reduceSum<16>(float* sdiff_row, float& mySum)
    {
        volatile float* smem = sdiff_row;

        smem[threadIdx.x] = mySum;
        
        if (threadIdx.x < 8) 
        {
            smem[threadIdx.x] = mySum += smem[threadIdx.x + 8]; 
            smem[threadIdx.x] = mySum += smem[threadIdx.x + 4]; 
            smem[threadIdx.x] = mySum += smem[threadIdx.x + 2];
            smem[threadIdx.x] = mySum += smem[threadIdx.x + 1];  
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Distance

    class L1Dist
    {
    public:
        __device__ L1Dist() : mySum(0.0f) {}

        __device__ void reduceIter(float val1, float val2)
        {
            mySum += fabs(val1 - val2);
        }

        template <int BLOCK_DIM_X>
        __device__ void reduceAll(float* sdiff_row)
        {
            reduceSum<BLOCK_DIM_X>(sdiff_row, mySum);
        }

        __device__ operator float() const
        {
            return mySum;
        }

    private:
        float mySum;
    };

    class L2Dist
    {
    public:
        __device__ L2Dist() : mySum(0.0f) {}

        __device__ void reduceIter(float val1, float val2)
        {
            float reg = val1 - val2;
            mySum += reg * reg;
        }

        template <int BLOCK_DIM_X>
        __device__ void reduceAll(float* sdiff_row)
        {
            reduceSum<BLOCK_DIM_X>(sdiff_row, mySum);
        }

        __device__ operator float() const
        {
            return sqrtf(mySum);
        }

    private:
        float mySum;
    };
    
    ///////////////////////////////////////////////////////////////////////////////
    // reduceDescDiff

    template <int BLOCK_DIM_X, typename Dist, typename T> 
    __device__ void reduceDescDiff(const T* queryDescs, const T* trainDescs, int desc_len, Dist& dist, 
        float* sdiff_row)
    {
        for (int i = threadIdx.x; i < desc_len; i += BLOCK_DIM_X)
            dist.reduceIter(queryDescs[i], trainDescs[i]);

        dist.reduceAll<BLOCK_DIM_X>(sdiff_row);
    }

///////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// Match //////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////
    
    ///////////////////////////////////////////////////////////////////////////////
    // loadDescsVals

    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN, typename T> 
    __device__ void loadDescsVals(const T* descs, int desc_len, float* queryVals, float* smem)
    {
        const int tid = threadIdx.y * blockDim.x + threadIdx.x;

        if (tid < desc_len)
        {
            smem[tid] = (float)descs[tid];
        }
        __syncthreads();

        #pragma unroll
        for (int i = threadIdx.x; i < MAX_DESCRIPTORS_LEN; i += BLOCK_DIM_X)
        {
            *queryVals = smem[i];
            ++queryVals;
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // reduceDescDiffCached

    template <int N> struct UnrollDescDiff
    {
        template <typename Dist, typename T>
        static __device__ void calcCheck(const float* queryVals, const T* trainDescs, int desc_len, 
            Dist& dist, int ind)
        {
            if (ind < desc_len)
            {
                dist.reduceIter(*queryVals, trainDescs[ind]);

                ++queryVals;

                UnrollDescDiff<N - 1>::calcCheck(queryVals, trainDescs, desc_len, dist, ind + blockDim.x);
            }
        }

        template <typename Dist, typename T>
        static __device__ void calcWithoutCheck(const float* queryVals, const T* trainDescs, Dist& dist)
        {
            dist.reduceIter(*queryVals, *trainDescs);

            ++queryVals;
            trainDescs += blockDim.x;

            UnrollDescDiff<N - 1>::calcWithoutCheck(queryVals, trainDescs, dist);
        }
    };
    template <> struct UnrollDescDiff<0>
    {
        template <typename Dist, typename T>
        static __device__ void calcCheck(const float* queryVals, const T* trainDescs, int desc_len, 
            Dist& dist, int ind)
        {
        }

        template <typename Dist, typename T>
        static __device__ void calcWithoutCheck(const float* queryVals, const T* trainDescs, Dist& dist)
        {
        }
    };

    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN, bool WITH_OUT_CHECK> struct DescDiffCalculator;
    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN> 
    struct DescDiffCalculator<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN, false>
    {
        template <typename Dist, typename T>
        static __device__ void calc(const float* queryVals, const T* trainDescs, int desc_len, Dist& dist)
        {
            UnrollDescDiff<MAX_DESCRIPTORS_LEN / BLOCK_DIM_X>::calcCheck(queryVals, trainDescs, desc_len, 
                dist, threadIdx.x);
        }
    };
    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN> 
    struct DescDiffCalculator<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN, true>
    {
        template <typename Dist, typename T>
        static __device__ void calc(const float* queryVals, const T* trainDescs, int desc_len, Dist& dist)
        {
            UnrollDescDiff<MAX_DESCRIPTORS_LEN / BLOCK_DIM_X>::calcWithoutCheck(queryVals, 
                trainDescs + threadIdx.x, dist);
        }
    };

    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN, bool DESC_LEN_EQ_MAX_LEN, typename Dist, typename T>
    __device__ void reduceDescDiffCached(const float* queryVals, const T* trainDescs, int desc_len, Dist& dist, 
        float* sdiff_row)
    {        
        DescDiffCalculator<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN, DESC_LEN_EQ_MAX_LEN>::calc(queryVals, 
            trainDescs, desc_len, dist);
        
        dist.reduceAll<BLOCK_DIM_X>(sdiff_row);
    }

    ///////////////////////////////////////////////////////////////////////////////
    // warpReduceMinIdxIdx

    template <int BLOCK_DIM_Y> 
    __device__ void warpReduceMinIdxIdx(float& myMin, int& myBestTrainIdx, int& myBestImgIdx, 
        volatile float* sdata, volatile int* strainIdx, volatile int* simgIdx);

    template <> 
    __device__ void warpReduceMinIdxIdx<16>(float& myMin, int& myBestTrainIdx, int& myBestImgIdx, 
        volatile float* smin, volatile int* strainIdx, volatile int* simgIdx)
    {
        const int tid = threadIdx.y * blockDim.x + threadIdx.x;

        if (tid < 8)
        {
            myMin = smin[tid];
            myBestTrainIdx = strainIdx[tid];
            myBestImgIdx = simgIdx[tid];

            float reg = smin[tid + 8];
            if (reg < myMin)
            {
                smin[tid] = myMin = reg;
                strainIdx[tid] = myBestTrainIdx = strainIdx[tid + 8];
                simgIdx[tid] = myBestImgIdx = simgIdx[tid + 8];
            }

            reg = smin[tid + 4];
            if (reg < myMin)
            {
                smin[tid] = myMin = reg;
                strainIdx[tid] = myBestTrainIdx = strainIdx[tid + 4];
                simgIdx[tid] = myBestImgIdx = simgIdx[tid + 4];
            }
        
            reg = smin[tid + 2];
            if (reg < myMin)
            {
                smin[tid] = myMin = reg;
                strainIdx[tid] = myBestTrainIdx = strainIdx[tid + 2];
                simgIdx[tid] = myBestImgIdx = simgIdx[tid + 2];
            }
        
            reg = smin[tid + 1];
            if (reg < myMin)
            {
                smin[tid] = myMin = reg;
                strainIdx[tid] = myBestTrainIdx = strainIdx[tid + 1];
                simgIdx[tid] = myBestImgIdx = simgIdx[tid + 1];
            }
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // findBestMatch

    template <int BLOCK_DIM_Y>
    __device__ void findBestMatch(float& myMin, int& myBestTrainIdx, int& myBestImgIdx, 
        float* smin, int* strainIdx, int* simgIdx)
    {
        if (threadIdx.x == 0)
        {
            smin[threadIdx.y] = myMin;
            strainIdx[threadIdx.y] = myBestTrainIdx;
            simgIdx[threadIdx.y] = myBestImgIdx;
        }
        __syncthreads();

        warpReduceMinIdxIdx<BLOCK_DIM_Y>(myMin, myBestTrainIdx, myBestImgIdx, smin, strainIdx, simgIdx);
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // ReduceDescCalculator

    template <int BLOCK_DIM_X, typename T>
    class ReduceDescCalculatorSimple
    {
    public:
        __device__ void prepare(const T* queryDescs_, int, float*)
        {
            queryDescs = queryDescs_;
        }

        template <typename Dist>
        __device__ void calc(const T* trainDescs, int desc_len, Dist& dist, float* sdiff_row) const
        {
            reduceDescDiff<BLOCK_DIM_X>(queryDescs, trainDescs, desc_len, dist, sdiff_row);
        }

    private:
        const T* queryDescs;
    };

    template <int BLOCK_DIM_X, int MAX_DESCRIPTORS_LEN, bool DESC_LEN_EQ_MAX_LEN, typename T>
    class ReduceDescCalculatorCached
    {
    public:
        __device__ void prepare(const T* queryDescs, int desc_len, float* smem)
        {
            loadDescsVals<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN>(queryDescs, desc_len, queryVals, smem);
        }

        template <typename Dist>
        __device__ void calc(const T* trainDescs, int desc_len, Dist& dist, float* sdiff_row) const
        {
            reduceDescDiffCached<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN, DESC_LEN_EQ_MAX_LEN>(queryVals, trainDescs, 
                desc_len, dist, sdiff_row);
        }

    private:
        float queryVals[MAX_DESCRIPTORS_LEN / BLOCK_DIM_X];
    };
    
    ///////////////////////////////////////////////////////////////////////////////
    // matchDescs loop

    template <typename Dist, typename ReduceDescCalculator, typename T, typename Mask>
    __device__ void matchDescs(int queryIdx, int imgIdx, const DevMem2D_<T>& trainDescs_,  
        const Mask& m, const ReduceDescCalculator& reduceDescCalc,
        float& myMin, int& myBestTrainIdx, int& myBestImgIdx, float* sdiff_row)
    {
        for (int trainIdx = threadIdx.y; trainIdx < trainDescs_.rows; trainIdx += blockDim.y)
        {
            if (m(queryIdx, trainIdx))
            {
                const T* trainDescs = trainDescs_.ptr(trainIdx);

                Dist dist;

                reduceDescCalc.calc(trainDescs, trainDescs_.cols, dist, sdiff_row);

                if (threadIdx.x == 0)
                {
                    if (dist < myMin)
                    {
                        myMin = dist;
                        myBestTrainIdx = trainIdx;
                        myBestImgIdx = imgIdx;
                    }
                }
            }
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Train collection loop strategy

    template <typename T>
    class SingleTrain
    {
    public:
        explicit SingleTrain(const DevMem2D_<T>& trainDescs_) : trainDescs(trainDescs_)
        {
        }

        template <typename Dist, typename ReduceDescCalculator, typename Mask>
        __device__ void loop(int queryIdx, Mask& m, const ReduceDescCalculator& reduceDescCalc, 
            float& myMin, int& myBestTrainIdx, int& myBestImgIdx, float* sdiff_row) const
        {
            matchDescs<Dist>(queryIdx, 0, trainDescs, m, reduceDescCalc, 
                myMin, myBestTrainIdx, myBestImgIdx, sdiff_row);
        }

        __device__ int desc_len() const
        {
            return trainDescs.cols;
        }

    private:
        DevMem2D_<T> trainDescs;
    };

    template <typename T>
    class TrainCollection
    {
    public:
        TrainCollection(const DevMem2D_<T>* trainCollection_, int nImg_, int desclen_) : 
            trainCollection(trainCollection_), nImg(nImg_), desclen(desclen_)
        {
        }

        template <typename Dist, typename ReduceDescCalculator, typename Mask>
        __device__ void loop(int queryIdx, Mask& m, const ReduceDescCalculator& reduceDescCalc, 
            float& myMin, int& myBestTrainIdx, int& myBestImgIdx, float* sdiff_row) const
        {
            for (int imgIdx = 0; imgIdx < nImg; ++imgIdx)
            {
                DevMem2D_<T> trainDescs = trainCollection[imgIdx];
                m.nextMask();
                matchDescs<Dist>(queryIdx, imgIdx, trainDescs, m, reduceDescCalc, 
                    myMin, myBestTrainIdx, myBestImgIdx, sdiff_row);
            }
        }

        __device__ int desc_len() const
        {
            return desclen;
        }

    private:
        const DevMem2D_<T>* trainCollection;
        int nImg;
        int desclen;
    };

    ///////////////////////////////////////////////////////////////////////////////
    // Match kernel

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename ReduceDescCalculator, typename Dist, typename T, 
        typename Train, typename Mask>
    __global__ void match(const PtrStep_<T> queryDescs_, const Train train, const Mask mask, 
        int* trainIdx, int* imgIdx, float* distance)
    {
        __shared__ float smem[BLOCK_DIM_X * BLOCK_DIM_Y];        
        
        const int queryIdx = blockIdx.x;
        
        int myBestTrainIdx = -1;
        int myBestImgIdx = -1;
        float myMin = numeric_limits_gpu<float>::max();

        {
            float* sdiff_row = smem + BLOCK_DIM_X * threadIdx.y;

            Mask m = mask;

            ReduceDescCalculator reduceDescCalc;

            reduceDescCalc.prepare(queryDescs_.ptr(queryIdx), train.desc_len(), smem);
        
            train.template loop<Dist>(queryIdx, m, reduceDescCalc, myMin, myBestTrainIdx, myBestImgIdx, sdiff_row);
        }
        __syncthreads();

        float* smin = smem;
        int* strainIdx = (int*)(smin + BLOCK_DIM_Y);
        int* simgIdx = strainIdx + BLOCK_DIM_Y;

        findBestMatch<BLOCK_DIM_Y>(myMin, myBestTrainIdx, myBestImgIdx, 
            smin, strainIdx, simgIdx);

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            imgIdx[queryIdx] = myBestImgIdx;
            trainIdx[queryIdx] = myBestTrainIdx;
            distance[queryIdx] = myMin;
        }
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Match kernel callers

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, 
        typename Train, typename Mask>
    void matchSimple_caller(const DevMem2D_<T>& queryDescs, const Train& train, 
        const Mask& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance)
    {
        StaticAssert<BLOCK_DIM_Y <= 64>::check(); // blockDimY vals must reduce by warp

        dim3 grid(queryDescs.rows, 1, 1);
        dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        match<BLOCK_DIM_X, BLOCK_DIM_Y, ReduceDescCalculatorSimple<BLOCK_DIM_X, T>, Dist, T>
            <<<grid, threads>>>(queryDescs, train, mask, trainIdx.data, 
            imgIdx.data, distance.data);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }
    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, int MAX_DESCRIPTORS_LEN, bool DESC_LEN_EQ_MAX_LEN, 
        typename Dist, typename T, typename Train, typename Mask>
    void matchCached_caller(const DevMem2D_<T>& queryDescs, const Train& train, 
        const Mask& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance)
    {
        StaticAssert<BLOCK_DIM_Y <= 64>::check();                                // blockDimY vals must reduce by warp
        StaticAssert<BLOCK_DIM_X * BLOCK_DIM_Y >= MAX_DESCRIPTORS_LEN>::check(); // block size must be greter than descriptors length
        StaticAssert<MAX_DESCRIPTORS_LEN % BLOCK_DIM_X == 0>::check();           // max descriptors length must divide to blockDimX

        dim3 grid(queryDescs.rows, 1, 1);
        dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);

        match<BLOCK_DIM_X, BLOCK_DIM_Y, 
              ReduceDescCalculatorCached<BLOCK_DIM_X, MAX_DESCRIPTORS_LEN, DESC_LEN_EQ_MAX_LEN, T>, 
              Dist, T>
              <<<grid, threads>>>(queryDescs, train, mask, trainIdx.data, 
              imgIdx.data, distance.data);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Match kernel chooser

    template <typename Dist, typename T, typename Train, typename Mask>
    void match_chooser(const DevMem2D_<T>& queryDescs, const Train& train, 
        const Mask& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance,
        bool cc_12)
    {
        if (queryDescs.cols < 64)
            matchCached_caller<16, 16, 64, false, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else if (queryDescs.cols == 64)
            matchCached_caller<16, 16, 64, true, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else if (queryDescs.cols < 128)
            matchCached_caller<16, 16, 128, false, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else if (queryDescs.cols == 128)
            matchCached_caller<16, 16, 128, true, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else if (queryDescs.cols < 256)
            matchCached_caller<16, 16, 256, false, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else if (queryDescs.cols == 256 && cc_12)
            matchCached_caller<16, 16, 256, true, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);
        else
            matchSimple_caller<16, 16, Dist>(queryDescs, train, mask, trainIdx, imgIdx, distance);

        cudaSafeCall( hipDeviceSynchronize() );
    }

    template <typename T>
    void matchSingleL1_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, 
        const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance,
        bool cc_12)
    {
        SingleTrain<T> train((DevMem2D_<T>)trainDescs);
        if (mask.data)
        {
            SingleMask m(mask);
            match_chooser<L1Dist>((DevMem2D_<T>)queryDescs, train, m, trainIdx, imgIdx, distance, cc_12);
        }
        else
        {
            match_chooser<L1Dist>((DevMem2D_<T>)queryDescs, train, WithOutMask(), trainIdx, imgIdx, distance, cc_12);
        }
    }

    template void matchSingleL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);

    template <typename T>
    void matchSingleL2_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, 
        const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, 
        bool cc_12)
    {
        SingleTrain<T> train((DevMem2D_<T>)trainDescs);
        if (mask.data)
        {
            SingleMask m(mask);
            match_chooser<L2Dist>((DevMem2D_<T>)queryDescs, train, m, trainIdx, imgIdx, distance, cc_12);
        }
        else
        {
            match_chooser<L2Dist>((DevMem2D_<T>)queryDescs, train, WithOutMask(), trainIdx, imgIdx, distance, cc_12);
        }
    }

    template void matchSingleL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchSingleL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);

    template <typename T>
    void matchCollectionL1_gpu(const DevMem2D& queryDescs, const DevMem2D& trainCollection, 
        const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, 
        const DevMem2Df& distance, bool cc_12)
    {
        TrainCollection<T> train((DevMem2D_<T>*)trainCollection.ptr(), trainCollection.cols, queryDescs.cols);
        if (maskCollection.data)
        {
            MaskCollection mask(maskCollection.data);
            match_chooser<L1Dist>((DevMem2D_<T>)queryDescs, train, mask, trainIdx, imgIdx, distance, cc_12);
        }
        else
        {
            match_chooser<L1Dist>((DevMem2D_<T>)queryDescs, train, WithOutMask(), trainIdx, imgIdx, distance, cc_12);
        }
    }

    template void matchCollectionL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);

    template <typename T>
    void matchCollectionL2_gpu(const DevMem2D& queryDescs, const DevMem2D& trainCollection, 
        const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, 
        const DevMem2Df& distance, bool cc_12)
    {
        TrainCollection<T> train((DevMem2D_<T>*)trainCollection.ptr(), trainCollection.cols, queryDescs.cols);
        if (maskCollection.data)
        {
            MaskCollection mask(maskCollection.data);
            match_chooser<L2Dist>((DevMem2D_<T>)queryDescs, train, mask, trainIdx, imgIdx, distance, cc_12);
        }
        else
        {
            match_chooser<L2Dist>((DevMem2D_<T>)queryDescs, train, WithOutMask(), trainIdx, imgIdx, distance, cc_12);
        }
    }

    template void matchCollectionL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    template void matchCollectionL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainCollection, const DevMem2D_<PtrStep>& maskCollection, const DevMem2Di& trainIdx, const DevMem2Di& imgIdx, const DevMem2Df& distance, bool cc_12);
    
///////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// Knn Match ////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////
    
    ///////////////////////////////////////////////////////////////////////////////
    // Calc distance kernel

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    __global__ void calcDistance(PtrStep_<T> queryDescs_, DevMem2D_<T> trainDescs_, Mask mask, PtrStepf distance)
    {
        __shared__ float sdiff[BLOCK_DIM_X * BLOCK_DIM_Y];

        float* sdiff_row = sdiff + BLOCK_DIM_X * threadIdx.y;
        
        const int queryIdx = blockIdx.x;
        const T* queryDescs = queryDescs_.ptr(queryIdx);

        const int trainIdx = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

        if (trainIdx < trainDescs_.rows)
        {
            const T* trainDescs = trainDescs_.ptr(trainIdx);

            float myDist = numeric_limits_gpu<float>::max();

            if (mask(queryIdx, trainIdx))
            {
                Dist dist;

                reduceDescDiff<BLOCK_DIM_X>(queryDescs, trainDescs, trainDescs_.cols, dist, sdiff_row);

                if (threadIdx.x == 0)
                    myDist = dist;
            }
            
            if (threadIdx.x == 0)
                distance.ptr(queryIdx)[trainIdx] = myDist;
        }
    }

    ///////////////////////////////////////////////////////////////////////////////
    // Calc distance kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    void calcDistance_caller(const DevMem2D_<T>& queryDescs, const DevMem2D_<T>& trainDescs, 
        const Mask& mask, const DevMem2Df& distance)
    {
        dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
        dim3 grid(queryDescs.rows, divUp(trainDescs.rows, BLOCK_DIM_Y), 1);

        calcDistance<BLOCK_DIM_X, BLOCK_DIM_Y, Dist, T><<<grid, threads>>>(
            queryDescs, trainDescs, mask, distance);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }
        
    ///////////////////////////////////////////////////////////////////////////////
    // warpReduceMinIdx

    template <int BLOCK_SIZE> 
    __device__ void warpReduceMinIdx(volatile float* sdist, volatile int* strainIdx, float& myMin, int tid)
    {
        if (tid < 32)
        {
            if (BLOCK_SIZE >= 64) 
            { 
                float reg = sdist[tid + 32];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 32];
                }
            }
            if (BLOCK_SIZE >= 32) 
            { 
                float reg = sdist[tid + 16];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 16];
                }
            }
            if (BLOCK_SIZE >= 16) 
            { 
                float reg = sdist[tid + 8];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 8];
                }
            }
            if (BLOCK_SIZE >= 8) 
            { 
                float reg = sdist[tid + 4];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 4];
                }
            }
            if (BLOCK_SIZE >= 4) 
            { 
                float reg = sdist[tid + 2];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 2];
                } 
            }
            if (BLOCK_SIZE >= 2) 
            { 
                float reg = sdist[tid + 1];

                if (reg < myMin)
                {
                    sdist[tid] = myMin = reg;
                    strainIdx[tid] = strainIdx[tid + 1];
                }
            }
        }
    }
    
    template <int BLOCK_SIZE> 
    __device__ void reduceMinIdx(const float* dist, int n, float* sdist, int* strainIdx)
    {
        const int tid = threadIdx.x;
        
        float myMin = numeric_limits_gpu<float>::max();
        int myMinIdx = -1;

        for (int i = tid; i < n; i += BLOCK_SIZE)
        {
            float reg = dist[i];
            if (reg < myMin)
            {
                myMin = reg;
                myMinIdx = i;
            }
        }

        sdist[tid] = myMin;
        strainIdx[tid] = myMinIdx;
        __syncthreads();

        if (BLOCK_SIZE >= 512 && tid < 256) 
        {
            float reg = sdist[tid + 256];

            if (reg < myMin)
            {
                sdist[tid] = myMin = reg;
                strainIdx[tid] = strainIdx[tid + 256];
            }
            __syncthreads(); 
        }
        if (BLOCK_SIZE >= 256 && tid < 128) 
        {
            float reg = sdist[tid + 128];

            if (reg < myMin)
            {
                sdist[tid] = myMin = reg;
                strainIdx[tid] = strainIdx[tid + 128];
            }
            __syncthreads(); 
        }
        if (BLOCK_SIZE >= 128 && tid < 64) 
        {
            float reg = sdist[tid + 64];

            if (reg < myMin)
            {
                sdist[tid] = myMin = reg;
                strainIdx[tid] = strainIdx[tid + 64];
            }
            __syncthreads(); 
        }
        
        warpReduceMinIdx<BLOCK_SIZE>(sdist, strainIdx, myMin, tid);
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // find knn match kernel

    template <int BLOCK_SIZE>
    __global__ void findBestMatch(DevMem2Df allDist_, int i, PtrStepi trainIdx_, PtrStepf distance_)
    {
        const int SMEM_SIZE = BLOCK_SIZE > 64 ? BLOCK_SIZE : 64;
        __shared__ float sdist[SMEM_SIZE];
        __shared__ int strainIdx[SMEM_SIZE];

        const int queryIdx = blockIdx.x;

        float* allDist = allDist_.ptr(queryIdx);
        int* trainIdx = trainIdx_.ptr(queryIdx);
        float* distance = distance_.ptr(queryIdx);

        reduceMinIdx<BLOCK_SIZE>(allDist, allDist_.cols, sdist, strainIdx);

        if (threadIdx.x == 0)
        {
            float dist = sdist[0];
            if (dist < numeric_limits_gpu<float>::max())
            {
                int bestIdx = strainIdx[0];
                allDist[bestIdx] = numeric_limits_gpu<float>::max();
                trainIdx[i] = bestIdx;
                distance[i] = dist;
            }
        }
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // find knn match kernel caller

    template <int BLOCK_SIZE>
    void findKnnMatch_caller(int knn, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist)
    {
        dim3 threads(BLOCK_SIZE, 1, 1);
        dim3 grid(trainIdx.rows, 1, 1);

        for (int i = 0; i < knn; ++i)
        {
            findBestMatch<BLOCK_SIZE><<<grid, threads>>>(allDist, i, trainIdx, distance);
            cudaSafeCall( hipGetLastError() );
        }
        
        cudaSafeCall( hipDeviceSynchronize() );
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // knn match caller

    template <typename T>
    void knnMatchL1_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn,
        const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist)
    {
        if (mask.data)
        {
            calcDistance_caller<16, 16, L1Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                SingleMask(mask), allDist);
        }
        else
        {
            calcDistance_caller<16, 16, L1Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                WithOutMask(), allDist);
        }

        findKnnMatch_caller<256>(knn, trainIdx, distance, allDist);
    }

    template void knnMatchL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);

    template <typename T>
    void knnMatchL2_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn,
        const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist)
    {
        if (mask.data)
        {
            calcDistance_caller<16, 16, L2Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                SingleMask(mask), allDist);
        }
        else
        {
            calcDistance_caller<16, 16, L2Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                WithOutMask(), allDist);
        }

        findKnnMatch_caller<256>(knn, trainIdx, distance, allDist);
    }

    template void knnMatchL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);
    template void knnMatchL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, int knn, const DevMem2D& mask, const DevMem2Di& trainIdx, const DevMem2Df& distance, const DevMem2Df& allDist);

///////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// Radius Match //////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////
    
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match kernel

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    __global__ void radiusMatch(PtrStep_<T> queryDescs_, DevMem2D_<T> trainDescs_, 
        float maxDistance, Mask mask, DevMem2Di trainIdx_, unsigned int* nMatches, PtrStepf distance)
    {
        #if defined (__CUDA_ARCH__) && __CUDA_ARCH__ >= 110

        __shared__ float smem[BLOCK_DIM_X * BLOCK_DIM_Y];

        float* sdiff_row = smem + BLOCK_DIM_X * threadIdx.y;
        
        const int queryIdx = blockIdx.x;
        const T* queryDescs = queryDescs_.ptr(queryIdx);

        const int trainIdx = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;

        if (trainIdx < trainDescs_.rows)
        {
            const T* trainDescs = trainDescs_.ptr(trainIdx);

            if (mask(queryIdx, trainIdx))
            {
                Dist dist;

                reduceDescDiff<BLOCK_DIM_X>(queryDescs, trainDescs, trainDescs_.cols, dist, sdiff_row);

                if (threadIdx.x == 0)
                {
                    if (dist < maxDistance)
                    {
                        unsigned int i = atomicInc(nMatches + queryIdx, (unsigned int) -1);
                        if (i < trainIdx_.cols)
                        {
                            distance.ptr(queryIdx)[i] = dist;
                            trainIdx_.ptr(queryIdx)[i] = trainIdx;
                        }
                    }
                }
            }
        }

        #endif
    }
        
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match kernel caller

    template <int BLOCK_DIM_X, int BLOCK_DIM_Y, typename Dist, typename T, typename Mask>
    void radiusMatch_caller(const DevMem2D_<T>& queryDescs, const DevMem2D_<T>& trainDescs, 
        float maxDistance, const Mask& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, 
        const DevMem2Df& distance)
    {
        dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
        dim3 grid(queryDescs.rows, divUp(trainDescs.rows, BLOCK_DIM_Y), 1);

        radiusMatch<BLOCK_DIM_X, BLOCK_DIM_Y, Dist, T><<<grid, threads>>>(
            queryDescs, trainDescs, maxDistance, mask, trainIdx, nMatches, distance);
        cudaSafeCall( hipGetLastError() );

        cudaSafeCall( hipDeviceSynchronize() );
    }
    
    ///////////////////////////////////////////////////////////////////////////////
    // Radius Match kernel chooser

    template <typename T>
    void radiusMatchL1_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance,
        const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance)
    {
        if (mask.data)
        {
            radiusMatch_caller<16, 16, L1Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                maxDistance, SingleMask(mask), trainIdx, nMatches, distance);
        }
        else
        {
            radiusMatch_caller<16, 16, L1Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                maxDistance, WithOutMask(), trainIdx, nMatches, distance);
        }
    }

    template void radiusMatchL1_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL1_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL1_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL1_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL1_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL1_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);

    template <typename T>
    void radiusMatchL2_gpu(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance,
        const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance)
    {
        if (mask.data)
        {
            radiusMatch_caller<16, 16, L2Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                maxDistance, SingleMask(mask), trainIdx, nMatches, distance);
        }
        else
        {
            radiusMatch_caller<16, 16, L2Dist>((DevMem2D_<T>)queryDescs, (DevMem2D_<T>)trainDescs, 
                maxDistance, WithOutMask(), trainIdx, nMatches, distance);
        }
    }

    template void radiusMatchL2_gpu<uchar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL2_gpu<schar >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL2_gpu<ushort>(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL2_gpu<short >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL2_gpu<int   >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
    template void radiusMatchL2_gpu<float >(const DevMem2D& queryDescs, const DevMem2D& trainDescs, float maxDistance, const DevMem2D& mask, const DevMem2Di& trainIdx, unsigned int* nMatches, const DevMem2Df& distance);
}}}
